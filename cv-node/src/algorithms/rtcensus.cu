#include "hip/hip_runtime.h"
/*
 * Author: Nicolas Pope and Sebastian Hahta (2019)
 * Implementation of algorithm presented in article(s):
 *
 * [1] Humenberger, Engelke, Kubinger: A fast stereo matching algorithm suitable
 *     for embedded real-time systems
 * [2] Humenberger, Zinner, Kubinger: Performance Evaluation of Census-Based
 *     Stereo Matching Algorithm on Embedded and Multi-Core Hardware
 *
 * Equation numbering uses [1] unless otherwise stated
 *
 */
 
#include <opencv2/core/cuda/common.hpp>

using namespace cv::cuda;
using namespace cv;

#define BLOCK_W 128
#define RADIUS 7
#define RADIUS2 2
#define ROWSperTHREAD 20

#define XHI(P1,P2) ((P1 <= P2) ? 0 : 1)

namespace ftl {
namespace gpu {

__device__ uint64_t sparse_census(unsigned char *arr, size_t u, size_t v, size_t w) {
	uint64_t r = 0;

	unsigned char t = arr[v*w+u];

	for (int n=-7; n<=7; n+=2) {
	auto u_ = u + n;
	for (int m=-7; m<=7; m+=2) {
		auto v_ = v + m;
		r <<= 1;
		r |= XHI(t, arr[v_*w+u_]);
	}
	}

	return r;
}

__device__ float fit_parabola(size_t pi, uint16_t p, uint16_t pl, uint16_t pr) {
	float a = pr - pl;
	float b = 2 * (2 * p - pl - pr);
	return static_cast<float>(pi) + (a / b);
}

__global__ void census_kernel(PtrStepSzb l, PtrStepSzb r, uint64_t *census) {	
	//extern __shared__ uint64_t census[];
	
	size_t u = (blockIdx.x * BLOCK_W + threadIdx.x + RADIUS);
	size_t v_start = blockIdx.y * ROWSperTHREAD + RADIUS;
	size_t v_end = v_start + ROWSperTHREAD;
	
	if (v_end >= l.rows) v_end = l.rows;
	if (u >= l.cols) return;
	
	size_t width = l.cols;
	
	for (size_t v=v_start; v<v_end; v++) {
	//for (size_t u=7; u<width-7; u++) {
		size_t ix = (u + v*width) * 2;
		uint64_t cenL = sparse_census(l.data, u, v, l.step);
		uint64_t cenR = sparse_census(r.data, u, v, r.step);
		
		census[ix] = cenL;
		census[ix + 1] = cenR;
		
		//disp(v,u) = (float)cenL;
	//}
	}
	
	//__syncthreads();
	
	return;
}
	
__global__ void disp_kernel(float *disp_l, float *disp_r, size_t width, size_t height, uint64_t *census, size_t ds) {	
	//extern __shared__ uint64_t census[];
	
	size_t u = (blockIdx.x * BLOCK_W) + threadIdx.x + RADIUS2;
	size_t v_start = (blockIdx.y * ROWSperTHREAD) + RADIUS2;
	size_t v_end = v_start + ROWSperTHREAD;
	
	if (v_end >= height) v_end = height;
	//if (u >= width-ds) return;
	
	for (size_t v=v_start; v<v_end; v++) {
	//for (size_t u=7; u<width-7; u++) {
	//const size_t eu = (sign>0) ? w-2-ds : w-2;

	//for (size_t v=7; v<height-7; v++) {
	//for (size_t u=7; u<width-7; u++) {
		//const size_t ix = v*w*ds+u*ds;
		
		uint16_t last_ham[2] = {65535,65535};
		uint16_t min_disp[2] = {65535,65535};
		uint16_t min_before[2] = {0,0};
		uint16_t min_after[2] = {0,0};
		size_t dix[2] = {0,0};
		
		for (size_t d=0; d<ds; d++) {
			uint16_t hamming1 = 0;
			uint16_t hamming2 = 0;
			
			//if (u+2+ds >= width) break;
		
			for (int n=-2; n<=2; n++) {
				const auto u_ = u + n;

				for (int m=-2; m<=2; m++) {
					const auto v_ = (v + m)*width;

					// Correct for disp_R
					auto l1 = census[(u_+v_)*2+1];
					auto r1 = census[(v_+(u_+d))*2];
					
					// Correct for disp_L
					auto l2 = census[(u_+v_)*2];
					auto r2 = census[(v_+(u_-d))*2+1];
					
					hamming1 += __popcll(r1^l1);
					hamming2 += __popcll(r2^l2);
				}
			}
			
			if (hamming1 < min_disp[0]) {
				min_before[0] = last_ham[0];
				min_disp[0] = hamming1;
				dix[0] = d;
			}
			if (dix[0] == d) min_after[0] = hamming1;
			last_ham[0] = hamming1;
			
			if (hamming2 < min_disp[1]) {
				min_before[1] = last_ham[1];
				min_disp[1] = hamming2;
				dix[1] = d;
			}
			if (dix[1] == d) min_after[1] = hamming2;
			last_ham[1] = hamming2;
		
		}
		
		float d1 = (dix[0] == 0 || dix[0] == ds-1) ? (float)dix[0] : fit_parabola(dix[0], min_disp[0], min_before[0], min_after[0]);
		float d2 = (dix[1] == 0 || dix[1] == ds-1) ? (float)dix[1] : fit_parabola(dix[1], min_disp[1], min_before[1], min_after[1]);
	
		//if (abs(d1-d2) <= 1.0) disp(v,u) = abs((d1+d2)/2);
		//else disp(v,u) = 0.0f;
		
		//disp(v,u) = d1;
	
		disp_l[v*width+u] = d2;
		disp_r[v*width+u] = d1;
	}
}

__global__ void consistency_kernel(float *d_sub_l, float *d_sub_r, PtrStepSz<float> disp) {
	size_t w = disp.cols;
	size_t h = disp.rows;
	//Mat result = Mat::zeros(Size(w,h), CV_32FC1);
	
	size_t u = (blockIdx.x * BLOCK_W) + threadIdx.x + RADIUS;
	size_t v_start = (blockIdx.y * ROWSperTHREAD) + RADIUS;
	size_t v_end = v_start + ROWSperTHREAD;
	
	if (v_end >= disp.rows) v_end = disp.rows;
	if (u >= w) return;
	
	for (size_t v=v_start; v<v_end; v++) {
	
		int a = (int)(d_sub_l[v*w+u]);
		if ((int)u-a < 0) continue;
		
		auto b = d_sub_r[v*w+u-a];
		
		if (abs(a-b) <= 1.0) disp(v,u) = abs((a+b)/2);
		else disp(v,u) = 0.0f;
	//}
	}

}

/*__global__ void test_kernel(const PtrStepSzb l, const PtrStepSzb r, PtrStepSz<float> disp)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < l.cols && y < l.rows) {
		const unsigned char lv = l(y, x);
		const unsigned char rv = r(y, x);
		disp(y, x) = (float)lv - (float)rv; //make_uchar1(v.z, v.y, v.x);
	}
}*/

void rtcensus_call(const PtrStepSzb &l, const PtrStepSzb &r, const PtrStepSz<float> &disp, size_t num_disp, const int &stream) {
	dim3 grid(1,1,1);
    dim3 threads(BLOCK_W, 1, 1);

	grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS, BLOCK_W);
	grid.y = cv::cuda::device::divUp(l.rows - 2 * RADIUS, ROWSperTHREAD);
	
	// TODO, reduce allocations
	uint64_t *census;
	float *disp_l;
	float *disp_r;
	hipMalloc(&census, sizeof(uint64_t)*l.cols*l.rows*2);
	//hipMemset(census, 0, sizeof(uint64_t)*l.cols*l.rows*2);
	hipMalloc(&disp_l, sizeof(float)*l.cols*l.rows);
	hipMalloc(&disp_r, sizeof(float)*l.cols*l.rows);
	
	//size_t smem_size = (2 * l.cols * l.rows) * sizeof(uint64_t);
	
	census_kernel<<<grid, threads>>>(l, r, census);
	cudaSafeCall( hipGetLastError() );
	
	grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS2, BLOCK_W);
	grid.y = cv::cuda::device::divUp(l.rows - 2 * RADIUS2, ROWSperTHREAD);
	
	//grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS - num_disp, BLOCK_W) - 1;
	disp_kernel<<<grid, threads>>>(disp_l, disp_r, l.cols, l.rows, census, num_disp);
	cudaSafeCall( hipGetLastError() );
	
	consistency_kernel<<<grid, threads>>>(disp_l, disp_r, disp);
	cudaSafeCall( hipGetLastError() );
	
	hipFree(disp_r);
	hipFree(disp_l);
	hipFree(census);
	
	//if (&stream == Stream::Null())
		cudaSafeCall( hipDeviceSynchronize() );
}

};
};
