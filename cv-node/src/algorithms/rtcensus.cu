#include "hip/hip_runtime.h"
/*
 * Author: Nicolas Pope and Sebastian Hahta (2019)
 * Implementation of algorithm presented in article(s):
 *
 * [1] Humenberger, Engelke, Kubinger: A fast stereo matching algorithm suitable
 *     for embedded real-time systems
 * [2] Humenberger, Zinner, Kubinger: Performance Evaluation of Census-Based
 *     Stereo Matching Algorithm on Embedded and Multi-Core Hardware
 *
 * Equation numbering uses [1] unless otherwise stated
 *
 */
 
#include <opencv2/core/cuda/common.hpp>

using namespace cv::cuda;
using namespace cv;

#define BLOCK_W 60
#define RADIUS 7
#define RADIUS2 2
#define ROWSperTHREAD 1

#define XHI(P1,P2) ((P1 <= P2) ? 0 : 1)

namespace ftl {
namespace gpu {

// --- SUPPORT -----------------------------------------------------------------

template <typename T>
hipTextureObject_t makeTexture2D(const PtrStepSzb &d) {
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = d.data;
	resDesc.res.pitch2D.pitchInBytes = d.step;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<T>();
	resDesc.res.pitch2D.width = d.cols;
	resDesc.res.pitch2D.height = d.rows;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex = 0;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
	return tex;
}

template <typename T>
hipTextureObject_t makeTexture2D(void *ptr, int pitch, int width, int height) {
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = ptr;
	resDesc.res.pitch2D.pitchInBytes = pitch;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc<T>();
	resDesc.res.pitch2D.width = width;
	resDesc.res.pitch2D.height = height;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex = 0;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
	return tex;
}

/*
 * Sparse 16x16 census (so 8x8) creating a 64bit mask
 * (14) & (15), based upon (9)
 */
__device__ uint64_t sparse_census(hipTextureObject_t tex, int u, int v) {
	uint64_t r = 0;

	unsigned char t = tex2D<unsigned char>(tex, u,v);

	for (int m=-7; m<=7; m+=2) {
		//auto start_ix = (v + m)*w + u;
		for (int n=-7; n<=7; n+=2) {
			r <<= 1;
			r |= XHI(t, tex2D<unsigned char>(tex, u+n, v+m));
		}
	}

	return r;
}

/*
 * Parabolic interpolation between matched disparities either side.
 * Results in subpixel disparity. (20).
 */
__device__ float fit_parabola(size_t pi, uint16_t p, uint16_t pl, uint16_t pr) {
	float a = pr - pl;
	float b = 2 * (2 * p - pl - pr);
	return static_cast<float>(pi) + (a / b);
}

// --- KERNELS -----------------------------------------------------------------

/*
 * Calculate census mask for left and right images together.
 */
__global__ void census_kernel(hipTextureObject_t l, hipTextureObject_t r,
		int w, int h, uint64_t *censusL, uint64_t *censusR,
		size_t pL, size_t pR) {	
	
	int u = (blockIdx.x * BLOCK_W + threadIdx.x + RADIUS);
	int v_start = blockIdx.y * ROWSperTHREAD + RADIUS;
	int v_end = v_start + ROWSperTHREAD;
	
	if (v_end+RADIUS >= h) v_end = h-RADIUS;
	if (u+RADIUS >= w) return;
	
	for (int v=v_start; v<v_end; v++) {
		//int ix = (u + v*pL);
		uint64_t cenL = sparse_census(l, u, v);
		uint64_t cenR = sparse_census(r, u, v);
		
		censusL[(u + v*pL)] = cenL;
		censusR[(u + v*pR)] = cenR;
	}
}

/* Convert vector uint2 (32bit x2) into a single uint64_t */
__forceinline__ __device__ uint64_t uint2asull (uint2 a) {
	uint64_t res;
	asm ("mov.b64 %0, {%1,%2};" : "=l"(res) : "r"(a.x), "r"(a.y));
	return res;
}

/*
 * Generate left and right disparity images from census data. (19)
 */
__global__ void disp_kernel(float *disp_l, float *disp_r,
		int pitchL, int pitchR,
		size_t width, size_t height,
		hipTextureObject_t censusL, hipTextureObject_t censusR,
		size_t ds) {	
	//extern __shared__ uint64_t cache[];

	const int gamma = 10;
	
	int u = (blockIdx.x * BLOCK_W) + threadIdx.x + RADIUS2;
	int v_start = (blockIdx.y * ROWSperTHREAD) + RADIUS2;
	int v_end = v_start + ROWSperTHREAD;
	int maxdisp = ds;
	
	// Local cache
	uint64_t l_cache_l1[5][5];
	uint64_t l_cache_l2[5][5];

	// Prepare the cache load
	//const int cache_thread_width = (BLOCK_W+ds / BLOCK_W + RADIUS2*2 + 1)*2;
	//uint64_t *cache_ptr = cache + (threadIdx.x * cache_thread_width);
	
	if (v_end >= height) v_end = height;
	if (u+maxdisp >= width) maxdisp = width-u;
	
	for (int v=v_start; v<v_end; v++) {
		/*const int cache_start = v*width*2 + cache_thread_width*blockIdx.x;
		for (int i=0; i<cache_thread_width; i+=2) {
			cache_ptr[i] = census[cache_start+i];
			cache_ptr[i+1] = census[cache_start+i+1];
		}
		
		__syncthreads();*/
		
		// Fill local cache for window 5x5
		// TODO Use shared memory?
		for (int m=-2; m<=2; m++) {
			for (int n=-2; n<=2; n++) {
				l_cache_l2[m+2][n+2] = uint2asull(tex2D<uint2>(censusL,u+n,v+m));
				l_cache_l1[m+2][n+2] = uint2asull(tex2D<uint2>(censusR,u+n,v+m));
			}
		}
		
		uint16_t last_ham1 = 65535;
		uint16_t last_ham2 = 65535;
		uint16_t min_disp1 = 65535;
		uint16_t min_disp2 = 65535;
		uint16_t min_disp1b = 65535;
		uint16_t min_disp2b = 65535;
		uint16_t min_before1 = 0;
		uint16_t min_before2 = 0;
		uint16_t min_after1 = 0;
		uint16_t min_after2 = 0;
		int dix1 = 0;
		int dix2 = 0;
		
		// TODO Use prediction textures to narrow range
		for (int d=0; d<maxdisp; d++) {
			uint16_t hamming1 = 0;
			uint16_t hamming2 = 0;
			
			//if (u+2+ds >= width) break;
		
			for (int m=-2; m<=2; m++) {
				const auto v_ = (v + m);
				for (int n=-2; n<=2; n++) {
					const auto u_ = u + n;
					
					auto l1 = l_cache_l1[m+2][n+2];
					auto l2 = l_cache_l2[m+2][n+2];
					
					// TODO Somehow might use shared memory
					auto r1 = uint2asull(tex2D<uint2>(censusL, u_+d, v_));
					auto r2 = uint2asull(tex2D<uint2>(censusR, u_-d, v_));
					
					hamming1 += __popcll(r1^l1);
					hamming2 += __popcll(r2^l2);
				}
			}
			
			if (hamming1 < min_disp1) {
				min_before1 = last_ham1;
				min_disp1 = hamming1;
				dix1 = d;
			} else if (hamming1 < min_disp1b) {
				min_disp1b = hamming1;
			}
			if (dix1 == d) min_after1 = hamming1;
			last_ham1 = hamming1;
			
			if (hamming2 < min_disp2) {
				min_before2 = last_ham2;
				min_disp2 = hamming2;
				dix2 = d;
			} else if (hamming2 < min_disp2b) {
				min_disp2b = hamming2;
			}
			if (dix2 == d) min_after2 = hamming2;
			last_ham2 = hamming2;
		
		}
		
		//float d1 = (dix1 == 0 || dix1 == ds-1) ? (float)dix1 : fit_parabola(dix1, min_disp1, min_before1, min_after1);
		//float d2 = (dix2 == 0 || dix2 == ds-1) ? (float)dix2 : fit_parabola(dix2, min_disp2, min_before2, min_after2);
	
		// TODO Allow for discontinuities with threshold
		float d1 = fit_parabola(dix1, min_disp1, min_before1, min_after1);
		float d2 = fit_parabola(dix2, min_disp2, min_before2, min_after2);
	
		// Confidence filter (25)
		// TODO choice of gamma to depend on disparity variance
		// Variance with next option, variance with neighbours, variance with past value
		disp_l[v*pitchL+u] = ((min_disp2b - min_disp2) >= gamma) ? d2 : NAN;
		disp_r[v*pitchR+u] = ((min_disp1b - min_disp1) >= gamma) ? d1 : NAN;

		// TODO If disparity is 0.0f, perhaps
		// Use previous value unless it conflicts with present
		// Use neighbour values if texture matches 
	}
}

__global__ void consistency_kernel(hipTextureObject_t d_sub_l,
		hipTextureObject_t d_sub_r, float *disp, int w, int h, int pitch) {
	//Mat result = Mat::zeros(Size(w,h), CV_32FC1);
	
	int u = (blockIdx.x * BLOCK_W) + threadIdx.x + RADIUS;
	int v_start = (blockIdx.y * ROWSperTHREAD) + RADIUS;
	int v_end = v_start + ROWSperTHREAD;
	
	if (v_end >= h) v_end = h;
	if (u >= w) return;
	
	for (int v=v_start; v<v_end; v++) {
	
		float a = (int)tex2D<float>(d_sub_l, u, v);
		if (u-a < 0) continue;
		
		auto b = tex2D<float>(d_sub_r, u-a, v);

		//disp(v,u) = a; //abs((a+b)/2);
		
		if (abs(a-b) <= 1.0) disp[v*pitch+u] = abs((a+b)/2); // was 1.0
		else disp[v*pitch+u] = NAN;
	//}
	}

}

#define FILTER_WINDOW_R	7
#define FILTER_SIM_THRESH 10

__global__ void filter_kernel(hipTextureObject_t t, hipTextureObject_t d, PtrStepSz<float> f) {
	size_t u = (blockIdx.x * BLOCK_W) + threadIdx.x + RADIUS;
	size_t v = blockIdx.y;

	float disp = tex2D<float>(d,u,v);
	if (!isnan(disp)) {
		f(v,u) = disp;
		return;
	}

	int pixel = tex2D<unsigned char>(t, u, v);
	float est = 0.0f;
	int nn = 0;

	for (int m=-FILTER_WINDOW_R; m<=FILTER_WINDOW_R; m++) {
		for (int n=-FILTER_WINDOW_R; n<=FILTER_WINDOW_R; n++) {
			int neigh = tex2D<unsigned char>(t, u+n, v+m);
			float ndisp = tex2D<float>(d,u+n,v+m);

			if (!isnan(ndisp) && (abs(neigh-pixel) <= FILTER_SIM_THRESH)) {
				est += ndisp;
				nn++;
			}
		}	
	}

	f(v,u) = (nn==0) ? NAN : est / nn;
}

void rtcensus_call(const PtrStepSzb &l, const PtrStepSzb &r, const PtrStepSz<float> &disp, size_t num_disp, const int &stream) {
	dim3 grid(1,1,1);
    dim3 threads(BLOCK_W, 1, 1);

	grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS, BLOCK_W);
	grid.y = cv::cuda::device::divUp(l.rows - 2 * RADIUS, ROWSperTHREAD);
	
	// TODO, reduce allocations
	uint64_t *censusL;
	uint64_t *censusR;
	size_t pitchL;
	size_t pitchR;

	float *disp_l;
	float *disp_r;
	size_t pitchDL;
	size_t pitchDR;

	float *disp_raw;
	size_t pitchD;
	
	cudaSafeCall( hipMallocPitch(&censusL, &pitchL, l.cols*sizeof(uint64_t), l.rows) );
	cudaSafeCall( hipMallocPitch(&censusR, &pitchR, r.cols*sizeof(uint64_t), r.rows) );
	
	//hipMemset(census, 0, sizeof(uint64_t)*l.cols*l.rows*2);
	cudaSafeCall( hipMallocPitch(&disp_l, &pitchDL, sizeof(float)*l.cols, l.rows) );
	cudaSafeCall( hipMallocPitch(&disp_r, &pitchDR, sizeof(float)*l.cols, l.rows) );

	cudaSafeCall( hipMallocPitch(&disp_raw, &pitchD, sizeof(float)*l.cols, l.rows) );
	
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
  
	hipTextureObject_t texLeft = makeTexture2D<unsigned char>(l);
	hipTextureObject_t texRight = makeTexture2D<unsigned char>(r);

	//size_t smem_size = (2 * l.cols * l.rows) * sizeof(uint64_t);
	
	// Calculate L and R census
	census_kernel<<<grid, threads>>>(texLeft, texRight, l.cols, l.rows, censusL, censusR, pitchL/sizeof(uint64_t), pitchR/sizeof(uint64_t));
	cudaSafeCall( hipGetLastError() );
	
	//cudaSafeCall( hipDeviceSynchronize() );
  
	hipTextureObject_t censusTexLeft = makeTexture2D<uint2>(censusL, pitchL, l.cols, l.rows);
	hipTextureObject_t censusTexRight = makeTexture2D<uint2>(censusR, pitchR, r.cols, r.rows);
	
	grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS2, BLOCK_W);
	grid.y = cv::cuda::device::divUp(l.rows - 2 * RADIUS2, ROWSperTHREAD);
	
	// Calculate L and R disparities
	disp_kernel<<<grid, threads>>>(disp_l, disp_r, pitchDL/sizeof(float), pitchDR/sizeof(float), l.cols, l.rows, censusTexLeft, censusTexRight, num_disp);
	cudaSafeCall( hipGetLastError() );

	hipTextureObject_t dispTexLeft = makeTexture2D<float>(disp_l, pitchDL, l.cols, l.rows);
	hipTextureObject_t dispTexRight = makeTexture2D<float>(disp_r, pitchDR, r.cols, r.rows);
	
	// Check consistency between L and R disparities.
	consistency_kernel<<<grid, threads>>>(dispTexLeft, dispTexRight, disp_raw, l.cols, l.rows, pitchD/sizeof(float));
	cudaSafeCall( hipGetLastError() );

	hipTextureObject_t dispTex = makeTexture2D<float>(disp_raw, pitchD, r.cols, r.rows);

	filter_kernel<<<grid, threads>>>(texLeft, dispTex, disp);
	cudaSafeCall( hipGetLastError() );
	
	//if (&stream == Stream::Null())
	cudaSafeCall( hipDeviceSynchronize() );
		
	cudaSafeCall( hipDestroyTextureObject (texLeft) );
	cudaSafeCall( hipDestroyTextureObject (texRight) );
	cudaSafeCall( hipDestroyTextureObject (censusTexLeft) );
	cudaSafeCall( hipDestroyTextureObject (censusTexRight) );
	cudaSafeCall( hipDestroyTextureObject (dispTexLeft) );
	cudaSafeCall( hipDestroyTextureObject (dispTexRight) );
	cudaSafeCall( hipDestroyTextureObject (dispTex) );
	hipFree(disp_r);
	hipFree(disp_l);
	hipFree(censusL);
	hipFree(censusR);
}

};
};
