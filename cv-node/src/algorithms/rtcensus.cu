#include "hip/hip_runtime.h"
/*
 * Author: Nicolas Pope and Sebastian Hahta (2019)
 * Implementation of algorithm presented in article(s):
 *
 * [1] Humenberger, Engelke, Kubinger: A fast stereo matching algorithm suitable
 *     for embedded real-time systems
 * [2] Humenberger, Zinner, Kubinger: Performance Evaluation of Census-Based
 *     Stereo Matching Algorithm on Embedded and Multi-Core Hardware
 *
 * Equation numbering uses [1] unless otherwise stated
 *
 */
 
#include <opencv2/core/cuda/common.hpp>

using namespace cv::cuda;
using namespace cv;

#define BLOCK_W 60
#define RADIUS 7
#define RADIUS2 2
#define ROWSperTHREAD 2

#define XHI(P1,P2) ((P1 <= P2) ? 0 : 1)

namespace ftl {
namespace gpu {

// --- SUPPORT -----------------------------------------------------------------

/*
 * Sparse 16x16 census (so 8x8) creating a 64bit mask
 * (14) & (15), based upon (9)
 */
__device__ uint64_t sparse_census(unsigned char *arr, size_t u, size_t v, size_t w) {
	uint64_t r = 0;

	unsigned char t = arr[v*w+u];

	for (int m=-7; m<=7; m+=2) {
		auto start_ix = (v + m)*w + u;
		for (int n=-7; n<=7; n+=2) {
			r <<= 1;
			r |= XHI(t, arr[start_ix+n]);
		}
	}

	return r;
}

/*
 * Parabolic interpolation between matched disparities either side.
 * Results in subpixel disparity. (20).
 */
__device__ float fit_parabola(size_t pi, uint16_t p, uint16_t pl, uint16_t pr) {
	float a = pr - pl;
	float b = 2 * (2 * p - pl - pr);
	return static_cast<float>(pi) + (a / b);
}

// --- KERNELS -----------------------------------------------------------------

/*
 * Calculate census mask for left and right images together.
 */
__global__ void census_kernel(PtrStepSzb l, PtrStepSzb r, uint64_t *census) {	
	//extern __shared__ uint64_t census[];
	
	size_t u = (blockIdx.x * BLOCK_W + threadIdx.x + RADIUS);
	size_t v_start = blockIdx.y * ROWSperTHREAD + RADIUS;
	size_t v_end = v_start + ROWSperTHREAD;
	
	if (v_end >= l.rows) v_end = l.rows;
	if (u >= l.cols) return;
	
	size_t width = l.cols;
	
	for (size_t v=v_start; v<v_end; v++) {
		size_t ix = (u + v*width) * 2;
		uint64_t cenL = sparse_census(l.data, u, v, l.step);
		uint64_t cenR = sparse_census(r.data, u, v, r.step);
		
		census[ix] = cenL;
		census[ix + 1] = cenR;
	}
}

/*
 * Generate left and right disparity images from census data. (19)
 */
__global__ void disp_kernel(float *disp_l, float *disp_r, size_t width, size_t height, uint64_t *census, size_t ds) {	
	//extern __shared__ uint64_t cache[];

	const int gamma = 100;
	
	size_t u = (blockIdx.x * BLOCK_W) + threadIdx.x + RADIUS2;
	size_t v_start = (blockIdx.y * ROWSperTHREAD) + RADIUS2;
	size_t v_end = v_start + ROWSperTHREAD;

	// Prepare the cache load
	//const int cache_thread_width = (BLOCK_W+ds / BLOCK_W + RADIUS2*2 + 1)*2;
	//uint64_t *cache_ptr = cache + (threadIdx.x * cache_thread_width);
	
	if (v_end >= height) v_end = height;
	//if (u >= width-ds) return;
	
	for (size_t v=v_start; v<v_end; v++) {
		/*const int cache_start = v*width*2 + cache_thread_width*blockIdx.x;
		for (int i=0; i<cache_thread_width; i+=2) {
			cache_ptr[i] = census[cache_start+i];
			cache_ptr[i+1] = census[cache_start+i+1];
		}
		
		__syncthreads();*/
		
		uint16_t last_ham1 = 65535;
		uint16_t last_ham2 = 65535;
		uint16_t min_disp1 = 65535;
		uint16_t min_disp2 = 65535;
		uint16_t min_disp1b = 65535;
		uint16_t min_disp2b = 65535;
		uint16_t min_before1 = 0;
		uint16_t min_before2 = 0;
		uint16_t min_after1 = 0;
		uint16_t min_after2 = 0;
		int dix1 = 0;
		int dix2 = 0;
		
		// TODO Use prediction textures to narrow range
		for (size_t d=0; d<ds; d++) {
			uint16_t hamming1 = 0;
			uint16_t hamming2 = 0;
			
			//if (u+2+ds >= width) break;
		
			for (int m=-2; m<=2; m++) {
				const auto v_ = (v + m)*width;
				for (int n=-2; n<=2; n++) {
					const auto u_ = u + n;

				
					

					auto l2 = census[(u_+v_)*2];
					auto l1 = census[(u_+v_)*2+1];
					
					auto r1 = census[(v_+(u_+d))*2];
					auto r2 = census[(v_+(u_-d))*2+1];
					
					hamming1 += __popcll(r1^l1);
					hamming2 += __popcll(r2^l2);
				}
			}
			
			if (hamming1 < min_disp1) {
				min_before1 = last_ham1;
				min_disp1 = hamming1;
				dix1 = d;
			} else if (hamming1 < min_disp1b) {
				min_disp1b = hamming1;
			}
			if (dix1 == d) min_after1 = hamming1;
			last_ham1 = hamming1;
			
			if (hamming2 < min_disp2) {
				min_before2 = last_ham2;
				min_disp2 = hamming2;
				dix2 = d;
			} else if (hamming2 < min_disp2b) {
				min_disp2b = hamming2;
			}
			if (dix2 == d) min_after2 = hamming2;
			last_ham2 = hamming2;
		
		}
		
		//float d1 = (dix1 == 0 || dix1 == ds-1) ? (float)dix1 : fit_parabola(dix1, min_disp1, min_before1, min_after1);
		//float d2 = (dix2 == 0 || dix2 == ds-1) ? (float)dix2 : fit_parabola(dix2, min_disp2, min_before2, min_after2);
	
		// TODO Allow for discontinuities with threshold
		float d1 = fit_parabola(dix1, min_disp1, min_before1, min_after1);
		float d2 = fit_parabola(dix2, min_disp2, min_before2, min_after2);
	
		// Confidence filter (25)
		// TODO choice of gamma to depend on disparity variance
		// Variance with next option, variance with neighbours, variance with past value
		disp_l[v*width+u] = ((min_disp2b - min_disp2) >= gamma) ? d2 : INFINITY;
		disp_r[v*width+u] = ((min_disp1b - min_disp1) >= gamma) ? d1 : INFINITY;

		// TODO If disparity is 0.0f, perhaps
		// Use previous value unless it conflicts with present
		// Use neighbour values if texture matches 
	}
}

__global__ void consistency_kernel(float *d_sub_l, float *d_sub_r, PtrStepSz<float> disp) {
	size_t w = disp.cols;
	size_t h = disp.rows;
	//Mat result = Mat::zeros(Size(w,h), CV_32FC1);
	
	size_t u = (blockIdx.x * BLOCK_W) + threadIdx.x + RADIUS;
	size_t v_start = (blockIdx.y * ROWSperTHREAD) + RADIUS;
	size_t v_end = v_start + ROWSperTHREAD;
	
	if (v_end >= disp.rows) v_end = disp.rows;
	if (u >= w) return;
	
	for (size_t v=v_start; v<v_end; v++) {
	
		int a = (int)(d_sub_l[v*w+u]);
		if ((int)u-a < 0) continue;
		
		auto b = d_sub_r[v*w+u-a];
		
		if (abs(a-b) <= 1.0) disp(v,u) = abs((a+b)/2); // was 1.0
		else disp(v,u) = INFINITY;
	//}
	}

}

void rtcensus_call(const PtrStepSzb &l, const PtrStepSzb &r, const PtrStepSz<float> &disp, size_t num_disp, const int &stream) {
	dim3 grid(1,1,1);
    dim3 threads(BLOCK_W, 1, 1);

	grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS, BLOCK_W);
	grid.y = cv::cuda::device::divUp(l.rows - 2 * RADIUS, ROWSperTHREAD);
	
	// TODO, reduce allocations
	uint64_t *census;
	float *disp_l;
	float *disp_r;
	hipMalloc(&census, sizeof(uint64_t)*l.cols*l.rows*2);
	//hipMemset(census, 0, sizeof(uint64_t)*l.cols*l.rows*2);
	hipMalloc(&disp_l, sizeof(float)*l.cols*l.rows);
	hipMalloc(&disp_r, sizeof(float)*l.cols*l.rows);
	
	//size_t smem_size = (2 * l.cols * l.rows) * sizeof(uint64_t);
	
	census_kernel<<<grid, threads>>>(l, r, census);
	cudaSafeCall( hipGetLastError() );
	
	grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS2, BLOCK_W);
	grid.y = cv::cuda::device::divUp(l.rows - 2 * RADIUS2, ROWSperTHREAD);
	
	//grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS - num_disp, BLOCK_W) - 1;
	disp_kernel<<<grid, threads>>>(disp_l, disp_r, l.cols, l.rows, census, num_disp);
	cudaSafeCall( hipGetLastError() );
	
	consistency_kernel<<<grid, threads>>>(disp_l, disp_r, disp);
	cudaSafeCall( hipGetLastError() );
	
	hipFree(disp_r);
	hipFree(disp_l);
	hipFree(census);
	
	//if (&stream == Stream::Null())
		cudaSafeCall( hipDeviceSynchronize() );
}

};
};
