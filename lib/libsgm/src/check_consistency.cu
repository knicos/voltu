#include "hip/hip_runtime.h"
/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <libsgm.h>
#include "internal.h"

namespace {
	template<typename SRC_T, typename DST_T>
	__global__ void check_consistency_kernel(DST_T* d_leftDisp, const DST_T* d_rightDisp, const uint8_t* d_mask, int width, int height, int src_pitch, int dst_pitch, bool subpixel)  {

		const int j = blockIdx.x * blockDim.x + threadIdx.x;
		const int i = blockIdx.y * blockDim.y + threadIdx.y;

		// left-right consistency check, only on leftDisp, but could be done for rightDisp too

		uint8_t mask = d_mask[i * src_pitch + j];
		int d = d_leftDisp[i * dst_pitch + j];
		if (subpixel) {
			d >>= sgm::StereoSGM::SUBPIXEL_SHIFT;
		}
		int k = j - d;
		if (mask != 0 || d <= 0 || (k >= 0 && k < width)) {
			int diff = abs(d_rightDisp[i * dst_pitch + k] - d);
			if (mask != 0 || diff > 1) {
				// masked or left-right inconsistent pixel -> invalid
				d_leftDisp[i * dst_pitch + j] = (256 << (sgm::StereoSGM::SUBPIXEL_SHIFT+1));
			}
		}
	}
}

namespace sgm {
	namespace details {

		void check_consistency(uint8_t* d_left_disp, const uint8_t* d_right_disp, const uint8_t* d_mask, int width, int height, int depth_bits, int src_pitch, int dst_pitch, bool subpixel, hipStream_t stream) {

			const dim3 blocks(width / 16, height / 16);
			const dim3 threads(16, 16);
			if (depth_bits == 16) {
				check_consistency_kernel<uint16_t> << < blocks, threads, 0, stream >> > (d_left_disp, d_right_disp, d_mask, width, height, src_pitch, dst_pitch, subpixel);
			}
			else if (depth_bits == 8) {
				check_consistency_kernel<uint8_t> << < blocks, threads, 0, stream >> > (d_left_disp, d_right_disp, d_mask, width, height, src_pitch, dst_pitch, subpixel);
			}

			CudaKernelCheck();
		}

		void check_consistency(uint16_t* d_left_disp, const uint16_t* d_right_disp, const uint8_t* d_mask, int width, int height, int depth_bits, int src_pitch, int dst_pitch, bool subpixel, hipStream_t stream) {

			const dim3 blocks(width / 16, height / 16);
			const dim3 threads(16, 16);
			if (depth_bits == 16) {
				check_consistency_kernel<uint16_t> << < blocks, threads, 0, stream >> > (d_left_disp, d_right_disp, d_mask, width, height, src_pitch, dst_pitch, subpixel);
			}
			else if (depth_bits == 8) {
				check_consistency_kernel<uint8_t> << < blocks, threads, 0, stream >> > (d_left_disp, d_right_disp, d_mask, width, height, src_pitch, dst_pitch, subpixel);
			}
			
			CudaKernelCheck();	
		}

	}
}
