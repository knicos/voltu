#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>

#include <opencv2/core/cuda/common.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/highgui.hpp>

#include "stereo.hpp"

#include "util_opencv.hpp"
#include "costs/census.hpp"
#include "dsi.hpp"

#include "wta.hpp"
#include "cost_aggregation.hpp"
#include "aggregations/cp_sgm.hpp"

#include "median_filter.hpp"
#include "dsi_tools.hpp"

#ifdef __GNUG__

#include <chrono>
#include <iostream>

static std::chrono::time_point<std::chrono::system_clock> start;

static void timer_set() {
		start = std::chrono::high_resolution_clock::now();
}

static void timer_print(const std::string &msg, const bool reset=true) {
	auto stop = std::chrono::high_resolution_clock::now();

	char buf[24];
	snprintf(buf, sizeof(buf), "%5i ms  ",
				(int) std::chrono::duration_cast<std::chrono::milliseconds>(stop-start).count());

	std::cout << buf <<  msg << "\n" << std::flush;
	if (reset) { timer_set(); }
}

#else

static void timer_set() {}
static void timer_print(const std::string &msg, const bool reset=true) {}

#endif

using cv::Mat;
using cv::Size;
using ftl::stereo::aggregations::ConfidencePriorSGM;

typedef CensusMatchingCost MatchingCost;

struct StereoCPCensusSgm::Impl {
	MatchingCost cost;
	Array2D<MatchingCost::Type> cost_min_paths;
	Array2D<MatchingCost::Type> uncertainty;
	Array2D<uchar> l;
    Array2D<uchar> r;
	Array2D<uchar> conf_prior;
	Array2D<float> disparity;

	PathAggregator<ConfidencePriorSGM<MatchingCost::DataType>> aggr;
	WinnerTakesAll<DisparitySpaceImage<MatchingCost::Type>,float> wta;

	Impl(int width, int height, int min_disp, int max_disp) :
		cost(width, height, min_disp, max_disp),
		cost_min_paths(width, height),
		uncertainty(width, height),
        l(width, height), r(width, height),
		conf_prior(width, height),
		disparity(width, height)
		{}

};

StereoCPCensusSgm::StereoCPCensusSgm() : impl_(nullptr) {
    impl_ = new Impl(0, 0, 0, 0);
}

void StereoCPCensusSgm::compute(cv::InputArray l, cv::InputArray r, cv::OutputArray disparity) {
	hipSetDevice(0);

	if (l.rows() != impl_->cost.height() || r.cols() != impl_->cost.width()) {
		delete impl_; impl_ = nullptr;
        impl_ = new Impl(l.cols(), l.rows(), params.d_min, params.d_max);
        impl_->conf_prior.toGpuMat().setTo(0);
	}

	mat2gray(l, impl_->l);
	mat2gray(r, impl_->r);
	timer_set();

	// CT
	impl_->cost.set(impl_->l, impl_->r);

	cudaSafeCall(hipDeviceSynchronize());
	if (params.debug) { timer_print("census transform"); }

	// cost aggregation
	ConfidencePriorSGM<MatchingCost::DataType> func = {impl_->cost.data(), impl_->cost_min_paths.data(), impl_->conf_prior.data(), params.P1, params.P2};
	auto &out = impl_->aggr(func, params.paths);

	cudaSafeCall(hipDeviceSynchronize());
	if (params.debug) { timer_print("Aggregation"); }

	impl_->wta(out, params.subpixel, params.lr_consistency);
	cudaSafeCall(hipDeviceSynchronize());
	if (params.debug) { timer_print("WTA"); }

	// Drory, A., Haubold, C., Avidan, S., & Hamprecht, F. A. (2014).
	// Semi-global matching: A principled derivation in terms of
	// message passing. Lecture Notes in Computer Science (Including Subseries
	// Lecture Notes in Artificial Intelligence and Lecture Notes in
	// Bioinformatics). https://doi.org/10.1007/978-3-319-11752-2_4

	Array2D<MatchingCost::Type> dsitmp_dev(l.cols(), l.rows());

	#if USE_GPU
	auto uncertainty = impl_->uncertainty.toGpuMat();
	cv::cuda::subtract(impl_->wta.min_cost.toGpuMat(), impl_->cost_min_paths.toGpuMat(), uncertainty);
	cv::cuda::compare(uncertainty, params.uniqueness, uncertainty, cv::CMP_GT);
	impl_->wta.disparity.toGpuMat().setTo(0, uncertainty);
	dsi_slice(out, impl_->wta.disparity, dsitmp_dev);
	cv::cuda::compare(dsitmp_dev.toGpuMat(), float(params.P2)+2.0f*(float(MatchingCost::COST_MAX)*0.5f), uncertainty, cv::CMP_GT);
	if (params.filtercosts) impl_->wta.disparity.toGpuMat().setTo(0, uncertainty);
	#else
	auto uncertainty = impl_->uncertainty.toMat();
	cv::subtract(impl_->wta.min_cost.toMat(), impl_->cost_min_paths.toMat(), uncertainty);
	cv::compare(uncertainty, params.uniqueness, uncertainty, cv::CMP_GT);
	impl_->wta.disparity.toMat().setTo(0, uncertainty);
	#endif

	median_filter(impl_->wta.disparity, impl_->disparity.toGpuMat());
	impl_->wta.disparity.toGpuMat().convertTo(impl_->conf_prior.toGpuMat(), CV_8UC1);
	impl_->disparity.toGpuMat().download(disparity);
	if (params.debug) { timer_print("median filter"); }

	show_dsi_slice(dsitmp_dev.toGpuMat());
}

StereoCPCensusSgm::~StereoCPCensusSgm() {
	if (impl_) {
		delete impl_;
		impl_ = nullptr;
	}
}
