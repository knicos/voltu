#include "stereo.hpp"
#include "stereosgm.hpp"
#include "../costs/census.hpp"

struct StereoCensusSgm::Impl : public StereoSgm<CensusMatchingCost, StereoCensusSgm::Parameters> {
	Array2D<uchar> l;
	Array2D<uchar> r;

	Impl(StereoCensusSgm::Parameters &params, int width, int height, int dmin, int dmax) :
		StereoSgm(params, width, height, dmin, dmax), l(width, height), r(width, height) {}
};

StereoCensusSgm::StereoCensusSgm() : impl_(nullptr) {
	impl_ = new Impl(params, 0, 0, 0, 0);
}

void StereoCensusSgm::compute(cv::InputArray l, cv::InputArray r, cv::OutputArray disparity) {

	//hipSetDevice(0);

	if (l.rows() != impl_->cost.height() || r.cols() != impl_->cost.width()) {
		delete impl_; impl_ = nullptr;
		impl_ = new Impl(params, l.cols(), l.rows(), params.d_min, params.d_max);
	}

	mat2gray(l, impl_->l);
	mat2gray(r, impl_->r);
	impl_->cost.setPattern(params.pattern);
	impl_->cost.set(impl_->l, impl_->r);

	cudaSafeCall(hipDeviceSynchronize());
	impl_->compute(disparity);
}

StereoCensusSgm::~StereoCensusSgm() {
	if (impl_) {
		delete impl_;
		impl_ = nullptr;
	}
}
