#include "stereo.hpp"
#include "stereosgm.hpp"
#include "../costs/tcensus.hpp"

struct StereoTCensusSgm::Impl : public StereoSgm<TCensusMatchingCost, StereoTCensusSgm::Parameters> {
	Array2D<uchar> l;
	Array2D<uchar> r;

	Impl(StereoTCensusSgm::Parameters &params, int width, int height, int dmin, int dmax) :
		StereoSgm(params, width, height, dmin, dmax), l(width, height), r(width, height) {}
};

StereoTCensusSgm::StereoTCensusSgm() : impl_(nullptr) {
	impl_ = new Impl(params, 0, 0, 0, 0);
}

void StereoTCensusSgm::compute(cv::InputArray l, cv::InputArray r, cv::OutputArray disparity) {

	//hipSetDevice(0);

	if (l.rows() != impl_->cost.height() || r.cols() != impl_->cost.width()) {
		delete impl_; impl_ = nullptr;
		impl_ = new Impl(params, l.cols(), l.rows(), params.d_min, params.d_max);
	}

	mat2gray(l, impl_->l);
	mat2gray(r, impl_->r);
	timer_set();

	impl_->cost.setT(params.t);
	impl_->cost.set(impl_->l, impl_->r);
	impl_->compute(disparity);
}

StereoTCensusSgm::~StereoTCensusSgm() {
	if (impl_) {
		delete impl_;
		impl_ = nullptr;
	}
}
