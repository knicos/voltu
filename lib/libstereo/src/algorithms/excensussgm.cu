#include "stereo.hpp"
#include "stereosgm.hpp"
#include "../costs/census.hpp"
#include "costs/scale.hpp"

#include <opencv2/cudafilters.hpp>

static void variance_mask(cv::InputArray in, cv::OutputArray out, int wsize=3) {
	if (in.isGpuMat() && out.isGpuMat()) {
		cv::cuda::GpuMat im;
		cv::cuda::GpuMat im2;
		cv::cuda::GpuMat mean;
		cv::cuda::GpuMat mean2;

		mean.create(in.size(), CV_32FC1);
		mean2.create(in.size(), CV_32FC1);
		im2.create(in.size(), CV_32FC1);

		if (in.type() != CV_32FC1) {
			in.getGpuMat().convertTo(im, CV_32FC1);
		}
		else {
			im = in.getGpuMat();
		}

		cv::cuda::multiply(im, im, im2);
		auto filter = cv::cuda::createBoxFilter(CV_32FC1, CV_32FC1, cv::Size(wsize,wsize));
		filter->apply(im, mean);   // E[X]
		filter->apply(im2, mean2); // E[X^2]
		cv::cuda::multiply(mean, mean, mean); // (E[X])^2

		// NOTE: floating point accuracy in subtraction
		// (cv::cuda::createBoxFilter only supports 8 bit integer types)
		cv::cuda::subtract(mean2, mean, out.getGpuMatRef()); // E[X^2] - (E[X])^2
	}
	else { throw std::exception(); /* todo CPU version */ }
}


typedef unsigned short CostType;
typedef WeightedCost<ExpandingCensusMatchingCost, CostType> MatchingCost;


struct StereoExCensusSgm::Impl : public StereoSgm<MatchingCost, StereoExCensusSgm::Parameters> {
	Array2D<uchar> l;
    Array2D<uchar> r;
    Array2D<float> variance;
    Array2D<float> variance_r;
    ExpandingCensusMatchingCost excensus;

	Impl(StereoExCensusSgm::Parameters &params, int width, int height, int dmin, int dmax) :
        StereoSgm(params, width, height, dmin, dmax), l(width, height), r(width, height),
        variance(width,height), variance_r(width,height),
        excensus(width, height, dmin, dmax) {
            cost.setCost(excensus);
            cost.setWeights(variance, variance_r);
        }
};

StereoExCensusSgm::StereoExCensusSgm() : impl_(nullptr) {
	impl_ = new Impl(params, 0, 0, 0, 0);
}

void StereoExCensusSgm::compute(cv::InputArray l, cv::InputArray r, cv::OutputArray disparity) {

	//hipSetDevice(0);

	if (l.rows() != impl_->cost.height() || r.cols() != impl_->cost.width()) {
		delete impl_; impl_ = nullptr;
		impl_ = new Impl(params, l.cols(), l.rows(), params.d_min, params.d_max);
	}

	mat2gray(l, impl_->l);
    mat2gray(r, impl_->r);
    
    cv::cuda::GpuMat var_l = impl_->variance.toGpuMat();
	variance_mask(impl_->l.toGpuMat(), var_l, params.var_window);
	cv::cuda::GpuMat var_r = impl_->variance_r.toGpuMat();
    variance_mask(impl_->r.toGpuMat(), var_r, params.var_window);
    
    cv::cuda::normalize(var_l, var_l, params.alpha, params.beta, cv::NORM_MINMAX, -1);
	cv::cuda::normalize(var_r, var_r, params.alpha, params.beta, cv::NORM_MINMAX, -1);

	impl_->excensus.setPattern(params.pattern);
    impl_->excensus.set(impl_->l, impl_->r);
    impl_->cost.set();

	cudaSafeCall(hipDeviceSynchronize());
    impl_->compute(disparity);
    
    Array2D<ExpandingCensusMatchingCost::Type> dsitmp_dev(l.cols(), l.rows());
	dsi_slice(impl_->cost, impl_->wta.disparity, dsitmp_dev);
	show_dsi_slice(dsitmp_dev.toGpuMat());
}

StereoExCensusSgm::~StereoExCensusSgm() {
	if (impl_) {
		delete impl_;
		impl_ = nullptr;
	}
}
