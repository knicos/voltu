#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/core/cuda/common.hpp>

#include "stereo.hpp"

#include "util_opencv.hpp"
#include "costs/mutual_information.hpp"
#include "dsi.hpp"

#include "wta.hpp"
#include "cost_aggregation.hpp"
#include "aggregations/standard_sgm.hpp"

#ifdef __GNUG__

#include <chrono>
#include <iostream>

static std::chrono::time_point<std::chrono::system_clock> start;

static void timer_set() {
		start = std::chrono::high_resolution_clock::now();
}

static void timer_print(const std::string &msg, const bool reset=true) {
	auto stop = std::chrono::high_resolution_clock::now();

	char buf[24];
	snprintf(buf, sizeof(buf), "%5i ms  ",
				(int) std::chrono::duration_cast<std::chrono::milliseconds>(stop-start).count());

	std::cout << buf <<  msg << "\n" << std::flush;
	if (reset) { timer_set(); }
}

#else

static void timer_set() {}
static void timer_print(const std::string &msg, const bool reset=true) {}

#endif

using cv::Mat;
using cv::Size;
using ftl::stereo::aggregations::StandardSGM;

struct StereoMiSgm::Impl {
	MutualInformationMatchingCost cost;

	Array2D<unsigned short> cost_min;
	Array2D<unsigned short> cost_min_paths;
	Array2D<unsigned short> uncertainty;
	Array2D<float> confidence;
	Array2D<float> disparity_r;
	Array2D<uchar> l;
	Array2D<uchar> r;

	Mat prior; // used only to calculate MI

	PathAggregator<StandardSGM<MutualInformationMatchingCost::DataType>> aggr;
	WinnerTakesAll<DSImage16U,float> wta;

	Impl(int width, int height, int min_disp, int max_disp) :
		cost(width, height, min_disp, max_disp),
		cost_min(width, height),
		cost_min_paths(width, height),
		uncertainty(width, height),
		confidence(width, height),
		disparity_r(width, height), l(width, height), r(width, height) {}

};

StereoMiSgm::StereoMiSgm() : impl_(nullptr) {
	impl_ = new Impl(0, 0, 0, 0);
}

void StereoMiSgm::setPrior(cv::InputArray prior) {
	if (prior.rows() != impl_->cost.height() || prior.cols() != impl_->cost.width()) {
		return;
	}
	if (prior.isGpuMat()) {
		prior.getGpuMat().download(impl_->prior);
	}
	else {
		prior.getMat().copyTo(impl_->prior);
	}
}


void StereoMiSgm::compute(cv::InputArray l, cv::InputArray r, cv::OutputArray disparity) {
	hipSetDevice(0);

	if (l.rows() != impl_->cost.height() || r.cols() != impl_->cost.width()) {
		delete impl_; impl_ = nullptr;
		impl_ = new Impl(l.cols(), l.rows(), params.d_min, params.d_max);
	}

	//todo prior
	if (impl_->prior.empty() || impl_->prior.size() != l.size()) {
		impl_->prior.create(l.rows(), l.cols(), CV_32FC1);
		cv::randu(impl_->prior, params.d_min, params.d_max);
	}

	impl_->uncertainty.toMat().setTo(0);

	timer_set();

	// CT
	if (l.isMat()) {
		impl_->cost.set(l.getMat(), r.getMat(), impl_->prior);
	}
	else if (l.isGpuMat()) {
		Mat l_;
		Mat r_;
		l.getGpuMat().download(l_);
		r.getGpuMat().download(r_);
		impl_->cost.set(l_, r_, impl_->prior);
	}
	if (params.debug) { timer_print("mutual information"); }

	cudaSafeCall(hipDeviceSynchronize());
	// cost aggregation
	//AggregationParameters aggr_params = {impl_->cost_min_paths.data(), params};
	StandardSGM<MutualInformationMatchingCost::DataType> func = {impl_->cost.data(), params.P1, params.P2};
	auto &out = impl_->aggr(func, AggregationDirections::ALL);  // params.paths

	cudaSafeCall(hipDeviceSynchronize());
	if (params.debug) { timer_print("Aggregation"); }

	impl_->wta(out, 0);
	cudaSafeCall(hipDeviceSynchronize());
	if (params.debug) { timer_print("WTA"); }
	if (disparity.isGpuMat()) {
		impl_->wta.disparity.toGpuMat(disparity.getGpuMatRef());
	}
	else {
		cv::Mat &disparity_ = disparity.getMatRef();
		impl_->wta.disparity.toMat(disparity_);
		cv::medianBlur(disparity_, disparity_, 3);
	}
	// confidence estimate

	// Drory, A., Haubold, C., Avidan, S., & Hamprecht, F. A. (2014).
	// Semi-global matching: A principled derivation in terms of
	// message passing. Lecture Notes in Computer Science (Including Subseries
	// Lecture Notes in Artificial Intelligence and Lecture Notes in
	// Bioinformatics). https://doi.org/10.1007/978-3-319-11752-2_4
	//cv::Mat uncertainty;
	//uncertainty = impl_->cost_min.toMat() - impl_->cost_min_paths.toMat();
	// confidence threshold
	// TODO: estimate confidence from uncertainty and plot ROC curve.
	//disparity.setTo(0.0f, uncertainty > params.uniqueness);
}

StereoMiSgm::~StereoMiSgm() {
	if (impl_) {
		delete impl_;
		impl_ = nullptr;
	}
}
