#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/core/cuda/common.hpp>

#include "stereo.hpp"

#include "util_opencv.hpp"
#include "costs/absolute_diff.hpp"
#include "costs/census.hpp"
#include "costs/dual.hpp"
#include "dsi.hpp"

#include "wta.hpp"
#include "cost_aggregation.hpp"
#include "aggregations/standard_sgm.hpp"

#ifdef __GNUG__

#include <chrono>
#include <iostream>

static std::chrono::time_point<std::chrono::system_clock> start;

static void timer_set() {
		start = std::chrono::high_resolution_clock::now();
}

static void timer_print(const std::string &msg, const bool reset=true) {
	auto stop = std::chrono::high_resolution_clock::now();

	char buf[24];
	snprintf(buf, sizeof(buf), "%5i ms  ",
				(int) std::chrono::duration_cast<std::chrono::milliseconds>(stop-start).count());

	std::cout << buf <<  msg << "\n" << std::flush;
	if (reset) { timer_set(); }
}

#else

static void timer_set() {}
static void timer_print(const std::string &msg, const bool reset=true) {}

#endif

using cv::Mat;
using cv::Size;
using ftl::stereo::aggregations::StandardSGM;

struct StereoADCensusSgm::Impl {
	DisparitySpaceImage<unsigned short> dsi;
	AbsDiffBT ad_cost;
	CensusMatchingCost census_cost;
	DualCosts<AbsDiffBT,CensusMatchingCost> cost;
	Array2D<unsigned short> cost_min;
	Array2D<unsigned short> cost_min_paths;
	Array2D<unsigned short> uncertainty;
	Array2D<float> confidence;
	Array2D<float> disparity_r;
	Array2D<uchar> l;
	Array2D<uchar> r;

	PathAggregator<StandardSGM<typename DualCosts<AbsDiffBT,CensusMatchingCost>::DataType>> aggr;
	WinnerTakesAll<DSImage16U,float> wta;

	Impl(int width, int height, int min_disp, int max_disp) :
		dsi(width, height, min_disp, max_disp),
		ad_cost(width, height, min_disp, max_disp),
		census_cost(width, height, min_disp, max_disp),
		cost(width, height, min_disp, max_disp, ad_cost, census_cost),
		cost_min(width, height),
		cost_min_paths(width, height),
		uncertainty(width, height),
		confidence(width, height),
		disparity_r(width, height), l(width, height), r(width, height) {}

};

StereoADCensusSgm::StereoADCensusSgm() : impl_(nullptr) {
	impl_ = new Impl(0, 0, 0, 0);
}

void StereoADCensusSgm::compute(cv::InputArray l, cv::InputArray r, cv::OutputArray disparity) {
	hipSetDevice(0);

	if (l.rows() != impl_->dsi.height() || r.cols() != impl_->dsi.width()) {
		delete impl_; impl_ = nullptr;
		impl_ = new Impl(l.cols(), l.rows(), params.d_min, params.d_max);
	}

	impl_->dsi.clear();
	impl_->uncertainty.toMat().setTo(0);

	mat2gray(l, impl_->l);
	mat2gray(r, impl_->r);

	timer_set();

	// CT
	impl_->census_cost.set(impl_->l, impl_->r);
	impl_->ad_cost.set(impl_->l, impl_->r);
	impl_->cost.set();

	cudaSafeCall(hipDeviceSynchronize());
	if (params.debug) { timer_print("census transform"); }

	// cost aggregation
	StandardSGM<DualCosts<AbsDiffBT,CensusMatchingCost>::DataType> func = {impl_->cost.data(), impl_->cost_min_paths.data(), params.P1, params.P2};
	auto &out = impl_->aggr(func, params.paths);

	cudaSafeCall(hipDeviceSynchronize());
	if (params.debug) { timer_print("Aggregation"); }

	impl_->wta(out, 0);
	cudaSafeCall(hipDeviceSynchronize());
	if (params.debug) { timer_print("WTA"); }
	if (disparity.isGpuMat()) {
		impl_->wta.disparity.toGpuMat(disparity.getGpuMatRef());
	}
	else {
		cv::Mat &disparity_ = disparity.getMatRef();
		impl_->wta.disparity.toMat(disparity_);
		cv::medianBlur(disparity_, disparity_, 3);
	}
	// confidence estimate

	// Drory, A., Haubold, C., Avidan, S., & Hamprecht, F. A. (2014).
	// Semi-global matching: A principled derivation in terms of
	// message passing. Lecture Notes in Computer Science (Including Subseries
	// Lecture Notes in Artificial Intelligence and Lecture Notes in
	// Bioinformatics). https://doi.org/10.1007/978-3-319-11752-2_4
	//cv::Mat uncertainty;
	//uncertainty = impl_->cost_min.toMat() - impl_->cost_min_paths.toMat();
	// confidence threshold
	// TODO: estimate confidence from uncertainty and plot ROC curve.
	//disparity.setTo(0.0f, uncertainty > params.uniqueness);
}

StereoADCensusSgm::~StereoADCensusSgm() {
	if (impl_) {
		delete impl_;
		impl_ = nullptr;
	}
}
