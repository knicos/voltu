#include "hip/hip_runtime.h"
#include <ftl/operators/mask_cuda.hpp>

#define T_PER_BLOCK 8

using ftl::cuda::Mask;

/* OLD VERSION */
__global__ void discontinuity_kernel(ftl::cuda::TextureObject<uint8_t> mask_out,
										ftl::cuda::TextureObject<uchar4> support,
										ftl::cuda::TextureObject<float> depth, 
										const cv::Size size, const double minDepth, const double maxDepth,
										float depthCoef, int radius) {
	
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < size.width && y < size.height) {
		Mask mask(0);

		const float d = depth.tex2D((int)x, (int)y);
		// Multiples of pixel size at given depth
		//const float threshold = (depthCoef / ((depthCoef / d) - (radius+disconDisparities-1))) - d;
		const float threshold = depthCoef * d;  // Where depthCoef = 1 / focal * N, N = number of pixel distances equal to a discon.

		if (d > minDepth && d < maxDepth) {
			/* Orts-Escolano S. et al. 2016. Holoportation: Virtual 3D teleportation in real-time. */

			// If colour cross support region terminates within the requested
			// radius, and the absolute depth difference on the other side is
			// greater than threshold, then is is a discontinuity.
			// Repeat for left, right, up and down.
			const uchar4 sup = support.tex2D((int)x, (int)y);
			if (sup.x <= radius) {
				float dS = depth.tex2D((int)x - sup.x - radius, (int)y);
				if (fabs(dS - d) > threshold) mask.isDiscontinuity(true);
			}
			if (sup.y <= radius) {
				float dS = depth.tex2D((int)x + sup.y + radius, (int)y);
				if (fabs(dS - d) > threshold) mask.isDiscontinuity(true);
			}
			if (sup.z <= radius) {
				float dS = depth.tex2D((int)x, (int)y - sup.z - radius);
				if (fabs(dS - d) > threshold) mask.isDiscontinuity(true);
			}
			if (sup.w <= radius) {
				float dS = depth.tex2D((int)x, (int)y + sup.w + radius);
				if (fabs(dS - d) > threshold) mask.isDiscontinuity(true);
			}

			// FIXME: The above results in a cross formation, need to test all 8 directions
		}
		
		mask_out(x,y) = (int)mask;
	}
}

/* New / Current version */
__global__ void discontinuity_kernel(ftl::cuda::TextureObject<uint8_t> mask_out,
		ftl::cuda::TextureObject<uchar4> support,
		ftl::cuda::TextureObject<float> depth, 
		const cv::Size size, const double minDepth, const double maxDepth,
		float depthCoef, float discon_thresh, float noise_thresh, float area_max) {

	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < size.width && y < size.height) {
		Mask mask(0);

		const float d = depth.tex2D((int)x, (int)y);
		// Multiples of pixel size at given depth
		//const float threshold = (depthCoef / ((depthCoef / d) - (radius+disconDisparities-1))) - d;
		const float threshold = depthCoef * d;  // Where depthCoef = 1 / focal * N, N = number of pixel distances equal to a discon.

		if (d > minDepth && d < maxDepth) {
			/* Orts-Escolano S. et al. 2016. Holoportation: Virtual 3D teleportation in real-time.
			 * This paper just says to remove values around discontinuities. */

			// Find max change in depth gradient in each direction
			const float g1 = fabsf((depth.tex2D(x-1, y) - d) - (d - depth.tex2D(x+1,y)));
			const float g2 = fabsf((depth.tex2D(x, y-1) - d) - (d - depth.tex2D(x,y+1)));
			const float g3 = fabsf((depth.tex2D(x-1, y-1) - d) - (d - depth.tex2D(x+1,y+1)));
			const float g4 = fabsf((depth.tex2D(x+1, y-1) - d) - (d - depth.tex2D(x-1,y+1)));
			const float g = max(g1,max(g2,(max(g3,g4))));

			// Calculate support window area
			const uchar4 sup = support.tex2D((int)x, (int)y);
			const float supx = min(sup.x,sup.y);
			const float supy = min(sup.z,sup.w);
			const float area = supx * supy;

			float grad_weight = min(1.0f, g / threshold);
			float area_weight = min(1.0f, area / area_max);

			if (grad_weight * (1.0f - area_weight) > discon_thresh) mask.isDiscontinuity(true);
			if (grad_weight * (area_weight) > noise_thresh) mask.isNoise(true);
		}

		mask_out(x,y) = (int)mask;
	}
}

void ftl::cuda::discontinuity(	ftl::cuda::TextureObject<uint8_t> &mask_out, ftl::cuda::TextureObject<uchar4> &support,
								ftl::cuda::TextureObject<float> &depth,
								const cv::Size size, const double minDepth, const double maxDepth,
								int discon, float depthCoef, hipStream_t stream) {
	
	const dim3 gridSize((size.width + T_PER_BLOCK - 1)/T_PER_BLOCK, (size.height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	discontinuity_kernel<<<gridSize, blockSize, 0, stream>>>(mask_out, support, depth, size, minDepth, maxDepth, depthCoef, discon);
	cudaSafeCall( hipGetLastError() );

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
#endif
}

void ftl::cuda::discontinuity(	ftl::cuda::TextureObject<uint8_t> &mask_out, ftl::cuda::TextureObject<uchar4> &support,
		ftl::cuda::TextureObject<float> &depth,
		const cv::Size size, const double minDepth, const double maxDepth,
		float depthCoef, float discon_thresh, float noise_thresh, float area_max, hipStream_t stream) {

	const dim3 gridSize((size.width + T_PER_BLOCK - 1)/T_PER_BLOCK, (size.height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	discontinuity_kernel<<<gridSize, blockSize, 0, stream>>>(mask_out, support, depth, size, minDepth, maxDepth, depthCoef, discon_thresh, noise_thresh, area_max);
	cudaSafeCall( hipGetLastError() );

	#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	#endif
}

// =============================================================================

template <int RADIUS>
__global__ void cull_mask_kernel(ftl::cuda::TextureObject<uint8_t> mask, ftl::cuda::TextureObject<float> depth, uint8_t id) {
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < depth.width()-RADIUS && y < depth.height()-RADIUS) {
		bool isdiscon = false;

		#pragma unroll
		for (int v=-RADIUS; v<=RADIUS; ++v) {
		#pragma unroll
		for (int u=-RADIUS; u<=RADIUS; ++u) {
			Mask m(mask.tex2D((int)x+u,(int)y+v));
			isdiscon = isdiscon || m.is(id);
		}
		}

		if (isdiscon) {
			depth(x,y) = 0.0f;
		}
	}
}

void ftl::cuda::cull_mask(ftl::cuda::TextureObject<uint8_t> &mask, ftl::cuda::TextureObject<float> &depth, uint8_t id, unsigned int radius, hipStream_t stream) {
	const dim3 gridSize((depth.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (depth.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	switch (radius) {
	case 0	: cull_mask_kernel<0><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
	case 1	: cull_mask_kernel<1><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
	case 2	: cull_mask_kernel<2><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
	case 3	: cull_mask_kernel<3><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
	case 4	: cull_mask_kernel<4><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
	case 5	: cull_mask_kernel<5><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
	default: break;
	}
	cudaSafeCall( hipGetLastError() );

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
#endif
}
