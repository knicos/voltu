#include "hip/hip_runtime.h"
#include <ftl/operators/mask_cuda.hpp>

#define T_PER_BLOCK 8

using ftl::cuda::Mask;

__global__ void discontinuity_kernel(ftl::cuda::TextureObject<uint8_t> mask_out,
		ftl::cuda::TextureObject<uchar4> support,
		ftl::cuda::TextureObject<float> depth, 
		const cv::Size size, const double minDepth, const double maxDepth,
		float depthCoef, float discon_thresh, float noise_thresh, float area_max) {

	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < size.width && y < size.height) {
		Mask mask(0);

		const float d = depth.tex2D((int)x, (int)y);
		// Multiples of pixel size at given depth
		//const float threshold = (depthCoef / ((depthCoef / d) - (radius+disconDisparities-1))) - d;
		const float threshold = depthCoef * d;  // Where depthCoef = 1 / focal * N, N = number of pixel distances equal to a discon.

		if (d > minDepth && d < maxDepth) {
			/* Orts-Escolano S. et al. 2016. Holoportation: Virtual 3D teleportation in real-time.
			 * This paper just says to remove values around discontinuities. */

			// Find max change in depth gradient in each direction
			const float g1 = fabsf((depth.tex2D(x-1, y) - d) - (d - depth.tex2D(x+1,y)));
			const float g2 = fabsf((depth.tex2D(x, y-1) - d) - (d - depth.tex2D(x,y+1)));
			const float g3 = fabsf((depth.tex2D(x-1, y-1) - d) - (d - depth.tex2D(x+1,y+1)));
			const float g4 = fabsf((depth.tex2D(x+1, y-1) - d) - (d - depth.tex2D(x-1,y+1)));
			const float g = max(g1,max(g2,(max(g3,g4))));

			// Calculate support window area
			//const uchar4 sup = support.tex2D((int)x, (int)y);
			const uchar4 sup = getScaledTex2D(x, y, support, depth);
			const float supx = min(sup.x,sup.y);
			const float supy = min(sup.z,sup.w);
			const float area = supx * supy;

			float grad_weight = min(1.0f, g / threshold);
			float area_weight = min(1.0f, area / area_max);

			if (grad_weight * (1.0f - area_weight) > discon_thresh) mask.isDiscontinuity(true);
			if (grad_weight * (area_weight) > noise_thresh) mask.isNoise(true);
		}

		mask_out(x,y) = (int)mask;
	}
}

void ftl::cuda::discontinuity(	ftl::cuda::TextureObject<uint8_t> &mask_out, ftl::cuda::TextureObject<uchar4> &support,
		ftl::cuda::TextureObject<float> &depth,
		const cv::Size size, const double minDepth, const double maxDepth,
		float depthCoef, float discon_thresh, float noise_thresh, float area_max, hipStream_t stream) {

	const dim3 gridSize((size.width + T_PER_BLOCK - 1)/T_PER_BLOCK, (size.height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	discontinuity_kernel<<<gridSize, blockSize, 0, stream>>>(mask_out, support, depth, size, minDepth, maxDepth, depthCoef, discon_thresh, noise_thresh, area_max);
	cudaSafeCall( hipGetLastError() );

	#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	#endif
}

// =============================================================================

__global__ void border_mask_kernel(uint8_t* __restrict__ mask_out,
		int pitch, int width, int height,
		int left, int right, int top, int bottom) {

	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		Mask mask(mask_out[x+y*pitch]);
		if (x < left || x >= width-right || y < top || y >= height-bottom) {
			mask.isBad(true);
			mask_out[x+y*pitch] = (int)mask;
		}
	}
}

void ftl::cuda::border_mask(ftl::cuda::TextureObject<uint8_t> &mask_out,
		int left, int right, int top, int bottom, hipStream_t stream) {

	static constexpr int THREADS_X = 128;
	static constexpr int THREADS_Y = 4;

	const dim3 gridSize((mask_out.width() + THREADS_X - 1)/THREADS_X, (mask_out.height() + THREADS_Y - 1)/THREADS_Y);
	const dim3 blockSize(THREADS_X, THREADS_Y);

	border_mask_kernel<<<gridSize, blockSize, 0, stream>>>(mask_out.devicePtr(), mask_out.pixelPitch(),
		mask_out.width(), mask_out.height(), left, right, top, bottom);
	cudaSafeCall( hipGetLastError() );

	#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	#endif
}

// =============================================================================

template <int RADIUS, bool INVERT>
__global__ void cull_mask_kernel(ftl::cuda::TextureObject<uint8_t> mask, ftl::cuda::TextureObject<float> depth, uint8_t id) {
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < depth.width() && y < depth.height()) {
		bool isdiscon = false;

		#pragma unroll
		for (int v=-RADIUS; v<=RADIUS; ++v) {
		#pragma unroll
		for (int u=-RADIUS; u<=RADIUS; ++u) {
			Mask m(mask.tex2D((int)x+u,(int)y+v));
			isdiscon = isdiscon || m.is(id);
		}
		}

		if ((!INVERT && isdiscon) || (INVERT && !isdiscon)) {
			depth(x,y) = 0.0f;
		}
	}
}

void ftl::cuda::cull_mask(ftl::cuda::TextureObject<uint8_t> &mask, ftl::cuda::TextureObject<float> &depth, uint8_t id, bool invert, unsigned int radius, hipStream_t stream) {
	const dim3 gridSize((depth.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (depth.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	if (invert) {
		switch (radius) {
		case 0	: cull_mask_kernel<0,true><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 1	: cull_mask_kernel<1,true><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 2	: cull_mask_kernel<2,true><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 3	: cull_mask_kernel<3,true><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 4	: cull_mask_kernel<4,true><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 5	: cull_mask_kernel<5,true><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		default: break;
		}
	} else {
		switch (radius) {
		case 0	: cull_mask_kernel<0,false><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 1	: cull_mask_kernel<1,false><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 2	: cull_mask_kernel<2,false><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 3	: cull_mask_kernel<3,false><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 4	: cull_mask_kernel<4,false><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		case 5	: cull_mask_kernel<5,false><<<gridSize, blockSize, 0, stream>>>(mask, depth, id); break;
		default: break;
		}
	}
	cudaSafeCall( hipGetLastError() );

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
#endif
}
