#include "hip/hip_runtime.h"
#include "mvmls_cuda.hpp"
#include <ftl/cuda/weighting.hpp>
#include <ftl/operators/mask_cuda.hpp>
#include <ftl/cuda/warp.hpp>

using ftl::cuda::TextureObject;
using ftl::rgbd::Camera;
using ftl::cuda::Mask;
using ftl::cuda::MvMLSParams;

#define T_PER_BLOCK 8
#define WARP_SIZE 32

#include "correspondence_common.hpp"


// ==== MultiViewMLS Aggregate =================================================

__device__ inline short3 getScreenPos(int x, int y, float d, const Camera &cam1, const Camera &cam2, const float4x4 &transform) {
    const float3 campos = transform * cam1.screenToCam(x,y,d);
    const int2 screen = cam2.camToScreen<int2>(campos);
    return make_short3(screen.x, screen.y, campos.z);
}

__device__ inline short2 packScreen(int x, int y, int id) {
    return make_short2((id << 12) + x, y);
}

__device__ inline short2 packScreen(const short3 &p, int id) {
    return make_short2((id << 12) + p.x, p.y);
}

__device__ inline int supportSize(uchar4 support) {
    return (support.x+support.y) * (support.z+support.w);
}

__device__ inline short2 choosePoint(uchar4 sup1, uchar4 sup2, float dot1, float dot2, short2 screen1, short2 screen2) {
    //return (float(supportSize(sup2))*dot1 > float(supportSize(sup1))*dot2) ? screen2 : screen1;
    return (dot1 > dot2) ? screen2 : screen1;
}

__device__ inline int unpackCameraID(short2 p) {
    return p.x >> 12;
}

/**
 * Identify which source has the best support region for a given pixel.
 */
__global__ void best_sources_kernel(
        TextureObject<half4> normals1,
        TextureObject<half4> normals2,
        TextureObject<uchar4> support1,
        TextureObject<uchar4> support2,
        TextureObject<float> depth1,
        TextureObject<float> depth2,
        TextureObject<short2> screen,
        float4x4 transform,
        //float3x3 transformR,
        ftl::rgbd::Camera cam1,
        ftl::rgbd::Camera cam2,
        int id1,
        int id2) {

    const int x = (blockIdx.x*blockDim.x + threadIdx.x);
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= 0 && y >= 0 && x < screen.width() && y < screen.height()) {
        const float d1 = depth1.tex2D(x,y);

        const short3 scr2 = getScreenPos(x, y, d1, cam1, cam2, transform);
        short2 bestPoint = packScreen(x,y,0);

        if (scr2.x >= 0 && scr2.y >= 0 && scr2.x < cam2.width && scr2.y < cam2.height) {
            uchar4 sup1 = support1.tex2D(x,y);
            uchar4 sup2 = support2.tex2D(scr2.x,scr2.y);
            const float d2 = depth2.tex2D(scr2.x,scr2.y);
            float3 n1 = transform.getFloat3x3() * make_float3(normals1.tex2D(x,y));
            float3 n2 = make_float3(normals2.tex2D(scr2.x,scr2.y));

            float3 camray = cam2.screenToCam(scr2.x,scr2.y,1.0f);
            camray /= length(camray);
            const float dot1 = dot(camray, n1);
            const float dot2 = dot(camray, n2);

            bestPoint = (fabs(scr2.z - d2) < 0.04f) ? choosePoint(sup1, sup2, dot1, dot2, packScreen(x,y,id1), packScreen(scr2,id2)) : packScreen(x,y,6);
            //bestPoint = choosePoint(sup1, sup2, dot1, dot2, packScreen(x,y,id1), packScreen(scr2,id2));
			//bestPoint = (d1 < d2) ? packScreen(x,y,id1) : packScreen(x,y,id2);
			
			bestPoint = (fabs(scr2.z - d2) < 0.04f) ? packScreen(scr2,id2) : packScreen(scr2,id1);
        }

        screen(x,y) = bestPoint;

        /*if (s.x >= 0 && s.y >= 0) {
            auto norm1 = make_float3(n1.tex2D(x,y));
            const auto norm2 = make_float3(n2.tex2D(s.x,s.y));
            //n2(s.x,s.y) = norm1;

            float3 cent1 = make_float3(c1.tex2D(x,y));
            const auto cent2 = make_float3(c2.tex2D(s.x,s.y));

            if (cent2.x+cent2.y+cent2.z > 0.0f && norm2.x+norm2.y+norm2.z > 0.0f) {
                norm1 += poseInv1.getFloat3x3() * (pose2.getFloat3x3() * norm2);
                n1(x,y) = make_float4(norm1, 0.0f);
				cent1 +=  poseInv1 * (pose2 * cent2);  // FIXME: Transform between camera spaces
				cent1 /= 2.0f;
                c1(x,y) = make_float4(cent1, 0.0f);
                //c2(s.x,s.y) = cent1;

				//contribs1(x,y) = contribs1.tex2D(x,y) + 1.0f;
            }
           // contribs2(s.x,s.y) = contribs2.tex2D(s.x,s.y) + 1.0f;
        }*/
    }
}

void ftl::cuda::best_sources(
        ftl::cuda::TextureObject<half4> &normals1,
        ftl::cuda::TextureObject<half4> &normals2,
        ftl::cuda::TextureObject<uchar4> &support1,
        ftl::cuda::TextureObject<uchar4> &support2,
        ftl::cuda::TextureObject<float> &depth1,
        ftl::cuda::TextureObject<float> &depth2,
        ftl::cuda::TextureObject<short2> &screen,
        const float4x4 &transform,
        const ftl::rgbd::Camera &cam1,
        const ftl::rgbd::Camera &cam2,
        int id1,
        int id2,
        hipStream_t stream) {

    const dim3 gridSize((screen.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (screen.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    best_sources_kernel<<<gridSize, blockSize, 0, stream>>>(normals1, normals2, support1, support2, depth1, depth2, screen, transform, cam1, cam2, id1, id2);
    cudaSafeCall( hipGetLastError() );
}

/**
 * Identify which source has the best support region for a given pixel.
 */
 __global__ void aggregate_sources_kernel(
		TextureObject<half4> n1,
		TextureObject<half4> n2,
		TextureObject<float4> c1,
		TextureObject<float4> c2,
		TextureObject<float> depth1,
		//TextureObject<float> depth2,
		//TextureObject<short2> screen,
		float4x4 transform,
		//float3x3 transformR,
		ftl::rgbd::Camera cam1,
		ftl::rgbd::Camera cam2) {

	const int x = (blockIdx.x*blockDim.x + threadIdx.x);
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= 0 && y >= 0 && x < n1.width() && y < n1.height()) {
		const float d1 = depth1.tex2D(x,y);

		if (d1 > cam1.minDepth && d1 < cam1.maxDepth) {
			//const short3 s = getScreenPos(x, y, d1, cam1, cam2, transform);
			const float3 camPos = transform * cam1.screenToCam(x, y, d1);
			const int2 s = cam2.camToScreen<int2>(camPos);

			if (s.x >= 0 && s.y >= 0 && s.x < n2.width() && s.y < n2.height()) {
				auto norm1 = make_float3(n1.tex2D(x,y));
				const auto norm2 = make_float3(n2.tex2D(s.x,s.y));
				//n2(s.x,s.y) = norm1;

				float3 cent1 = make_float3(c1.tex2D(x,y));
				const auto cent2 = transform.getInverse() * make_float3(c2.tex2D(s.x,s.y));

				//printf("MERGING %f\n", length(cent2-cent1));

				if (cent2.x+cent2.y+cent2.z > 0.0f && norm2.x+norm2.y+norm2.z > 0.0f && length(cent2-cent1) < 0.04f) {
					norm1 += norm2;
					norm1 /= 2.0f;
					n1(x,y) = make_half4(norm1, 0.0f);
					cent1 += cent2;
					cent1 /= 2.0f;
					c1(x,y) = make_float4(cent1, 0.0f);
					//c2(s.x,s.y) = cent1;

					//contribs1(x,y) = contribs1.tex2D(x,y) + 1.0f;
				}
			// contribs2(s.x,s.y) = contribs2.tex2D(s.x,s.y) + 1.0f;
			}
		}
	}
}

void ftl::cuda::aggregate_sources(
		ftl::cuda::TextureObject<half4> &n1,
		ftl::cuda::TextureObject<half4> &n2,
		ftl::cuda::TextureObject<float4> &c1,
		ftl::cuda::TextureObject<float4> &c2,
		ftl::cuda::TextureObject<float> &depth1,
		//ftl::cuda::TextureObject<float> &depth2,
		//ftl::cuda::TextureObject<short2> &screen,
		const float4x4 &transform,
		const ftl::rgbd::Camera &cam1,
		const ftl::rgbd::Camera &cam2,
		hipStream_t stream) {

	const dim3 gridSize((n1.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (n1.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	aggregate_sources_kernel<<<gridSize, blockSize, 0, stream>>>(n1, n2, c1, c2, depth1, transform, cam1, cam2);
	cudaSafeCall( hipGetLastError() );
}

__device__ static uchar4 HSVtoRGB(int H, float S, float V) {
	const float C = S * V;
	const float X = C * (1 - fabs(fmodf(H / 60.0f, 2) - 1));
	const float m = V - C;
	float Rs, Gs, Bs;

	if(H >= 0 && H < 60) {
		Rs = C;
		Gs = X;
		Bs = 0;	
	}
	else if(H >= 60 && H < 120) {	
		Rs = X;
		Gs = C;
		Bs = 0;	
	}
	else if(H >= 120 && H < 180) {
		Rs = 0;
		Gs = C;
		Bs = X;	
	}
	else if(H >= 180 && H < 240) {
		Rs = 0;
		Gs = X;
		Bs = C;	
	}
	else if(H >= 240 && H < 300) {
		Rs = X;
		Gs = 0;
		Bs = C;	
	}
	else {
		Rs = C;
		Gs = 0;
		Bs = X;	
	}

	return make_uchar4((Bs + m) * 255, (Gs + m) * 255, (Rs + m) * 255, 0);
}

/**
 * Render each pixel is a colour corresponding to the source camera with the
 * best support window.
 */
 __global__ void vis_best_sources_kernel(
        TextureObject<short2> screen,
        TextureObject<uchar4> colour,
        int myid,
        int count) {

    const int x = (blockIdx.x*blockDim.x + threadIdx.x);
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= 0 && y >= 0 && x < colour.width() && y < colour.height()) {
        short2 s = screen.tex2D(x,y);
        int id = unpackCameraID(s);

        uchar4 c = HSVtoRGB((360 / count) * id, 0.6f, 0.85f);
        if (myid != id) colour(x,y) = c;
        //colour(x,y) = c;
    }
}

void ftl::cuda::vis_best_sources(
        ftl::cuda::TextureObject<short2> &screen,
        ftl::cuda::TextureObject<uchar4> &colour,
        int myid,
        int count,
        hipStream_t stream) {

    const dim3 gridSize((colour.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (colour.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    vis_best_sources_kernel<<<gridSize, blockSize, 0, stream>>>(screen, colour, myid, count);
    cudaSafeCall( hipGetLastError() );
}

/*void ftl::cuda::aggregate_sources(
        ftl::cuda::TextureObject<float4> &n1,
        ftl::cuda::TextureObject<float4> &n2,
        ftl::cuda::TextureObject<float4> &c1,
        ftl::cuda::TextureObject<float4> &c2,
        ftl::cuda::TextureObject<float> &contribs1,
        ftl::cuda::TextureObject<float> &contribs2,
		ftl::cuda::TextureObject<short2> &screen,
		const float4x4 &poseInv1,
		const float4x4 &pose2,
        hipStream_t stream) {

    const dim3 gridSize((screen.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (screen.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    aggregate_sources_kernel<<<gridSize, blockSize, 0, stream>>>(n1, n2, c1, c2, contribs1, contribs2, screen, poseInv1, pose2);
    cudaSafeCall( hipGetLastError() );
}*/

// ==== Normalise aggregations =================================================

__global__ void normalise_aggregations_kernel(
        TextureObject<half4> norms,
        TextureObject<float4> cents,
        TextureObject<float> contribs) {

    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x < norms.width() && y < norms.height()) {
        const float contrib = contribs.tex2D((int)x,(int)y);

        const auto a = make_float3(norms.tex2D((int)x,(int)y));
        const auto b = cents.tex2D(x,y);
        //const float4 normal = normals.tex2D((int)x,(int)y);

		//out(x,y) = (contrib == 0.0f) ? make<B>(a) : make<B>(a / contrib);

        if (contrib > 0.0f) {
            norms(x,y) = make_half4(a / (contrib+1.0f), 1.0f);
            cents(x,y) = b / (contrib+1.0f);
        }
    }
}

void ftl::cuda::normalise_aggregations(TextureObject<half4> &norms, TextureObject<float4> &cents, TextureObject<float> &contribs, hipStream_t stream) {
    const dim3 gridSize((norms.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norms.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    normalise_aggregations_kernel<<<gridSize, blockSize, 0, stream>>>(norms, cents, contribs);
    cudaSafeCall( hipGetLastError() );
}
