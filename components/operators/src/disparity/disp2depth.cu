#include "hip/hip_runtime.h"
#include <ftl/cuda_common.hpp>
#include <ftl/rgbd/camera.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <ftl/operators/cuda/disparity.hpp>
#include <ftl/operators/cuda/mask.hpp>

#ifndef PINF
#define PINF __int_as_float(0x7f800000)
#endif

template<typename T_in, typename T_out>
__global__ void d2d_kernel(cv::cuda::PtrStepSz<T_in> disp, cv::cuda::PtrStepSz<T_out> depth,
		const ftl::rgbd::Camera cam, const float scale) {

	for (STRIDE_Y(v,disp.rows)) {
	for (STRIDE_X(u,disp.cols)) {
		short d = disp(v,u);
		depth(v,u) = (d == 0) ? 0.0f : ((cam.baseline*cam.fx) / ((float(d)*scale) - cam.doffs));
	}
	}
}

namespace ftl {
namespace cuda {

	template<typename T_in, typename T_out>
	void disparity_to_depth(const cv::cuda::GpuMat &disparity, cv::cuda::GpuMat &depth,
				const ftl::rgbd::Camera &c, float scale, hipStream_t &stream) {
		dim3 grid(1,1,1);
		dim3 threads(128, 1, 1);
		grid.x = cv::cuda::device::divUp(disparity.cols, 128);
		grid.y = cv::cuda::device::divUp(disparity.rows, 1);
		d2d_kernel<T_in, T_out><<<grid, threads, 0, stream>>>(
			disparity, depth, c, scale);
		cudaSafeCall( hipGetLastError() );
	}

	template void disparity_to_depth<short, float>(const cv::cuda::GpuMat&, cv::cuda::GpuMat&, const ftl::rgbd::Camera&, float, hipStream_t&);
	template void disparity_to_depth<float, float>(const cv::cuda::GpuMat&, cv::cuda::GpuMat&, const ftl::rgbd::Camera&, float, hipStream_t&);
}
}

//==============================================================================

template<typename T_in, typename T_out>
__global__ void d2drev_kernel(cv::cuda::PtrStepSz<T_in> disp, cv::cuda::PtrStepSz<T_out> depth,
	const ftl::rgbd::Camera cam, const float scale) {

	for (STRIDE_Y(v,disp.rows)) {
	for (STRIDE_X(u,disp.cols)) {
		float d = depth(v,u);
		float disparity = (d > cam.maxDepth || d < cam.minDepth) ? 0.0f : ((cam.baseline*cam.fx) / d) + cam.doffs;
		disp(v,u) = T_out(disparity*scale);
	}}
}

namespace ftl {
namespace cuda {

	template<typename T_in, typename T_out>
	void depth_to_disparity(const cv::cuda::GpuMat &depth, cv::cuda::GpuMat &disparity,
				const ftl::rgbd::Camera &c, float scale, hipStream_t &stream) {
		dim3 grid(1,1,1);
		dim3 threads(128, 1, 1);
		grid.x = cv::cuda::device::divUp(disparity.cols, 128);
		grid.y = cv::cuda::device::divUp(disparity.rows, 1);
		d2drev_kernel<T_in, T_out><<<grid, threads, 0, stream>>>(
			disparity, depth, c, scale);
		cudaSafeCall( hipGetLastError() );
	}

	template void depth_to_disparity<float, float>(const cv::cuda::GpuMat&, cv::cuda::GpuMat&, const ftl::rgbd::Camera&, float, hipStream_t&);
	template void depth_to_disparity<float, short>(const cv::cuda::GpuMat&, cv::cuda::GpuMat&, const ftl::rgbd::Camera&, float, hipStream_t&);

}
}

// =============================================================================

__global__ void remove_occ_kernel(cv::cuda::PtrStepSz<float> depth, cv::cuda::PtrStepSz<float> depthR,
	ftl::rgbd::Camera cam)
{
	for (STRIDE_Y(v,depth.rows)) {
	for (STRIDE_X(u,depth.cols)) {
		float d = depth(v,u);
		int disparity = int((d > cam.maxDepth || d < cam.minDepth) ? 0.0f : ((cam.baseline*cam.fx) / d) + cam.doffs);

		if (disparity > 0 && u-disparity > 0) {
			float dR = depthR(v,u-disparity);
			if (fabsf(d-dR) > 0.01f*d) {
				depth(v,u) = 0.0f;
			}
		}
	}
	}
}

void ftl::cuda::remove_occlusions(cv::cuda::GpuMat &depth, const cv::cuda::GpuMat &depthR,
			const ftl::rgbd::Camera &c, hipStream_t stream) {
	dim3 grid(1,1,1);
	dim3 threads(128, 4, 1);
	grid.x = cv::cuda::device::divUp(depth.cols, 128);
	grid.y = cv::cuda::device::divUp(depth.rows, 4);
	remove_occ_kernel<<<grid, threads, 0, stream>>>(
		depth, depthR, c);
	cudaSafeCall( hipGetLastError() );
}

__global__ void mask_occ_kernel(cv::cuda::PtrStepSz<float> depth,
	cv::cuda::PtrStepSz<float> depthR,
	cv::cuda::PtrStepSz<uchar> mask,
	ftl::rgbd::Camera cam)
{
	for (STRIDE_Y(v,depth.rows)) {
	for (STRIDE_X(u,depth.cols)) {
		float d = depth(v,u);
		int disparity = int((d > cam.maxDepth || d < cam.minDepth) ? 0.0f : ((cam.baseline*cam.fx) / d) + cam.doffs);

		if (disparity > 0 && u-disparity > 0) {
			float dR = depthR(v,u-disparity);
			if (fabsf(d-dR) > 0.01f*d) {
				mask(v,u) = mask(v,u) | ftl::cuda::Mask::kMask_Occlusion;
			}
		}
	}
	}
}

void ftl::cuda::mask_occlusions(const cv::cuda::GpuMat &depth, const cv::cuda::GpuMat &depthR,
			cv::cuda::GpuMat &mask,
			const ftl::rgbd::Camera &c, hipStream_t stream) {
	dim3 grid(1,1,1);
	dim3 threads(128, 4, 1);
	grid.x = cv::cuda::device::divUp(depth.cols, 128);
	grid.y = cv::cuda::device::divUp(depth.rows, 4);
	mask_occ_kernel<<<grid, threads, 0, stream>>>(
		depth, depthR, mask, c);
	cudaSafeCall( hipGetLastError() );
}
