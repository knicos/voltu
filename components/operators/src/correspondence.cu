#include "hip/hip_runtime.h"
#include "mvmls_cuda.hpp"
#include <ftl/cuda/weighting.hpp>
#include <ftl/cuda/mask.hpp>

using ftl::cuda::TextureObject;
using ftl::rgbd::Camera;
using ftl::cuda::Mask;
using ftl::cuda::MvMLSParams;

#define T_PER_BLOCK 8

template<int FUNCTION>
__device__ float weightFunction(const ftl::cuda::MvMLSParams &params, float dweight, float cweight);

template <>
__device__ inline float weightFunction<0>(const ftl::cuda::MvMLSParams &params, float dweight, float cweight) {
	return (params.cost_ratio * (cweight) + (1.0f - params.cost_ratio) * dweight);
}

template <>
__device__ inline float weightFunction<1>(const ftl::cuda::MvMLSParams &param, float dweight, float cweight) {
	return (cweight * cweight * dweight);
}

template <>
__device__ inline float weightFunction<2>(const ftl::cuda::MvMLSParams &param, float dweight, float cweight) {
	return (dweight * dweight * cweight);
}

template <>
__device__ inline float weightFunction<3>(const ftl::cuda::MvMLSParams &params, float dweight, float cweight) {
	return (dweight == 0.0f) ? 0.0f : (params.cost_ratio * (cweight) + (1.0f - params.cost_ratio) * dweight);
}

template <>
__device__ inline float weightFunction<4>(const ftl::cuda::MvMLSParams &params, float dweight, float cweight) {
	return cweight;
}

template <>
__device__ inline float weightFunction<5>(const ftl::cuda::MvMLSParams &params, float dweight, float cweight) {
	return (cweight > 0.0f) ? dweight : 0.0f;
}

template<int COR_STEPS, int FUNCTION> 
__global__ void corresponding_point_kernel(
        TextureObject<float> d1,
        TextureObject<float> d2,
        TextureObject<uchar4> c1,
        TextureObject<uchar4> c2,
        TextureObject<short2> screenOut,
		TextureObject<float> conf,
		TextureObject<int> mask,
        float4x4 pose1,
        float4x4 pose1_inv,
        float4x4 pose2,  // Inverse
        Camera cam1,
        Camera cam2, ftl::cuda::MvMLSParams params) {

    // Each warp picks point in p1
    //const int tid = (threadIdx.x + threadIdx.y * blockDim.x);
	const int x = (blockIdx.x*blockDim.x + threadIdx.x); // / WARP_SIZE;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= 0 && y >=0 && x < screenOut.width() && y < screenOut.height()) {
        screenOut(x,y) = make_short2(-1,-1);
    
        //const float3 world1 = make_float3(p1.tex2D(x, y));
        const float depth1 = d1.tex2D(x,y); //(pose1_inv * world1).z;  // Initial starting depth
        if (depth1 < cam1.minDepth || depth1 > cam1.maxDepth) return;

        // TODO: Temporary hack to ensure depth1 is present
        //const float4 temp = vout.tex2D(x,y);
        //vout(x,y) =  make_float4(depth1, 0.0f, temp.z, temp.w);
        
        const float3 world1 = pose1 * cam1.screenToCam(x,y,depth1);

        const auto colour1 = c1.tex2D((float)x+0.5f, (float)y+0.5f);

        //float bestdepth = 0.0f;
        short2 bestScreen = make_short2(-1,-1);
		float bestdepth = 0.0f;
		float bestdepth2 = 0.0f;
        float bestweight = 0.0f;
        float bestcolour = 0.0f;
        float bestdweight = 0.0f;
        float totalcolour = 0.0f;
        int count = 0;
        float contrib = 0.0f;
        
        const float step_interval = params.range / (COR_STEPS / 2);
        
        const float3 rayStep_world = pose1.getFloat3x3() * cam1.screenToCam(x,y,step_interval);
        const float3 rayStart_2 = pose2 * world1;
        const float3 rayStep_2 = pose2.getFloat3x3() * rayStep_world;

        // Project to p2 using cam2
        // Each thread takes a possible correspondence and calculates a weighting
        //const int lane = tid % WARP_SIZE;
        for (int i=0; i<COR_STEPS; ++i) {
            const int j = i - (COR_STEPS/2);
            const float depth_adjust = (float)j * step_interval;

            // Calculate adjusted depth 3D point in camera 2 space
            const float3 worldPos = world1 + j * rayStep_world; //(pose1 * cam1.screenToCam(x, y, depth_adjust));
            const float3 camPos = rayStart_2 + j * rayStep_2; //pose2 * worldPos;
			const float2 screen = cam2.camToScreen<float2>(camPos);
			
			float weight = (screen.x >= cam2.width || screen.y >= cam2.height) ? 0.0f : 1.0f;

			// Generate a colour correspondence value
            const auto colour2 = c2.tex2D(screen.x, screen.y);
            const float cweight = ftl::cuda::colourWeighting(colour1, colour2, params.colour_smooth);

            // Generate a depth correspondence value
			const float depth2 = d2.tex2D(int(screen.x+0.5f), int(screen.y+0.5f));
			
			if (FUNCTION == 1) {
				weight *= ftl::cuda::weighting(fabs(depth2 - camPos.z), cweight*params.spatial_smooth);
			} else {
				const float dweight = ftl::cuda::weighting(fabs(depth2 - camPos.z), params.spatial_smooth);
            	weight *= weightFunction<FUNCTION>(params, dweight, cweight);
			}
            //const float dweight = ftl::cuda::weighting(fabs(depth_adjust), 10.0f*params.range);

            //weight *= weightFunction<FUNCTION>(params, dweight, cweight);

            ++count;
            contrib += weight;
            bestcolour = max(cweight, bestcolour);
            //bestdweight = max(dweight, bestdweight);
            totalcolour += cweight;
			bestdepth = (weight > bestweight) ? depth_adjust : bestdepth;
			//bestdepth2 = (weight > bestweight) ? camPos.z : bestdepth2;
			//bestScreen = (weight > bestweight) ? make_short2(screen.x+0.5f, screen.y+0.5f) : bestScreen;
			bestweight = max(bestweight, weight);
                //bestweight = weight;
                //bestdepth = depth_adjust;
                //bestScreen = make_short2(screen.x+0.5f, screen.y+0.5f);
            //}
        }

        const float avgcolour = totalcolour/(float)count;
        const float confidence = bestcolour / totalcolour; //bestcolour - avgcolour;
        
        //Mask m(mask.tex2D(x,y));

        //if (bestweight > 0.0f) {
            float old = conf.tex2D(x,y);

            if (bestweight * confidence > old) {
				d1(x,y) = 0.4f*bestdepth + depth1;
				//d2(bestScreen.x, bestScreen.y) = bestdepth2;
                //screenOut(x,y) = bestScreen;
                conf(x,y) = bestweight * confidence;
            }
        //}
        
        // If a good enough match is found, mark dodgy depth as solid
        //if ((m.isFilled() || m.isDiscontinuity()) && (bestweight > params.match_threshold)) mask(x,y) = 0;
    }
}

void ftl::cuda::correspondence(
        TextureObject<float> &d1,
        TextureObject<float> &d2,
        TextureObject<uchar4> &c1,
        TextureObject<uchar4> &c2,
        TextureObject<short2> &screen,
		TextureObject<float> &conf,
		TextureObject<int> &mask,
        float4x4 &pose1,
        float4x4 &pose1_inv,
        float4x4 &pose2,
        const Camera &cam1,
        const Camera &cam2, const MvMLSParams &params, int func,
        hipStream_t stream) {

	const dim3 gridSize((d1.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (d1.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    //printf("COR SIZE %d,%d\n", p1.width(), p1.height());

	switch (func) {
    case 0: corresponding_point_kernel<16,0><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, screen, conf, mask, pose1, pose1_inv, pose2, cam1, cam2, params); break;
	case 1: corresponding_point_kernel<16,1><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, screen, conf, mask, pose1, pose1_inv, pose2, cam1, cam2, params); break;
	case 2: corresponding_point_kernel<16,2><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, screen, conf, mask, pose1, pose1_inv, pose2, cam1, cam2, params); break;
	case 3: corresponding_point_kernel<16,3><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, screen, conf, mask, pose1, pose1_inv, pose2, cam1, cam2, params); break;
	case 4: corresponding_point_kernel<16,4><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, screen, conf, mask, pose1, pose1_inv, pose2, cam1, cam2, params); break;
	case 5: corresponding_point_kernel<16,5><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, screen, conf, mask, pose1, pose1_inv, pose2, cam1, cam2, params); break;
	}

    cudaSafeCall( hipGetLastError() );
}

// ==== Remove zero-confidence =================================================

__global__ void zero_confidence_kernel(
		TextureObject<float> conf,
		TextureObject<float> depth) {

	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < depth.width() && y < depth.height()) {
		const float c = conf.tex2D((int)x,(int)y);

		if (c == 0.0f) {
			depth(x,y) = 1000.0f;	
		}
	}
}

void ftl::cuda::zero_confidence(TextureObject<float> &conf, TextureObject<float> &depth, hipStream_t stream) {
	const dim3 gridSize((depth.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (depth.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	zero_confidence_kernel<<<gridSize, blockSize, 0, stream>>>(conf, depth);
	cudaSafeCall( hipGetLastError() );
}


// ==== MultiViewMLS Aggregate =================================================

__device__ inline short3 getScreenPos(int x, int y, float d, const Camera &cam1, const Camera &cam2, const float4x4 &transform) {
    const float3 campos = transform * cam1.screenToCam(x,y,d);
    const int2 screen = cam2.camToScreen<int2>(campos);
    return make_short3(screen.x, screen.y, campos.z);
}

__device__ inline short2 packScreen(int x, int y, int id) {
    return make_short2((id << 12) + x, y);
}

__device__ inline short2 packScreen(const short3 &p, int id) {
    return make_short2((id << 12) + p.x, p.y);
}

__device__ inline int supportSize(uchar4 support) {
    return (support.x+support.y) * (support.z+support.w);
}

__device__ inline short2 choosePoint(uchar4 sup1, uchar4 sup2, float dot1, float dot2, short2 screen1, short2 screen2) {
    //return (float(supportSize(sup2))*dot1 > float(supportSize(sup1))*dot2) ? screen2 : screen1;
    return (dot1 > dot2) ? screen2 : screen1;
}

__device__ inline int unpackCameraID(short2 p) {
    return p.x >> 12;
}

/**
 * Identify which source has the best support region for a given pixel.
 */
__global__ void best_sources_kernel(
        TextureObject<float4> normals1,
        TextureObject<float4> normals2,
        TextureObject<uchar4> support1,
        TextureObject<uchar4> support2,
        TextureObject<float> depth1,
        TextureObject<float> depth2,
        TextureObject<short2> screen,
        float4x4 transform,
        //float3x3 transformR,
        ftl::rgbd::Camera cam1,
        ftl::rgbd::Camera cam2,
        int id1,
        int id2) {

    const int x = (blockIdx.x*blockDim.x + threadIdx.x);
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= 0 && y >= 0 && x < screen.width() && y < screen.height()) {
        const float d1 = depth1.tex2D(x,y);

        const short3 scr2 = getScreenPos(x, y, d1, cam1, cam2, transform);
        short2 bestPoint = packScreen(x,y,0);

        if (scr2.x >= 0 && scr2.y >= 0 && scr2.x < cam2.width && scr2.y < cam2.height) {
            uchar4 sup1 = support1.tex2D(x,y);
            uchar4 sup2 = support2.tex2D(scr2.x,scr2.y);
            const float d2 = depth2.tex2D(scr2.x,scr2.y);
            float3 n1 = transform.getFloat3x3() * make_float3(normals1.tex2D(x,y));
            float3 n2 = make_float3(normals2.tex2D(scr2.x,scr2.y));

            float3 camray = cam2.screenToCam(scr2.x,scr2.y,1.0f);
            camray /= length(camray);
            const float dot1 = dot(camray, n1);
            const float dot2 = dot(camray, n2);

            bestPoint = (fabs(scr2.z - d2) < 0.04f) ? choosePoint(sup1, sup2, dot1, dot2, packScreen(x,y,id1), packScreen(scr2,id2)) : packScreen(x,y,6);
            //bestPoint = choosePoint(sup1, sup2, dot1, dot2, packScreen(x,y,id1), packScreen(scr2,id2));
			//bestPoint = (d1 < d2) ? packScreen(x,y,id1) : packScreen(x,y,id2);
			
			bestPoint = (fabs(scr2.z - d2) < 0.04f) ? packScreen(scr2,id2) : packScreen(scr2,id1);
        }

        screen(x,y) = bestPoint;

        /*if (s.x >= 0 && s.y >= 0) {
            auto norm1 = make_float3(n1.tex2D(x,y));
            const auto norm2 = make_float3(n2.tex2D(s.x,s.y));
            //n2(s.x,s.y) = norm1;

            float3 cent1 = make_float3(c1.tex2D(x,y));
            const auto cent2 = make_float3(c2.tex2D(s.x,s.y));

            if (cent2.x+cent2.y+cent2.z > 0.0f && norm2.x+norm2.y+norm2.z > 0.0f) {
                norm1 += poseInv1.getFloat3x3() * (pose2.getFloat3x3() * norm2);
                n1(x,y) = make_float4(norm1, 0.0f);
				cent1 +=  poseInv1 * (pose2 * cent2);  // FIXME: Transform between camera spaces
				cent1 /= 2.0f;
                c1(x,y) = make_float4(cent1, 0.0f);
                //c2(s.x,s.y) = cent1;

				//contribs1(x,y) = contribs1.tex2D(x,y) + 1.0f;
            }
           // contribs2(s.x,s.y) = contribs2.tex2D(s.x,s.y) + 1.0f;
        }*/
    }
}

void ftl::cuda::best_sources(
        ftl::cuda::TextureObject<float4> &normals1,
        ftl::cuda::TextureObject<float4> &normals2,
        ftl::cuda::TextureObject<uchar4> &support1,
        ftl::cuda::TextureObject<uchar4> &support2,
        ftl::cuda::TextureObject<float> &depth1,
        ftl::cuda::TextureObject<float> &depth2,
        ftl::cuda::TextureObject<short2> &screen,
        const float4x4 &transform,
        const ftl::rgbd::Camera &cam1,
        const ftl::rgbd::Camera &cam2,
        int id1,
        int id2,
        hipStream_t stream) {

    const dim3 gridSize((screen.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (screen.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    best_sources_kernel<<<gridSize, blockSize, 0, stream>>>(normals1, normals2, support1, support2, depth1, depth2, screen, transform, cam1, cam2, id1, id2);
    cudaSafeCall( hipGetLastError() );
}

/**
 * Identify which source has the best support region for a given pixel.
 */
 __global__ void aggregate_sources_kernel(
		TextureObject<float4> n1,
		TextureObject<float4> n2,
		TextureObject<float4> c1,
		TextureObject<float4> c2,
		TextureObject<float> depth1,
		//TextureObject<float> depth2,
		//TextureObject<short2> screen,
		float4x4 transform,
		//float3x3 transformR,
		ftl::rgbd::Camera cam1,
		ftl::rgbd::Camera cam2) {

	const int x = (blockIdx.x*blockDim.x + threadIdx.x);
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= 0 && y >= 0 && x < n1.width() && y < n1.height()) {
		const float d1 = depth1.tex2D(x,y);

		if (d1 > cam1.minDepth && d1 < cam1.maxDepth) {
			//const short3 s = getScreenPos(x, y, d1, cam1, cam2, transform);
			const float3 camPos = transform * cam1.screenToCam(x, y, d1);
			const int2 s = cam2.camToScreen<int2>(camPos);

			if (s.x >= 0 && s.y >= 0 && s.x < n2.width() && s.y < n2.height()) {
				auto norm1 = make_float3(n1.tex2D(x,y));
				const auto norm2 = make_float3(n2.tex2D(s.x,s.y));
				//n2(s.x,s.y) = norm1;

				float3 cent1 = make_float3(c1.tex2D(x,y));
				const auto cent2 = transform.getInverse() * make_float3(c2.tex2D(s.x,s.y));

				//printf("MERGING %f\n", length(cent2-cent1));

				if (cent2.x+cent2.y+cent2.z > 0.0f && norm2.x+norm2.y+norm2.z > 0.0f && length(cent2-cent1) < 0.04f) {
					norm1 += norm2;
					norm1 /= 2.0f;
					n1(x,y) = make_float4(norm1, 0.0f);
					cent1 += cent2;
					cent1 /= 2.0f;
					c1(x,y) = make_float4(cent1, 0.0f);
					//c2(s.x,s.y) = cent1;

					//contribs1(x,y) = contribs1.tex2D(x,y) + 1.0f;
				}
			// contribs2(s.x,s.y) = contribs2.tex2D(s.x,s.y) + 1.0f;
			}
		}
	}
}

void ftl::cuda::aggregate_sources(
		ftl::cuda::TextureObject<float4> &n1,
		ftl::cuda::TextureObject<float4> &n2,
		ftl::cuda::TextureObject<float4> &c1,
		ftl::cuda::TextureObject<float4> &c2,
		ftl::cuda::TextureObject<float> &depth1,
		//ftl::cuda::TextureObject<float> &depth2,
		//ftl::cuda::TextureObject<short2> &screen,
		const float4x4 &transform,
		const ftl::rgbd::Camera &cam1,
		const ftl::rgbd::Camera &cam2,
		hipStream_t stream) {

	const dim3 gridSize((n1.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (n1.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	aggregate_sources_kernel<<<gridSize, blockSize, 0, stream>>>(n1, n2, c1, c2, depth1, transform, cam1, cam2);
	cudaSafeCall( hipGetLastError() );
}

__device__ static uchar4 HSVtoRGB(int H, float S, float V) {
	const float C = S * V;
	const float X = C * (1 - fabs(fmodf(H / 60.0f, 2) - 1));
	const float m = V - C;
	float Rs, Gs, Bs;

	if(H >= 0 && H < 60) {
		Rs = C;
		Gs = X;
		Bs = 0;	
	}
	else if(H >= 60 && H < 120) {	
		Rs = X;
		Gs = C;
		Bs = 0;	
	}
	else if(H >= 120 && H < 180) {
		Rs = 0;
		Gs = C;
		Bs = X;	
	}
	else if(H >= 180 && H < 240) {
		Rs = 0;
		Gs = X;
		Bs = C;	
	}
	else if(H >= 240 && H < 300) {
		Rs = X;
		Gs = 0;
		Bs = C;	
	}
	else {
		Rs = C;
		Gs = 0;
		Bs = X;	
	}

	return make_uchar4((Bs + m) * 255, (Gs + m) * 255, (Rs + m) * 255, 0);
}

/**
 * Render each pixel is a colour corresponding to the source camera with the
 * best support window.
 */
 __global__ void vis_best_sources_kernel(
        TextureObject<short2> screen,
        TextureObject<uchar4> colour,
        int myid,
        int count) {

    const int x = (blockIdx.x*blockDim.x + threadIdx.x);
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= 0 && y >= 0 && x < colour.width() && y < colour.height()) {
        short2 s = screen.tex2D(x,y);
        int id = unpackCameraID(s);

        uchar4 c = HSVtoRGB((360 / count) * id, 0.6f, 0.85f);
        if (myid != id) colour(x,y) = c;
        //colour(x,y) = c;
    }
}

void ftl::cuda::vis_best_sources(
        ftl::cuda::TextureObject<short2> &screen,
        ftl::cuda::TextureObject<uchar4> &colour,
        int myid,
        int count,
        hipStream_t stream) {

    const dim3 gridSize((colour.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (colour.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    vis_best_sources_kernel<<<gridSize, blockSize, 0, stream>>>(screen, colour, myid, count);
    cudaSafeCall( hipGetLastError() );
}

/*void ftl::cuda::aggregate_sources(
        ftl::cuda::TextureObject<float4> &n1,
        ftl::cuda::TextureObject<float4> &n2,
        ftl::cuda::TextureObject<float4> &c1,
        ftl::cuda::TextureObject<float4> &c2,
        ftl::cuda::TextureObject<float> &contribs1,
        ftl::cuda::TextureObject<float> &contribs2,
		ftl::cuda::TextureObject<short2> &screen,
		const float4x4 &poseInv1,
		const float4x4 &pose2,
        hipStream_t stream) {

    const dim3 gridSize((screen.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (screen.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    aggregate_sources_kernel<<<gridSize, blockSize, 0, stream>>>(n1, n2, c1, c2, contribs1, contribs2, screen, poseInv1, pose2);
    cudaSafeCall( hipGetLastError() );
}*/

// ==== Normalise aggregations =================================================

__global__ void normalise_aggregations_kernel(
        TextureObject<float4> norms,
        TextureObject<float4> cents,
        TextureObject<float> contribs) {

    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x < norms.width() && y < norms.height()) {
        const float contrib = contribs.tex2D((int)x,(int)y);

        const auto a = norms.tex2D((int)x,(int)y);
        const auto b = cents.tex2D(x,y);
        //const float4 normal = normals.tex2D((int)x,(int)y);

		//out(x,y) = (contrib == 0.0f) ? make<B>(a) : make<B>(a / contrib);

        if (contrib > 0.0f) {
            norms(x,y) = a / (contrib+1.0f);
            cents(x,y) = b / (contrib+1.0f);
        }
    }
}

void ftl::cuda::normalise_aggregations(TextureObject<float4> &norms, TextureObject<float4> &cents, TextureObject<float> &contribs, hipStream_t stream) {
    const dim3 gridSize((norms.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norms.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    normalise_aggregations_kernel<<<gridSize, blockSize, 0, stream>>>(norms, cents, contribs);
    cudaSafeCall( hipGetLastError() );
}

