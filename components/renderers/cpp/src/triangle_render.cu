#include "hip/hip_runtime.h"
#include <ftl/render/splat_params.hpp>
#include "splatter_cuda.hpp"
#include <ftl/rgbd/camera.hpp>
#include <ftl/cuda_common.hpp>

using ftl::rgbd::Camera;
using ftl::cuda::TextureObject;
using ftl::render::SplatParams;

#define T_PER_BLOCK 8

__device__ inline float length2(int dx, int dy) { return dx*dx + dy*dy; }

__device__ inline float cross(const float2 &a, const float2 &b) {
	return a.x*b.y - a.y*b.x;
}

__device__ inline bool within(float x) {
    return 0.0f <= x <= 1.0f;
}

__device__ inline bool operator==(const float2 &a, const float2 &b) {
	return a.x == b.x && a.y == b.y;
}

__device__ inline bool insideTriangle(const float2 &a, const float2 &b, const float2 &c, const float2 &p)
{   
    float det = (b.y - c.y)*(a.x - c.x) + (c.x - b.x)*(a.y - c.y);
    float factor_alpha = (b.y - c.y)*(p.x - c.x) + (c.x - b.x)*(p.y - c.y);
    float factor_beta = (c.y - a.y)*(p.x - c.x) + (a.x - c.x)*(p.y - c.y);
    float alpha = factor_alpha / det;
    float beta = factor_beta / det;
    float gamma = 1.0 - alpha - beta;

    return p == a || p == b || p == c || (within(alpha) && within(beta) && within(gamma));
}

__device__ inline void swap(short2 &a, short2 &b) {
	short2 t = a;
	a = b;
	b = t;
}

__device__ void drawLine(TextureObject<int> &depth_out, int y, int x1, int x2, float d) {
	for (int x=x1; x<=x2; ++x) {
		if (x < 0) continue;
		if (x >= depth_out.width()) return;
		atomicMin(&depth_out(x,y), int(d*1000.0f));
	}
}

/* See: https://github.com/bcrusco/CUDA-Rasterizer */

/**
 * Calculate the signed area of a given triangle.
 */
__device__ static inline
 float calculateSignedArea(const short2 &a, const short2 &b, const short2 &c) {
	 return 0.5f * (float(c.x - a.x) * float(b.y - a.y) - float(b.x - a.x) * float(c.y - a.y));
 }

/**
 * Helper function for calculating barycentric coordinates.
 */
 __device__ static inline
 float calculateBarycentricCoordinateValue(const short2 &a, const short2 &b, const short2 &c, const short2 (&tri)[3]) {
	 return calculateSignedArea(a,b,c) / calculateSignedArea(tri[0], tri[1], tri[2]);
 }
 
 /**
  * Calculate barycentric coordinates.
  * TODO: Update to handle triangles coming in and not the array
  */
__device__ static
 float3 calculateBarycentricCoordinate(const short2 (&tri)[3], const short2 &point) {
	 float beta = calculateBarycentricCoordinateValue(tri[0], point, tri[2], tri);
	 float gamma = calculateBarycentricCoordinateValue(tri[0], tri[1], point, tri);
	 float alpha = 1.0 - beta - gamma;
	 return make_float3(alpha, beta, gamma);
 }
 
 /**
  * Check if a barycentric coordinate is within the boundaries of a triangle.
  */
 __host__ __device__ static
 bool isBarycentricCoordInBounds(const float3 &barycentricCoord) {
	 return barycentricCoord.x >= 0.0 && barycentricCoord.x <= 1.0 &&
			barycentricCoord.y >= 0.0 && barycentricCoord.y <= 1.0 &&
			barycentricCoord.z >= 0.0 && barycentricCoord.z <= 1.0;
 }

 /**
 * For a given barycentric coordinate, compute the corresponding z position
 * (i.e. depth) on the triangle.
 */
__device__ static
float getZAtCoordinate(const float3 &barycentricCoord, const float (&tri)[3]) {
	return (barycentricCoord.x * tri[0]
		+ barycentricCoord.y * tri[1]
		+ barycentricCoord.z * tri[2]);
}

/*
 * Convert source screen position to output screen coordinates.
 */
 template <int A, int B>
 __global__ void triangle_render_1_kernel(
        TextureObject<float> depth_in,
        TextureObject<int> depth_out,
		TextureObject<short2> screen, SplatParams params) {
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x < 1 || x >= depth_in.width()-1 || y < 1 || y >= depth_in.height()-1) return;

    float d[3];
    d[0] = depth_in.tex2D(x,y);
    d[1] = depth_in.tex2D(x+A,y);
    d[2] = depth_in.tex2D(x,y+B);

    // Is this triangle valid
	if (fabs(d[0] - d[1]) > 0.04f || fabs(d[0] - d[2]) > 0.04f) return;
	if (d[0] < params.camera.minDepth || d[0] > params.camera.maxDepth) return;

    short2 v[3];
    v[0] = screen.tex2D(x,y);
    v[1] = screen.tex2D(x+A,y);
	v[2] = screen.tex2D(x,y+B);

	// Attempt to back face cull, but not great
	//if ((v[1].x - v[0].x) * A < 0 || (v[2].y - v[0].y) * B < 0) return;

	const int minX = min(v[0].x, min(v[1].x, v[2].x));
	const int minY = min(v[0].y, min(v[1].y, v[2].y));
	const int maxX = max(v[0].x, max(v[1].x, v[2].x));
	const int maxY = max(v[0].y, max(v[1].y, v[2].y));

	// Remove really large triangles
	if ((maxX - minX) * (maxY - minY) > 200) return;

	for (int sy=minY; sy <= maxY; ++sy) {
		for (int sx=minX; sx <= maxX; ++sx) {
			if (sx >= params.camera.width || sx < 0 || sy >= params.camera.height || sy < 0) continue;

			float3 baryCentricCoordinate = calculateBarycentricCoordinate(v, make_short2(sx, sy));

			if (isBarycentricCoordInBounds(baryCentricCoordinate)) {
				float new_depth = getZAtCoordinate(baryCentricCoordinate, d);
				atomicMin(&depth_out(sx,sy), int(new_depth*100000.0f));
			}
		}
	}
}

void ftl::cuda::triangle_render1(TextureObject<float> &depth_in, TextureObject<int> &depth_out, TextureObject<short2> &screen, const SplatParams &params, hipStream_t stream) {
    const dim3 gridSize((depth_in.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (depth_in.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	triangle_render_1_kernel<1,1><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, screen, params);
	triangle_render_1_kernel<1,-1><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, screen, params);
	triangle_render_1_kernel<-1,1><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, screen, params);
	triangle_render_1_kernel<-1,-1><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, screen, params);
    cudaSafeCall( hipGetLastError() );
}
