#include "hip/hip_runtime.h"
#include <ftl/render/splat_params.hpp>
#include "splatter_cuda.hpp"
#include <ftl/rgbd/camera.hpp>
#include <ftl/cuda_common.hpp>

#include <ftl/cuda/weighting.hpp>

using ftl::rgbd::Camera;
using ftl::cuda::TextureObject;
using ftl::render::SplatParams;

#define T_PER_BLOCK 8

__device__ inline float length2(int dx, int dy) { return dx*dx + dy*dy; }

__device__ inline float cross(const float2 &a, const float2 &b) {
	return a.x*b.y - a.y*b.x;
}

__device__ inline bool within(float x) {
    return 0.0f <= x <= 1.0f;
}

__device__ inline bool operator==(const float2 &a, const float2 &b) {
	return a.x == b.x && a.y == b.y;
}

__device__ inline bool insideTriangle(const float2 &a, const float2 &b, const float2 &c, const float2 &p)
{   
    float det = (b.y - c.y)*(a.x - c.x) + (c.x - b.x)*(a.y - c.y);
    float factor_alpha = (b.y - c.y)*(p.x - c.x) + (c.x - b.x)*(p.y - c.y);
    float factor_beta = (c.y - a.y)*(p.x - c.x) + (a.x - c.x)*(p.y - c.y);
    float alpha = factor_alpha / det;
    float beta = factor_beta / det;
    float gamma = 1.0 - alpha - beta;

    return p == a || p == b || p == c || (within(alpha) && within(beta) && within(gamma));
}

__device__ inline void swap(short2 &a, short2 &b) {
	short2 t = a;
	a = b;
	b = t;
}

__device__ void drawLine(TextureObject<int> &depth_out, int y, int x1, int x2, float d) {
	for (int x=x1; x<=x2; ++x) {
		if (x < 0) continue;
		if (x >= depth_out.width()) return;
		atomicMin(&depth_out(x,y), int(d*1000.0f));
	}
}

/* See: https://github.com/bcrusco/CUDA-Rasterizer */

/**
 * Calculate the signed area of a given triangle.
 */
__device__ static inline
 float calculateSignedArea(const short2 &a, const short2 &b, const short2 &c) {
	 return 0.5f * (float(c.x - a.x) * float(b.y - a.y) - float(b.x - a.x) * float(c.y - a.y));
 }

/**
 * Helper function for calculating barycentric coordinates.
 */
 __device__ static inline
 float calculateBarycentricCoordinateValue(const short2 &a, const short2 &b, const short2 &c, const short2 (&tri)[3]) {
	 return calculateSignedArea(a,b,c) / calculateSignedArea(tri[0], tri[1], tri[2]);
 }
 
 /**
  * Calculate barycentric coordinates.
  * TODO: Update to handle triangles coming in and not the array
  */
__device__ static
 float3 calculateBarycentricCoordinate(const short2 (&tri)[3], const short2 &point) {
	 float beta = calculateBarycentricCoordinateValue(tri[0], point, tri[2], tri);
	 float gamma = calculateBarycentricCoordinateValue(tri[0], tri[1], point, tri);
	 float alpha = 1.0f - beta - gamma;
	 return make_float3(alpha, beta, gamma);
 }
 
 /**
  * Check if a barycentric coordinate is within the boundaries of a triangle.
  */
 __host__ __device__ static
 bool isBarycentricCoordInBounds(const float3 &barycentricCoord) {
	 return barycentricCoord.x >= -0.0001f && //barycentricCoord.x <= 1.0f &&
			barycentricCoord.y >= -0.0001f && //barycentricCoord.y <= 1.0f &&
			barycentricCoord.z >= -0.0001f; // &&barycentricCoord.z <= 1.0f;
 }

 /**
 * For a given barycentric coordinate, compute the corresponding z position
 * (i.e. depth) on the triangle.
 */
__device__ static
float getZAtCoordinate(const float3 &barycentricCoord, const float (&tri)[3]) {
	return (barycentricCoord.x * tri[0]
		+ barycentricCoord.y * tri[1]
		+ barycentricCoord.z * tri[2]);
}

/*
 * Convert source screen position to output screen coordinates.
 */
 template <int A, int B>
 __global__ void triangle_render_1_kernel(
        TextureObject<float> depth_in,
        TextureObject<int> depth_out,
		TextureObject<short2> screen, SplatParams params) {
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x < 1 || x >= depth_in.width()-1 || y < 1 || y >= depth_in.height()-1) return;

    float d[3];
    d[0] = depth_in.tex2D(x,y);
    d[1] = depth_in.tex2D(x+A,y);
    d[2] = depth_in.tex2D(x,y+B);

    // Is this triangle valid
	if (fabs(d[0] - d[1]) > params.depthThreshold || fabs(d[0] - d[2]) > params.depthThreshold) return;
	if (d[0] < params.camera.minDepth || d[0] > params.camera.maxDepth) return;

    short2 v[3];
    v[0] = screen.tex2D(x,y);
    v[1] = screen.tex2D(x+A,y);
	v[2] = screen.tex2D(x,y+B);

	// Attempt to back face cull, but not great
	//if ((v[1].x - v[0].x) * A < 0 || (v[2].y - v[0].y) * B < 0) return;

	const int minX = min(v[0].x, min(v[1].x, v[2].x));
	const int minY = min(v[0].y, min(v[1].y, v[2].y));
	const int maxX = max(v[0].x, max(v[1].x, v[2].x));
	const int maxY = max(v[0].y, max(v[1].y, v[2].y));

	// Ensure the points themselves are drawn
	//atomicMin(&depth_out(v[0].x,v[0].y), int(d[0]*100000.0f));
	//atomicMin(&depth_out(v[1].x,v[1].y), int(d[1]*100000.0f));
	//atomicMin(&depth_out(v[2].x,v[2].y), int(d[2]*100000.0f));

	// Remove really large triangles
	if ((maxX - minX) * (maxY - minY) > params.triangle_limit) return;

	for (int sy=minY; sy <= maxY; ++sy) {
		for (int sx=minX; sx <= maxX; ++sx) {
			if (sx >= params.camera.width || sx < 0 || sy >= params.camera.height || sy < 0) continue;

			float3 baryCentricCoordinate = calculateBarycentricCoordinate(v, make_short2(sx, sy));

			if (isBarycentricCoordInBounds(baryCentricCoordinate)) {
				float new_depth = getZAtCoordinate(baryCentricCoordinate, d);
				atomicMin(&depth_out(sx,sy), int(new_depth*100000.0f));
			}
		}
	}
}

void ftl::cuda::triangle_render1(TextureObject<float> &depth_in, TextureObject<int> &depth_out, TextureObject<short2> &screen, const SplatParams &params, hipStream_t stream) {
    const dim3 gridSize((depth_in.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (depth_in.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	triangle_render_1_kernel<1,1><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, screen, params);
	triangle_render_1_kernel<1,-1><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, screen, params);
	triangle_render_1_kernel<-1,1><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, screen, params);
	triangle_render_1_kernel<-1,-1><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, screen, params);
    cudaSafeCall( hipGetLastError() );
}

// ==== Merge convert ===========

__global__ void merge_convert_kernel(
		TextureObject<int> depth_in,
		TextureObject<float> depth_out,
		float alpha) {
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < 0 || x >= depth_in.width() || y < 0 || y >= depth_in.height()) return;

	float a = float(depth_in.tex2D(x,y))*alpha;
	float b = depth_out.tex2D(x,y);
	depth_out(x,y) = min(a,b);
}

void ftl::cuda::merge_convert_depth(TextureObject<int> &depth_in, TextureObject<float> &depth_out, float alpha, hipStream_t stream) {
	const dim3 gridSize((depth_in.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (depth_in.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	merge_convert_kernel<<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, alpha);
	cudaSafeCall( hipGetLastError() );
}

// ==== BLENDER ========

/*
 * Merge two depth maps together
 */
 __global__ void mesh_blender_kernel(
        TextureObject<int> depth_in,
		TextureObject<int> depth_out,
		ftl::rgbd::Camera camera,
		float alpha) {
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x < 0 || x >= depth_in.width() || y < 0 || y >= depth_in.height()) return;

	int a = depth_in.tex2D(x,y);
	int b = depth_out.tex2D(x,y);

	float mindepth = (float)min(a,b) / 100000.0f;
	float maxdepth = (float)max(a,b) / 100000.0f;
	float weight = ftl::cuda::weighting(maxdepth-mindepth, alpha);

	//depth_out(x,y) = (int)(((float)mindepth + (float)maxdepth*weight) / (1.0f + weight) * 100000.0f);

	float depth = (mindepth + maxdepth*weight) / (1.0f + weight);
	depth_out(x,y) = (int)(depth * 100000.0f);
}

void ftl::cuda::mesh_blender(TextureObject<int> &depth_in, TextureObject<int> &depth_out, const ftl::rgbd::Camera &camera, float alpha, hipStream_t stream) {
    const dim3 gridSize((depth_in.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (depth_in.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	mesh_blender_kernel<<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, camera, alpha);
    cudaSafeCall( hipGetLastError() );
}
