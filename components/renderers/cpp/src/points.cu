#include "hip/hip_runtime.h"
#include <ftl/cuda/points.hpp>

#define T_PER_BLOCK 8

template <int RADIUS>
__global__ void point_cloud_kernel(ftl::cuda::TextureObject<float4> output, ftl::cuda::TextureObject<float> depth, ftl::rgbd::Camera params, float4x4 pose)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < params.width && y < params.height) {
		output(x,y) = make_float4(MINF, MINF, MINF, MINF);

		const float d = depth.tex2D((int)x, (int)y);
		float p = d;

		if (d >= params.minDepth && d <= params.maxDepth) {
			/* Orts-Escolano S. et al. 2016. Holoportation: Virtual 3D teleportation in real-time. */
			// Is there a discontinuity nearby?
			for (int u=-RADIUS; u<=RADIUS; ++u) {
				for (int v=-RADIUS; v<=RADIUS; ++v) {
					// If yes, the flag using w = -1
					if (fabs(depth.tex2D((int)x+u, (int)y+v) - d) > 0.1f) p = -1.0f;
				}
			}

			output(x,y) = make_float4(pose * params.screenToCam(x, y, d), p);
		}
	}
}

template <>
__global__ void point_cloud_kernel<0>(ftl::cuda::TextureObject<float4> output, ftl::cuda::TextureObject<float> depth, ftl::rgbd::Camera params, float4x4 pose)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < params.width && y < params.height) {
		output(x,y) = make_float4(MINF, MINF, MINF, MINF);

		float d = depth.tex2D((int)x, (int)y);

		if (d >= params.minDepth && d <= params.maxDepth) {
			output(x,y) = make_float4(pose * params.screenToCam(x, y, d), d);
		}
	}
}

void ftl::cuda::point_cloud(ftl::cuda::TextureObject<float4> &output, ftl::cuda::TextureObject<float> &depth, const ftl::rgbd::Camera &params, const float4x4 &pose, uint discon, hipStream_t stream) {
	const dim3 gridSize((params.width + T_PER_BLOCK - 1)/T_PER_BLOCK, (params.height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	switch (discon) {
	case 4 :	point_cloud_kernel<4><<<gridSize, blockSize, 0, stream>>>(output, depth, params, pose); break;
	case 3 :	point_cloud_kernel<3><<<gridSize, blockSize, 0, stream>>>(output, depth, params, pose); break;
	case 2 :	point_cloud_kernel<2><<<gridSize, blockSize, 0, stream>>>(output, depth, params, pose); break;
	case 1 :	point_cloud_kernel<1><<<gridSize, blockSize, 0, stream>>>(output, depth, params, pose); break;
	default:	point_cloud_kernel<0><<<gridSize, blockSize, 0, stream>>>(output, depth, params, pose);
	}
	cudaSafeCall( hipGetLastError() );

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
#endif
}

//==============================================================================

__device__ bool isClipped(const float4 &p, const ftl::cuda::ClipSpace &clip) {
	const float3 tp = clip.origin * make_float3(p);
	return fabs(tp.x) > clip.size.x || fabs(tp.y) > clip.size.y || fabs(tp.z) > clip.size.z;
}

__global__ void clipping_kernel(ftl::cuda::TextureObject<float4> points, ftl::cuda::ClipSpace clip)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < points.width() && y < points.height()) {
		float4 p = points(x,y);

		if (isClipped(p, clip)) {
			points(x,y) = make_float4(MINF, MINF, MINF, MINF);
		}
	}
}

void ftl::cuda::clipping(ftl::cuda::TextureObject<float4> &points,
		const ClipSpace &clip, hipStream_t stream) {

	const dim3 gridSize((points.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (points.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	clipping_kernel<<<gridSize, blockSize, 0, stream>>>(points, clip);
	cudaSafeCall( hipGetLastError() );
}
