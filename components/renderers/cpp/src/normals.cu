#include "hip/hip_runtime.h"
#include <ftl/cuda/normals.hpp>
#include <ftl/cuda/weighting.hpp>

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

__global__ void computeNormals_kernel(ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<float4> input) {
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= input.width() || y >= input.height()) return;

	output(x,y) = make_float4(0, 0, 0, 0);

	if(x > 0 && x < input.width()-1 && y > 0 && y < input.height()-1) {
		const float3 CC = make_float3(input.tex2D((int)x+0, (int)y+0)); //[(y+0)*width+(x+0)];
		const float3 PC = make_float3(input.tex2D((int)x+0, (int)y+1)); //[(y+1)*width+(x+0)];
		const float3 CP = make_float3(input.tex2D((int)x+1, (int)y+0)); //[(y+0)*width+(x+1)];
		const float3 MC = make_float3(input.tex2D((int)x+0, (int)y-1)); //[(y-1)*width+(x+0)];
		const float3 CM = make_float3(input.tex2D((int)x-1, (int)y+0)); //[(y+0)*width+(x-1)];

		if(CC.x != MINF && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF) {
			const float3 n = cross(PC-MC, CP-CM);
			const float  l = length(n);

			if(l > 0.0f) {
				output(x,y) = make_float4(n/-l, 1.0f);
			}
		}
	}
}

__device__ inline bool isValid(const ftl::rgbd::Camera &camera, const float3 &d) {
	return d.z >= camera.minDepth && d.z <= camera.maxDepth;
}

__global__ void computeNormals_kernel(ftl::cuda::TextureObject<float4> output,
		ftl::cuda::TextureObject<int> input, ftl::rgbd::Camera camera, float3x3 pose) {
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= input.width() || y >= input.height()) return;

	output(x,y) = make_float4(0, 0, 0, 0);

	if(x > 0 && x < input.width()-1 && y > 0 && y < input.height()-1) {
		const float3 CC = camera.screenToCam(x+0, y+0, (float)input.tex2D((int)x+0, (int)y+0) / 1000.0f);
		const float3 PC = camera.screenToCam(x+0, y+1, (float)input.tex2D((int)x+0, (int)y+1) / 1000.0f);
		const float3 CP = camera.screenToCam(x+1, y+0, (float)input.tex2D((int)x+1, (int)y+0) / 1000.0f);
		const float3 MC = camera.screenToCam(x+0, y-1, (float)input.tex2D((int)x+0, (int)y-1) / 1000.0f);
		const float3 CM = camera.screenToCam(x-1, y+0, (float)input.tex2D((int)x-1, (int)y+0) / 1000.0f);

		//if(CC.z <  && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF) {
		if (isValid(camera,CC) && isValid(camera,PC) && isValid(camera,CP) && isValid(camera,MC) && isValid(camera,CM)) {
			const float3 n = cross(PC-MC, CP-CM);
			const float  l = length(n);

			if(l > 0.0f) {
				output(x,y) = make_float4((n/-l), 1.0f);
			}
		}
	}
}

template <int RADIUS>
__global__ void smooth_normals_kernel(ftl::cuda::TextureObject<float4> norms,
        ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<float4> points,
        ftl::rgbd::Camera camera, float3x3 pose, float smoothing) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= points.width() || y >= points.height()) return;

    const float3 p0 = make_float3(points.tex2D((int)x,(int)y));
    float3 nsum = make_float3(0.0f);
    float contrib = 0.0f;

    if (p0.x == MINF) return;

    for (int v=-RADIUS; v<=RADIUS; ++v) {
        for (int u=-RADIUS; u<=RADIUS; ++u) {
            const float3 p = make_float3(points.tex2D((int)x+u,(int)y+v));
            if (p.x == MINF) continue;
            const float s = ftl::cuda::spatialWeighting(p0, p, smoothing);
            //const float s = 1.0f;

            if (s > 0.0f) {
                const float4 n = norms.tex2D((int)x+u,(int)y+v);
                if (n.w > 0.0f) {
                    nsum += make_float3(n) * s;
                    contrib += s;
                }
            }
        }
    }

    // Compute dot product of normal with camera to obtain measure of how
    // well this point faces the source camera, a measure of confidence
    float3 ray = pose * camera.screenToCam(x, y, 1.0f);
    ray = ray / length(ray);
    nsum /= contrib;
    nsum /= length(nsum);

    output(x,y) = (contrib > 0.0f) ? make_float4(nsum, dot(nsum, ray)) : make_float4(0.0f);
}

template <int RADIUS>
__global__ void smooth_normals_kernel(ftl::cuda::TextureObject<float4> norms,
        ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<int> depth,
        ftl::rgbd::Camera camera, float3x3 pose, float smoothing) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= depth.width() || y >= depth.height()) return;

    const float3 p0 = camera.screenToCam(x,y, (float)depth.tex2D((int)x,(int)y) / 1000.0f);
    float3 nsum = make_float3(0.0f);
    float contrib = 0.0f;

    if (p0.z < camera.minDepth || p0.z > camera.maxDepth) return;

    for (int v=-RADIUS; v<=RADIUS; ++v) {
        for (int u=-RADIUS; u<=RADIUS; ++u) {
            const float3 p = camera.screenToCam(x+u,y+v, (float)depth.tex2D((int)x+u,(int)y+v) / 1000.0f);
            if (p.z < camera.minDepth || p.z > camera.maxDepth) continue;
            const float s = ftl::cuda::spatialWeighting(p0, p, smoothing);
            //const float s = 1.0f;

            //if (s > 0.0f) {
                const float4 n = norms.tex2D((int)x+u,(int)y+v);
                if (n.w > 0.0f) {
                    nsum += make_float3(n) * s;
                    contrib += s;
                }
            //}
        }
    }

    // Compute dot product of normal with camera to obtain measure of how
    // well this point faces the source camera, a measure of confidence
    float3 ray = camera.screenToCam(x, y, 1.0f);
    ray = ray / length(ray);
    nsum /= contrib;
    nsum /= length(nsum);

    output(x,y) = (contrib > 0.0f) ? make_float4(pose*nsum, 1.0f) : make_float4(0.0f);
}

void ftl::cuda::normals(ftl::cuda::TextureObject<float4> &output,
        ftl::cuda::TextureObject<float4> &temp,
		ftl::cuda::TextureObject<float4> &input,
		int radius,
		float smoothing,
        const ftl::rgbd::Camera &camera,
        const float3x3 &pose,hipStream_t stream) {
	const dim3 gridSize((input.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (input.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormals_kernel<<<gridSize, blockSize, 0, stream>>>(temp, input);
    cudaSafeCall( hipGetLastError() );

	switch (radius) {
	case 9: smooth_normals_kernel<9><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
	case 7: smooth_normals_kernel<7><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
	case 5: smooth_normals_kernel<5><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
	case 3: smooth_normals_kernel<3><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
	}
    cudaSafeCall( hipGetLastError() );

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}

void ftl::cuda::normals(ftl::cuda::TextureObject<float4> &output,
		ftl::cuda::TextureObject<float4> &temp,
		ftl::cuda::TextureObject<int> &input,
		int radius,
		float smoothing,
		const ftl::rgbd::Camera &camera,
		const float3x3 &pose_inv, const float3x3 &pose,hipStream_t stream) {
	const dim3 gridSize((input.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (input.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormals_kernel<<<gridSize, blockSize, 0, stream>>>(temp, input, camera, pose);
	cudaSafeCall( hipGetLastError() );

	switch (radius) {
	case 7: smooth_normals_kernel<7><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	case 5: smooth_normals_kernel<5><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	case 3: smooth_normals_kernel<3><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	}
	cudaSafeCall( hipGetLastError() );

	#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
	#endif
}

//==============================================================================

__global__ void vis_normals_kernel(ftl::cuda::TextureObject<float4> norm,
        ftl::cuda::TextureObject<uchar4> output,
        float3 direction, uchar4 diffuse, uchar4 ambient) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= norm.width() || y >= norm.height()) return;

    output(x,y) = make_uchar4(0,0,0,0);
    float3 ray = direction;
    ray = ray / length(ray);
    float3 n = make_float3(norm.tex2D((int)x,(int)y));
    float l = length(n);
    if (l == 0) return;
    n /= l;

    const float d = max(dot(ray, n), 0.0f);
    output(x,y) = make_uchar4(
		min(255.0f, diffuse.x*d + ambient.x),
		min(255.0f, diffuse.y*d + ambient.y),
		min(255.0f, diffuse.z*d + ambient.z), 255);
}

void ftl::cuda::normal_visualise(ftl::cuda::TextureObject<float4> &norm,
        ftl::cuda::TextureObject<uchar4> &output,
        const float3 &light, const uchar4 &diffuse, const uchar4 &ambient,
        hipStream_t stream) {

    const dim3 gridSize((norm.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norm.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    vis_normals_kernel<<<gridSize, blockSize, 0, stream>>>(norm, output, light, diffuse, ambient);

    cudaSafeCall( hipGetLastError() );
#ifdef _DEBUG
    cudaSafeCall(hipDeviceSynchronize());
    //cutilCheckMsg(__FUNCTION__);
#endif
}

//==============================================================================

__global__ void filter_normals_kernel(ftl::cuda::TextureObject<float4> norm,
        ftl::cuda::TextureObject<float4> output,
        ftl::rgbd::Camera camera, float4x4 pose, float thresh) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= norm.width() || y >= norm.height()) return;

    float3 ray = pose.getFloat3x3() * camera.screenToCam(x,y,1.0f);
    ray = ray / length(ray);
    float3 n = make_float3(norm.tex2D((int)x,(int)y));
    float l = length(n);
    if (l == 0) {
        output(x,y) = make_float4(MINF);
        return;
    }
    n /= l;

    const float d = dot(ray, n);
    if (d <= thresh) {
        output(x,y) = make_float4(MINF);
    }
}

void ftl::cuda::normal_filter(ftl::cuda::TextureObject<float4> &norm,
        ftl::cuda::TextureObject<float4> &output,
        const ftl::rgbd::Camera &camera, const float4x4 &pose,
        float thresh,
        hipStream_t stream) {

    const dim3 gridSize((norm.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norm.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    filter_normals_kernel<<<gridSize, blockSize, 0, stream>>>(norm, output, camera, pose, thresh);

    cudaSafeCall( hipGetLastError() );
    #ifdef _DEBUG
    cudaSafeCall(hipDeviceSynchronize());
    //cutilCheckMsg(__FUNCTION__);
    #endif
}

//==============================================================================

__global__ void transform_normals_kernel(ftl::cuda::TextureObject<float4> norm,
        float3x3 pose) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= norm.width() || y >= norm.height()) return;

    float3 normal = pose * make_float3(norm.tex2D((int)x,(int)y));
    normal /= length(normal);
    norm(x,y) = make_float4(normal, 0.0f);
}

void ftl::cuda::transform_normals(ftl::cuda::TextureObject<float4> &norm,
        const float3x3 &pose,
        hipStream_t stream) {

    const dim3 gridSize((norm.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norm.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    transform_normals_kernel<<<gridSize, blockSize, 0, stream>>>(norm, pose);

    cudaSafeCall( hipGetLastError() );
    #ifdef _DEBUG
    cudaSafeCall(hipDeviceSynchronize());
    //cutilCheckMsg(__FUNCTION__);
    #endif
}
