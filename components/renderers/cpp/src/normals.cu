#include "hip/hip_runtime.h"
#include <ftl/cuda/normals.hpp>
#include <ftl/cuda/weighting.hpp>

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

__global__ void computeNormals_kernel(ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<float4> input) {
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= input.width() || y >= input.height()) return;

	output(x,y) = make_float4(0, 0, 0, 0);

	if(x > 0 && x < input.width()-1 && y > 0 && y < input.height()-1) {
		const float3 CC = make_float3(input.tex2D((int)x+0, (int)y+0)); //[(y+0)*width+(x+0)];
		const float3 PC = make_float3(input.tex2D((int)x+0, (int)y+1)); //[(y+1)*width+(x+0)];
		const float3 CP = make_float3(input.tex2D((int)x+1, (int)y+0)); //[(y+0)*width+(x+1)];
		const float3 MC = make_float3(input.tex2D((int)x+0, (int)y-1)); //[(y-1)*width+(x+0)];
		const float3 CM = make_float3(input.tex2D((int)x-1, (int)y+0)); //[(y+0)*width+(x-1)];

		if(CC.x != MINF && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF) {
			const float3 n = cross(PC-MC, CP-CM);
			const float  l = length(n);

			if(l > 0.0f) {
				output(x,y) = make_float4(n/-l, 1.0f);
			}
		}
	}
}

__device__ inline bool isValid(const ftl::rgbd::Camera &camera, const float3 &d) {
	return d.z >= camera.minDepth && d.z <= camera.maxDepth;
}

__global__ void computeNormals_kernel(ftl::cuda::TextureObject<float4> output,
		ftl::cuda::TextureObject<int> input, ftl::rgbd::Camera camera, float3x3 pose) {
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= input.width() || y >= input.height()) return;

	output(x,y) = make_float4(0, 0, 0, 0);

	if(x > 0 && x < input.width()-1 && y > 0 && y < input.height()-1) {
		const float3 CC = camera.screenToCam(x+0, y+0, (float)input.tex2D((int)x+0, (int)y+0) / 100000.0f);
		const float3 PC = camera.screenToCam(x+0, y+1, (float)input.tex2D((int)x+0, (int)y+1) / 100000.0f);
		const float3 CP = camera.screenToCam(x+1, y+0, (float)input.tex2D((int)x+1, (int)y+0) / 100000.0f);
		const float3 MC = camera.screenToCam(x+0, y-1, (float)input.tex2D((int)x+0, (int)y-1) / 100000.0f);
		const float3 CM = camera.screenToCam(x-1, y+0, (float)input.tex2D((int)x-1, (int)y+0) / 100000.0f);

		//if(CC.z <  && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF) {
		if (isValid(camera,CC) && isValid(camera,PC) && isValid(camera,CP) && isValid(camera,MC) && isValid(camera,CM)) {
			const float3 n = cross(PC-MC, CP-CM);
			const float  l = length(n);

			if(l > 0.0f) {
				output(x,y) = make_float4((n/-l), 1.0f);
			}
		}
	}
}

__global__ void computeNormals_kernel(ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<float> input, ftl::rgbd::Camera camera) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= input.width() || y >= input.height()) return;

    output(x,y) = make_float4(0, 0, 0, 0);

    if(x > 0 && x < input.width()-1 && y > 0 && y < input.height()-1) {
        const float3 CC = camera.screenToCam(x+0, y+0, input.tex2D((int)x+0, (int)y+0));
        const float3 PC = camera.screenToCam(x+0, y+1, input.tex2D((int)x+0, (int)y+1));
        const float3 CP = camera.screenToCam(x+1, y+0, input.tex2D((int)x+1, (int)y+0));
        const float3 MC = camera.screenToCam(x+0, y-1, input.tex2D((int)x+0, (int)y-1));
        const float3 CM = camera.screenToCam(x-1, y+0, input.tex2D((int)x-1, (int)y+0));

        //if(CC.z <  && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF) {
        if (isValid(camera,CC) && isValid(camera,PC) && isValid(camera,CP) && isValid(camera,MC) && isValid(camera,CM)) {
            const float3 n = cross(PC-MC, CP-CM);
            const float  l = length(n);

            if(l > 0.0f) {
                output(x,y) = make_float4((n/-l), 1.0f);
            }
        }
    }
}

template <int RADIUS>
__global__ void smooth_normals_kernel(ftl::cuda::TextureObject<float4> norms,
        ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<float4> points,
        ftl::rgbd::Camera camera, float3x3 pose, float smoothing) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= points.width() || y >= points.height()) return;

    const float3 p0 = make_float3(points.tex2D((int)x,(int)y));
    float3 nsum = make_float3(0.0f);
    float contrib = 0.0f;

    output(x,y) = make_float4(0.0f,0.0f,0.0f,0.0f);

    if (p0.x == MINF) return;

    for (int v=-RADIUS; v<=RADIUS; ++v) {
        for (int u=-RADIUS; u<=RADIUS; ++u) {
            const float3 p = make_float3(points.tex2D((int)x+u,(int)y+v));
            if (p.x == MINF) continue;
            const float s = ftl::cuda::spatialWeighting(p0, p, smoothing);
            //const float s = 1.0f;

            if (s > 0.0f) {
                const float4 n = norms.tex2D((int)x+u,(int)y+v);
                if (n.w > 0.0f) {
                    nsum += make_float3(n) * s;
                    contrib += s;
                }
            }
        }
    }

    // Compute dot product of normal with camera to obtain measure of how
    // well this point faces the source camera, a measure of confidence
    float3 ray = pose * camera.screenToCam(x, y, 1.0f);
    ray = ray / length(ray);
    nsum /= contrib;
    nsum /= length(nsum);

    output(x,y) = (contrib > 0.0f) ? make_float4(nsum, dot(nsum, ray)) : make_float4(0.0f);
}

template <int RADIUS>
__global__ void smooth_normals_kernel(ftl::cuda::TextureObject<float4> norms,
        ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<int> depth,
        ftl::rgbd::Camera camera, float3x3 pose, float smoothing) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= depth.width() || y >= depth.height()) return;

    const float3 p0 = camera.screenToCam(x,y, (float)depth.tex2D((int)x,(int)y) / 100000.0f);
    float3 nsum = make_float3(0.0f);
    float contrib = 0.0f;

    output(x,y) = make_float4(0.0f,0.0f,0.0f,0.0f);

    if (p0.z < camera.minDepth || p0.z > camera.maxDepth) return;

    for (int v=-RADIUS; v<=RADIUS; ++v) {
        for (int u=-RADIUS; u<=RADIUS; ++u) {
            const float3 p = camera.screenToCam(x+u,y+v, (float)depth.tex2D((int)x+u,(int)y+v) / 100000.0f);
            if (p.z < camera.minDepth || p.z > camera.maxDepth) continue;
            const float s = ftl::cuda::spatialWeighting(p0, p, smoothing);
            //const float s = 1.0f;

            //if (s > 0.0f) {
                const float4 n = norms.tex2D((int)x+u,(int)y+v);
                if (n.w > 0.0f) {
                    nsum += make_float3(n) * s;
                    contrib += s;
                }
            //}
        }
    }

    // Compute dot product of normal with camera to obtain measure of how
    // well this point faces the source camera, a measure of confidence
    float3 ray = camera.screenToCam(x, y, 1.0f);
    ray = ray / length(ray);
    nsum /= contrib;
    nsum /= length(nsum);

    output(x,y) = (contrib > 0.0f) ? make_float4(pose*nsum, 1.0f) : make_float4(0.0f);
}

template <>
__global__ void smooth_normals_kernel<0>(ftl::cuda::TextureObject<float4> norms,
        ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<int> depth,
        ftl::rgbd::Camera camera, float3x3 pose, float smoothing) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= depth.width() || y >= depth.height()) return;

    output(x,y) = make_float4(0.0f,0.0f,0.0f,0.0f);

    const float3 p0 = camera.screenToCam(x,y, (float)depth.tex2D((int)x,(int)y) / 1000.0f);

    if (p0.z < camera.minDepth || p0.z > camera.maxDepth) return;

    // Compute dot product of normal with camera to obtain measure of how
    // well this point faces the source camera, a measure of confidence
    //float3 ray = camera.screenToCam(x, y, 1.0f);
    //ray = ray / length(ray);
    //nsum /= contrib;
    //nsum /= length(nsum);

    const float4 n = norms.tex2D((int)x,(int)y);
    output(x,y) = n;
}

template <int RADIUS>
__global__ void smooth_normals_kernel(ftl::cuda::TextureObject<float4> norms,
        ftl::cuda::TextureObject<float4> output,
        ftl::cuda::TextureObject<float> depth,
        ftl::rgbd::Camera camera, float3x3 pose, float smoothing) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= depth.width() || y >= depth.height()) return;

    const float3 p0 = camera.screenToCam(x,y, depth.tex2D((int)x,(int)y));
    float3 nsum = make_float3(0.0f);
    float contrib = 0.0f;

    output(x,y) = make_float4(0.0f,0.0f,0.0f,0.0f);

    if (p0.z < camera.minDepth || p0.z > camera.maxDepth) return;

    for (int v=-RADIUS; v<=RADIUS; ++v) {
        for (int u=-RADIUS; u<=RADIUS; ++u) {
            const float3 p = camera.screenToCam(x+u,y+v, depth.tex2D((int)x+u,(int)y+v));
            if (p.z < camera.minDepth || p.z > camera.maxDepth) continue;
            const float s = ftl::cuda::spatialWeighting(p0, p, smoothing);
            //const float s = 1.0f;

            //if (s > 0.0f) {
                const float4 n = norms.tex2D((int)x+u,(int)y+v);
                if (n.w > 0.0f) {
                    nsum += make_float3(n) * s;
                    contrib += s;
                }
            //}
        }
    }

    // Compute dot product of normal with camera to obtain measure of how
    // well this point faces the source camera, a measure of confidence
    float3 ray = camera.screenToCam(x, y, 1.0f);
    ray = ray / length(ray);
    nsum /= contrib;
    nsum /= length(nsum);

    output(x,y) = (contrib > 0.0f) ? make_float4(pose*nsum, 1.0f) : make_float4(0.0f);
}

void ftl::cuda::normals(ftl::cuda::TextureObject<float4> &output,
        ftl::cuda::TextureObject<float4> &temp,
		ftl::cuda::TextureObject<float4> &input,
		int radius,
		float smoothing,
        const ftl::rgbd::Camera &camera,
        const float3x3 &pose,hipStream_t stream) {
	const dim3 gridSize((input.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (input.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormals_kernel<<<gridSize, blockSize, 0, stream>>>(temp, input);
    cudaSafeCall( hipGetLastError() );

	switch (radius) {
	case 9: smooth_normals_kernel<9><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
	case 7: smooth_normals_kernel<7><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
	case 5: smooth_normals_kernel<5><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
    case 3: smooth_normals_kernel<3><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
    case 2: smooth_normals_kernel<2><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
    case 1: smooth_normals_kernel<1><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
    case 0: smooth_normals_kernel<0><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing); break;
	}
    cudaSafeCall( hipGetLastError() );

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}

void ftl::cuda::normals(ftl::cuda::TextureObject<float4> &output,
		ftl::cuda::TextureObject<float4> &temp,
		ftl::cuda::TextureObject<int> &input,
		int radius,
		float smoothing,
		const ftl::rgbd::Camera &camera,
		const float3x3 &pose_inv, const float3x3 &pose,hipStream_t stream) {
	const dim3 gridSize((input.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (input.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormals_kernel<<<gridSize, blockSize, 0, stream>>>(temp, input, camera, pose);
	cudaSafeCall( hipGetLastError() );

	switch (radius) {
	case 7: smooth_normals_kernel<7><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	case 5: smooth_normals_kernel<5><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
    case 3: smooth_normals_kernel<3><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
    case 2: smooth_normals_kernel<2><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
    case 1: smooth_normals_kernel<1><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	}
	cudaSafeCall( hipGetLastError() );

	#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
	#endif
}

void ftl::cuda::normals(ftl::cuda::TextureObject<float4> &output,
		ftl::cuda::TextureObject<float4> &temp,
		ftl::cuda::TextureObject<float> &input,
		int radius,
		float smoothing,
		const ftl::rgbd::Camera &camera,
		const float3x3 &pose_inv, const float3x3 &pose,hipStream_t stream) {
	const dim3 gridSize((input.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (input.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormals_kernel<<<gridSize, blockSize, 0, stream>>>(temp, input, camera);
	cudaSafeCall( hipGetLastError() );

	switch (radius) {
	case 7: smooth_normals_kernel<7><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	case 5: smooth_normals_kernel<5><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	case 3: smooth_normals_kernel<3><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	case 2: smooth_normals_kernel<2><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	case 1: smooth_normals_kernel<1><<<gridSize, blockSize, 0, stream>>>(temp, output, input, camera, pose, smoothing);
	}
	cudaSafeCall( hipGetLastError() );

	#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
	#endif
}

void ftl::cuda::normals(ftl::cuda::TextureObject<float4> &output,
        ftl::cuda::TextureObject<float> &input,
        const ftl::rgbd::Camera &camera,
        hipStream_t stream) {
    const dim3 gridSize((input.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (input.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    computeNormals_kernel<<<gridSize, blockSize, 0, stream>>>(output, input, camera);
    cudaSafeCall( hipGetLastError() );

    #ifdef _DEBUG
    cudaSafeCall(hipDeviceSynchronize());
    //cutilCheckMsg(__FUNCTION__);
    #endif
}

// =============================================================================

__global__ void computeNormals_dot_kernel(ftl::cuda::TextureObject<float> output,
		ftl::cuda::TextureObject<float> input, ftl::rgbd::Camera camera) {
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= input.width() || y >= input.height()) return;

	float out = 0.0f;

	if(x > 0 && x < input.width()-1 && y > 0 && y < input.height()-1) {
		float d = input.tex2D((int)x+0, (int)y+0);
		const float3 CC = camera.screenToCam(x+0, y+0, d);
		const float3 PC = camera.screenToCam(x+0, y+1, input.tex2D((int)x+0, (int)y+1));
		const float3 CP = camera.screenToCam(x+1, y+0, input.tex2D((int)x+1, (int)y+0));
		const float3 MC = camera.screenToCam(x+0, y-1, input.tex2D((int)x+0, (int)y-1));
		const float3 CM = camera.screenToCam(x-1, y+0, input.tex2D((int)x-1, (int)y+0));

		//if(CC.z <  && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF) {
		if (isValid(camera,CC) && isValid(camera,PC) && isValid(camera,CP) && isValid(camera,MC) && isValid(camera,CM)) {
			float3 n = cross(PC-MC, CP-CM);
			const float  l = length(n);

			if(l > 0.0f) {
				n = n / -l;
				float3 ray = camera.screenToCam(x, y, d);
				ray = ray / length(ray);
				out = dot(ray,n);
			}
		}
	}

	output(x,y) = out;
}

void ftl::cuda::normals_dot(ftl::cuda::TextureObject<float> &output,
		ftl::cuda::TextureObject<float> &input,
		const ftl::rgbd::Camera &camera,
		hipStream_t stream) {
	const dim3 gridSize((input.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (input.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormals_dot_kernel<<<gridSize, blockSize, 0, stream>>>(output, input, camera);
	cudaSafeCall( hipGetLastError() );

	#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
	#endif
}

//==============================================================================

__global__ void vis_normals_kernel(ftl::cuda::TextureObject<float4> norm,
        ftl::cuda::TextureObject<uchar4> output,
        float3 direction, uchar4 diffuse, uchar4 ambient) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= norm.width() || y >= norm.height()) return;

    output(x,y) = make_uchar4(0,0,0,0);
    float3 ray = direction;
    ray = ray / length(ray);
    float3 n = make_float3(norm.tex2D((int)x,(int)y));
    float l = length(n);
    if (l == 0) return;
    n /= l;

    const float d = max(dot(ray, n), 0.0f);
    output(x,y) = make_uchar4(
		min(255.0f, diffuse.x*d + ambient.x),
		min(255.0f, diffuse.y*d + ambient.y),
		min(255.0f, diffuse.z*d + ambient.z), 255);
}

void ftl::cuda::normal_visualise(ftl::cuda::TextureObject<float4> &norm,
        ftl::cuda::TextureObject<uchar4> &output,
        const float3 &light, const uchar4 &diffuse, const uchar4 &ambient,
        hipStream_t stream) {

    const dim3 gridSize((norm.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norm.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    vis_normals_kernel<<<gridSize, blockSize, 0, stream>>>(norm, output, light, diffuse, ambient);

    cudaSafeCall( hipGetLastError() );
#ifdef _DEBUG
    cudaSafeCall(hipDeviceSynchronize());
    //cutilCheckMsg(__FUNCTION__);
#endif
}

//==============================================================================

__global__ void cool_blue_kernel(ftl::cuda::TextureObject<float4> norm,
        ftl::cuda::TextureObject<uchar4> output,
        uchar4 colouring, float3x3 pose) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= norm.width() || y >= norm.height()) return;

    //output(x,y) = make_uchar4(0,0,0,0);
    float3 ray = pose * make_float3(0.0f,0.0f,1.0f);
    ray = ray / length(ray);
    float3 n = make_float3(norm.tex2D((int)x,(int)y));
    float l = length(n);
    if (l == 0) return;
    n /= l;

    const float d = 1.0f - max(dot(ray, n), 0.0f);
    uchar4 original = output(x,y); //.tex2D(x,y);

    output(x,y) = make_uchar4(
        min(255.0f, colouring.x*d + original.x),
        min(255.0f, colouring.y*d + original.y),
        min(255.0f, colouring.z*d + original.z), 255);
}

void ftl::cuda::cool_blue(ftl::cuda::TextureObject<float4> &norm,
        ftl::cuda::TextureObject<uchar4> &output,
        const uchar4 &colouring, const float3x3 &pose,
        hipStream_t stream) {

    const dim3 gridSize((norm.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norm.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    cool_blue_kernel<<<gridSize, blockSize, 0, stream>>>(norm, output, colouring, pose);

    cudaSafeCall( hipGetLastError() );
    #ifdef _DEBUG
    cudaSafeCall(hipDeviceSynchronize());
    //cutilCheckMsg(__FUNCTION__);
    #endif
}

//==============================================================================

__global__ void filter_normals_kernel(ftl::cuda::TextureObject<float4> norm,
        ftl::cuda::TextureObject<float4> output,
        ftl::rgbd::Camera camera, float4x4 pose, float thresh) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= norm.width() || y >= norm.height()) return;

    float3 ray = pose.getFloat3x3() * camera.screenToCam(x,y,1.0f);
    ray = ray / length(ray);
    float3 n = make_float3(norm.tex2D((int)x,(int)y));
    float l = length(n);
    if (l == 0) {
        output(x,y) = make_float4(MINF);
        return;
    }
    n /= l;

    const float d = dot(ray, n);
    if (d <= thresh) {
        output(x,y) = make_float4(MINF);
    }
}

void ftl::cuda::normal_filter(ftl::cuda::TextureObject<float4> &norm,
        ftl::cuda::TextureObject<float4> &output,
        const ftl::rgbd::Camera &camera, const float4x4 &pose,
        float thresh,
        hipStream_t stream) {

    const dim3 gridSize((norm.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norm.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    filter_normals_kernel<<<gridSize, blockSize, 0, stream>>>(norm, output, camera, pose, thresh);

    cudaSafeCall( hipGetLastError() );
    #ifdef _DEBUG
    cudaSafeCall(hipDeviceSynchronize());
    //cutilCheckMsg(__FUNCTION__);
    #endif
}

//==============================================================================

__global__ void transform_normals_kernel(ftl::cuda::TextureObject<float4> norm,
        float3x3 pose) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= norm.width() || y >= norm.height()) return;

    float3 normal = pose * make_float3(norm.tex2D((int)x,(int)y));
    normal /= length(normal);
    norm(x,y) = make_float4(normal, 0.0f);
}

void ftl::cuda::transform_normals(ftl::cuda::TextureObject<float4> &norm,
        const float3x3 &pose,
        hipStream_t stream) {

    const dim3 gridSize((norm.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (norm.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    transform_normals_kernel<<<gridSize, blockSize, 0, stream>>>(norm, pose);

    cudaSafeCall( hipGetLastError() );
    #ifdef _DEBUG
    cudaSafeCall(hipDeviceSynchronize());
    //cutilCheckMsg(__FUNCTION__);
    #endif
}
