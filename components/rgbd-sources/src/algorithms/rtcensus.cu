#include "hip/hip_runtime.h"
/*
 * Author: Nicolas Pope and Sebastian Hahta (2019)
 * Implementation of algorithm presented in article(s):
 *
 * [1] Humenberger, Engelke, Kubinger: A fast stereo matching algorithm suitable
 *     for embedded real-time systems
 * [2] Humenberger, Zinner, Kubinger: Performance Evaluation of Census-Based
 *     Stereo Matching Algorithm on Embedded and Multi-Core Hardware
 * [3] Humenberger, Engelke, Kubinger: A Census-Based Stereo Vision Algorithm Using Modified Semi-Global Matching
 *     and Plane Fitting to Improve Matching Quality.
 *
 * Equation numbering uses [1] unless otherwise stated
 *
 */
 
#include <ftl/cuda_common.hpp>
#include "../cuda_algorithms.hpp"

using namespace cv::cuda;
using namespace cv;


#define BLOCK_W 60
#define RADIUS 7
#define RADIUS2 2
#define ROWSperTHREAD 1

namespace ftl {
namespace gpu {

// --- SUPPORT -----------------------------------------------------------------


/*
 * Parabolic interpolation between matched disparities either side.
 * Results in subpixel disparity. (20).
 */
__device__ float fit_parabola(size_t pi, uint16_t p, uint16_t pl, uint16_t pr) {
	float a = pr - pl;
	float b = 2 * (2 * p - pl - pr);
	return static_cast<float>(pi) + (a / b);
}

// --- KERNELS -----------------------------------------------------------------

/* Convert vector uint2 (32bit x2) into a single uint64_t */
__forceinline__ __device__ uint64_t uint2asull (uint2 a) {
	uint64_t res;
	asm ("mov.b64 %0, {%1,%2};" : "=l"(res) : "r"(a.x), "r"(a.y));
	return res;
}

/*
 * Generate left and right disparity images from census data. (18)(19)(25)
 */
__global__ void disp_kernel(float *disp_l, float *disp_r,
		int pitchL, int pitchR,
		size_t width, size_t height,
		hipTextureObject_t censusL, hipTextureObject_t censusR,
		size_t ds) {	
	//extern __shared__ uint64_t cache[];

	const int gamma = 35;
	
	int u = (blockIdx.x * BLOCK_W) + threadIdx.x + RADIUS2;
	int v_start = (blockIdx.y * ROWSperTHREAD) + RADIUS2;
	int v_end = v_start + ROWSperTHREAD;
	int maxdisp = ds;
	
	// Local cache
	uint64_t l_cache_l1[5][5];
	uint64_t l_cache_l2[5][5];
	
	if (v_end >= height) v_end = height;
	if (u+maxdisp >= width) maxdisp = width-u;
	
	for (int v=v_start; v<v_end; v++) {
		for (int m=-2; m<=2; m++) {
			for (int n=-2; n<=2; n++) {
				l_cache_l2[m+2][n+2] = uint2asull(tex2D<uint2>(censusL,u+n,v+m));
				l_cache_l1[m+2][n+2] = uint2asull(tex2D<uint2>(censusR,u+n,v+m));
			}
		}
		
		uint16_t last_ham1 = 65535;
		uint16_t last_ham2 = 65535;
		uint16_t min_disp1 = 65535;
		uint16_t min_disp2 = 65535;
		uint16_t min_disp1b = 65535;
		uint16_t min_disp2b = 65535;
		uint16_t min_before1 = 0;
		uint16_t min_before2 = 0;
		uint16_t min_after1 = 0;
		uint16_t min_after2 = 0;
		int dix1 = 0;
		int dix2 = 0;
		
		// (19)
		for (int d=0; d<maxdisp; d++) {
			uint16_t hamming1 = 0;
			uint16_t hamming2 = 0;
			
			//if (u+2+ds >= width) break;
		
			for (int m=-2; m<=2; m++) {
				const auto v_ = (v + m);
				for (int n=-2; n<=2; n++) {
					const auto u_ = u + n;
					
					// (18)
					auto l1 = l_cache_l1[m+2][n+2];
					auto l2 = l_cache_l2[m+2][n+2];
					auto r1 = uint2asull(tex2D<uint2>(censusL, u_+d, v_));
					auto r2 = uint2asull(tex2D<uint2>(censusR, u_-d, v_));
					hamming1 += __popcll(r1^l1);
					hamming2 += __popcll(r2^l2);
				}
			}
			
			// Find the two minimum costs
			if (hamming1 < min_disp1) {
				min_before1 = last_ham1;
				min_disp1 = hamming1;
				dix1 = d;
			} else if (hamming1 < min_disp1b) {
				min_disp1b = hamming1;
			}
			if (dix1 == d) min_after1 = hamming1;
			last_ham1 = hamming1;
			
			if (hamming2 < min_disp2) {
				min_before2 = last_ham2;
				min_disp2 = hamming2;
				dix2 = d;
			} else if (hamming2 < min_disp2b) {
				min_disp2b = hamming2;
			}
			if (dix2 == d) min_after2 = hamming2;
			last_ham2 = hamming2;
		
		}
		
		//float d1 = (dix1 == 0 || dix1 == ds-1) ? (float)dix1 : fit_parabola(dix1, min_disp1, min_before1, min_after1);
		//float d2 = (dix2 == 0 || dix2 == ds-1) ? (float)dix2 : fit_parabola(dix2, min_disp2, min_before2, min_after2);
	
		// TODO Allow for discontinuities with threshold
		// Subpixel disparity (20)
		float d1 = fit_parabola(dix1, min_disp1, min_before1, min_after1);
		float d2 = fit_parabola(dix2, min_disp2, min_before2, min_after2);
	
		// Confidence filter based on (25)
		disp_l[v*pitchL+u] = ((min_disp2b - min_disp2) >= gamma) ? d2 : NAN;
		disp_r[v*pitchR+u] = ((min_disp1b - min_disp1) >= gamma) ? d1 : NAN;
	}
}

void rtcensus_call(const PtrStepSz<uchar4> &l, const PtrStepSz<uchar4> &r, const PtrStepSz<float> &disp, size_t num_disp, const int &stream) {
	// Make all the required texture steps
	// TODO Could reduce re-allocations by caching these
	ftl::cuda::TextureObject<uchar4> texLeft(l);
	ftl::cuda::TextureObject<uchar4> texRight(r);
	ftl::cuda::TextureObject<uint2> censusTexLeft(l.cols, l.rows);
	ftl::cuda::TextureObject<uint2> censusTexRight(r.cols, r.rows);
	ftl::cuda::TextureObject<float> dispTexLeft(l.cols, l.rows);
	ftl::cuda::TextureObject<float> dispTexRight(r.cols, r.rows);
	ftl::cuda::TextureObject<float> dispTex(r.cols, r.rows);
	ftl::cuda::TextureObject<float> output(disp);
	
	// Calculate the census for left and right (14)(15)(16)
	ftl::cuda::sparse_census(texLeft, texRight, censusTexLeft, censusTexRight);

	dim3 grid(1,1,1);
    dim3 threads(BLOCK_W, 1, 1);
	grid.x = cv::cuda::device::divUp(l.cols - 2 * RADIUS2, BLOCK_W);
	grid.y = cv::cuda::device::divUp(l.rows - 2 * RADIUS2, ROWSperTHREAD);
	
	// Calculate L and R disparities (18)(19)(20)(21)(22)(25)
	disp_kernel<<<grid, threads>>>(
		dispTexLeft.devicePtr(), dispTexRight.devicePtr(),
		dispTexLeft.pitch()/sizeof(float), dispTexRight.pitch()/sizeof(float),
		l.cols, l.rows,
		censusTexLeft.cudaTexture(), censusTexRight.cudaTexture(),
		num_disp);
	cudaSafeCall( hipGetLastError() );
	
	// Check consistency between L and R disparities. (23)(24)
	consistency(dispTexLeft, dispTexRight, dispTex);

	// TM in (7) of paper [3]. Eq (26) in [1] is wrong.
	texture_filter(texLeft, dispTex, output, num_disp, 10.0);

	cudaSafeCall( hipDeviceSynchronize() );
	
	texLeft.free();
	texRight.free();
	censusTexLeft.free();
	censusTexRight.free();
	dispTexLeft.free();
	dispTexRight.free();
	dispTex.free();
	output.free();
}

};
};
