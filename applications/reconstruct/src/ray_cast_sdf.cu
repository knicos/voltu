#include "hip/hip_runtime.h"

//#include <hip/hip_runtime.h>

#include <ftl/cuda_matrix_util.hpp>

#include <ftl/depth_camera.hpp>
#include <ftl/voxel_hash.hpp>
#include <ftl/ray_cast_util.hpp>

#define T_PER_BLOCK 8
#define NUM_GROUPS_X 1024

#define NUM_CUDA_BLOCKS  10000

//texture<float, hipTextureType2D, hipReadModeElementType> rayMinTextureRef;
//texture<float, hipTextureType2D, hipReadModeElementType> rayMaxTextureRef;

__global__ void renderKernel(ftl::voxhash::HashData hashData, RayCastData rayCastData, RayCastParams rayCastParams) 
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	//const RayCastParams& rayCastParams = c_rayCastParams;

	if (x < rayCastParams.m_width && y < rayCastParams.m_height) {
		rayCastData.d_depth[y*rayCastParams.m_width+x] = MINF;
		rayCastData.d_depth3[y*rayCastParams.m_width+x] = make_float3(MINF,MINF,MINF);
		rayCastData.d_normals[y*rayCastParams.m_width+x] = make_float4(MINF,MINF,MINF,MINF);
		rayCastData.d_colors[y*rayCastParams.m_width+x] = make_uchar3(0,0,0);

		float3 camDir = normalize(rayCastParams.camera.kinectProjToCamera(x, y, 1.0f));
		float3 worldCamPos = rayCastParams.m_viewMatrixInverse * make_float3(0.0f, 0.0f, 0.0f);
		float4 w = rayCastParams.m_viewMatrixInverse * make_float4(camDir, 0.0f);
		float3 worldDir = normalize(make_float3(w.x, w.y, w.z));

		////use ray interval splatting
		//float minInterval = tex2D(rayMinTextureRef, x, y);
		//float maxInterval = tex2D(rayMaxTextureRef, x, y);

		//don't use ray interval splatting
		float minInterval = rayCastParams.m_minDepth;
		float maxInterval = rayCastParams.m_maxDepth;

		//if (minInterval == 0 || minInterval == MINF) minInterval = rayCastParams.m_minDepth;
		//if (maxInterval == 0 || maxInterval == MINF) maxInterval = rayCastParams.m_maxDepth;
		//TODO MATTHIAS: shouldn't this return in the case no interval is found?
		if (minInterval == 0 || minInterval == MINF) return;
		if (maxInterval == 0 || maxInterval == MINF) return;

		// debugging 
		//if (maxInterval < minInterval) {
		//	printf("ERROR (%d,%d): [ %f, %f ]\n", x, y, minInterval, maxInterval);
		//}

		rayCastData.traverseCoarseGridSimpleSampleAll(hashData, rayCastParams, worldCamPos, worldDir, camDir, make_int3(x,y,1), minInterval, maxInterval);
	} 
}

extern "C" void renderCS(const ftl::voxhash::HashData& hashData, const RayCastData &rayCastData, const RayCastParams &rayCastParams, hipStream_t stream) 
{

	const dim3 gridSize((rayCastParams.m_width + T_PER_BLOCK - 1)/T_PER_BLOCK, (rayCastParams.m_height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	//hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	//hipBindTextureToArray(rayMinTextureRef, rayCastData.d_rayIntervalSplatMinArray, channelDesc);
	//hipBindTextureToArray(rayMaxTextureRef, rayCastData.d_rayIntervalSplatMaxArray, channelDesc);

	//printf("Ray casting render...\n");

	renderKernel<<<gridSize, blockSize, 0, stream>>>(hashData, rayCastData, rayCastParams);

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}


/////////////////////////////////////////////////////////////////////////
// ray interval splatting
/////////////////////////////////////////////////////////////////////////

/*__global__ void resetRayIntervalSplatKernel(RayCastData data) 
{
	uint idx = blockIdx.x + blockIdx.y * NUM_GROUPS_X;
	data.point_cloud_[idx] = make_float3(MINF);
}

extern "C" void resetRayIntervalSplatCUDA(RayCastData& data, const RayCastParams& params)
{
	const dim3 gridSize(NUM_GROUPS_X, (params.m_maxNumVertices + NUM_GROUPS_X - 1) / NUM_GROUPS_X, 1); // ! todo check if need third dimension?
	const dim3 blockSize(1, 1, 1);

	resetRayIntervalSplatKernel<<<gridSize, blockSize>>>(data);

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}*/

/*__global__ void rayIntervalSplatKernel(ftl::voxhash::HashData hashData, DepthCameraData depthCameraData, RayCastData rayCastData, DepthCameraData cameraData) 
{
	uint idx = blockIdx.x + blockIdx.y * NUM_GROUPS_X;

	const ftl::voxhash::HashEntry& entry = hashData.d_hashCompactified[idx];
	if (entry.ptr != ftl::voxhash::FREE_ENTRY) {
		//if (!hashData.isSDFBlockInCameraFrustumApprox(entry.pos)) return;
		const RayCastParams &params = c_rayCastParams;
		const float4x4& viewMatrix = params.m_viewMatrix;

		float3 worldCurrentVoxel = hashData.SDFBlockToWorld(entry.pos);

		float3 MINV = worldCurrentVoxel - c_hashParams.m_virtualVoxelSize / 2.0f;

		float3 maxv = MINV+SDF_BLOCK_SIZE*c_hashParams.m_virtualVoxelSize;

		float3 proj000 = cameraData.cameraToKinectProj(viewMatrix * make_float3(MINV.x, MINV.y, MINV.z));
		float3 proj100 = cameraData.cameraToKinectProj(viewMatrix * make_float3(maxv.x, MINV.y, MINV.z));
		float3 proj010 = cameraData.cameraToKinectProj(viewMatrix * make_float3(MINV.x, maxv.y, MINV.z));
		float3 proj001 = cameraData.cameraToKinectProj(viewMatrix * make_float3(MINV.x, MINV.y, maxv.z));
		float3 proj110 = cameraData.cameraToKinectProj(viewMatrix * make_float3(maxv.x, maxv.y, MINV.z));
		float3 proj011 = cameraData.cameraToKinectProj(viewMatrix * make_float3(MINV.x, maxv.y, maxv.z));
		float3 proj101 = cameraData.cameraToKinectProj(viewMatrix * make_float3(maxv.x, MINV.y, maxv.z));
		float3 proj111 = cameraData.cameraToKinectProj(viewMatrix * make_float3(maxv.x, maxv.y, maxv.z));

		// Tree Reduction Min
		float3 min00 = fminf(proj000, proj100);
		float3 min01 = fminf(proj010, proj001);
		float3 min10 = fminf(proj110, proj011);
		float3 min11 = fminf(proj101, proj111);

		float3 min0 = fminf(min00, min01);
		float3 min1 = fminf(min10, min11);

		float3 minFinal = fminf(min0, min1);

		// Tree Reduction Max
		float3 max00 = fmaxf(proj000, proj100);
		float3 max01 = fmaxf(proj010, proj001);
		float3 max10 = fmaxf(proj110, proj011);
		float3 max11 = fmaxf(proj101, proj111);

		float3 max0 = fmaxf(max00, max01);
		float3 max1 = fmaxf(max10, max11);

		float3 maxFinal = fmaxf(max0, max1);

		float depth = maxFinal.z;
		if(params.m_splatMinimum == 1) {
			depth = minFinal.z;
		}
		float depthWorld = cameraData.kinectProjToCameraZ(depth);

		//uint addr = idx*4;
		//rayCastData.d_vertexBuffer[addr] = make_float4(maxFinal.x, minFinal.y, depth, depthWorld);
		//rayCastData.d_vertexBuffer[addr+1] = make_float4(minFinal.x, minFinal.y, depth, depthWorld);
		//rayCastData.d_vertexBuffer[addr+2] = make_float4(maxFinal.x, maxFinal.y, depth, depthWorld);
		//rayCastData.d_vertexBuffer[addr+3] = make_float4(minFinal.x, maxFinal.y, depth, depthWorld);

		// Note (Nick) : Changed to create point cloud instead of vertex.
		uint addr = idx;
		rayCastData.point_cloud_[addr] = make_float3(maxFinal.x, maxFinal.y, depth);
		//printf("Ray: %f\n", depth);

		uint addr = idx*6;
		rayCastData.d_vertexBuffer[addr] = make_float4(maxFinal.x, minFinal.y, depth, depthWorld);
		rayCastData.d_vertexBuffer[addr+1] = make_float4(minFinal.x, minFinal.y, depth, depthWorld);
		rayCastData.d_vertexBuffer[addr+2] = make_float4(maxFinal.x, maxFinal.y, depth, depthWorld);
		rayCastData.d_vertexBuffer[addr+3] = make_float4(minFinal.x, minFinal.y, depth, depthWorld);
		rayCastData.d_vertexBuffer[addr+4] = make_float4(maxFinal.x, maxFinal.y, depth, depthWorld);
		rayCastData.d_vertexBuffer[addr+5] = make_float4(minFinal.x, maxFinal.y, depth, depthWorld);
	}
}

extern "C" void rayIntervalSplatCUDA(const ftl::voxhash::HashData& hashData, const DepthCameraData& cameraData, const RayCastData &rayCastData, const RayCastParams &rayCastParams) 
{
	//printf("Ray casting...\n");
	const dim3 gridSize(NUM_GROUPS_X, (rayCastParams.m_numOccupiedSDFBlocks + NUM_GROUPS_X - 1) / NUM_GROUPS_X, 1);
	const dim3 blockSize(1, 1, 1);

	rayIntervalSplatKernel<<<gridSize, blockSize>>>(hashData, cameraData, rayCastData, cameraData);

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}  */
