#include "hip/hip_runtime.h"
#include "ilw_cuda.hpp"

#define T_PER_BLOCK 8

using ftl::cuda::Mask;

template <int RADIUS>
__global__ void discontinuity_kernel(ftl::cuda::TextureObject<int> mask_out, ftl::cuda::TextureObject<float> depth,
										const cv::Size size, const double minDepth, const double maxDepth) {
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < size.width && y < size.height) {
		Mask mask(0);

		const float d = depth.tex2D((int)x, (int)y);

		// Calculate depth between 0.0 and 1.0
		//float p = (d - params.minDepth) / (params.maxDepth - params.minDepth);

		if (d >= minDepth && d <= maxDepth) {
			/* Orts-Escolano S. et al. 2016. Holoportation: Virtual 3D teleportation in real-time. */
			// Is there a discontinuity nearby?
			for (int u=-RADIUS; u<=RADIUS; ++u) {
				for (int v=-RADIUS; v<=RADIUS; ++v) {
					// If yes, the flag using w = -1
					if (fabs(depth.tex2D((int)x+u, (int)y+v) - d) > 0.1f) mask.isDiscontinuity(true);
				}
			}
		}
		
		mask_out(x,y) = (int)mask;
	}
}

void ftl::cuda::discontinuity(ftl::cuda::TextureObject<int> &mask_out, ftl::cuda::TextureObject<float> &depth,
								const cv::Size size, const double minDepth, const double maxDepth,
								uint discon, hipStream_t stream) {
			
	const dim3 gridSize((size.width + T_PER_BLOCK - 1)/T_PER_BLOCK, (size.height + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	switch (discon) {
	case 5 :	discontinuity_kernel<5><<<gridSize, blockSize, 0, stream>>>(mask_out, depth, size, minDepth, maxDepth); break;
	case 4 :	discontinuity_kernel<4><<<gridSize, blockSize, 0, stream>>>(mask_out, depth, size, minDepth, maxDepth); break;
	case 3 :	discontinuity_kernel<3><<<gridSize, blockSize, 0, stream>>>(mask_out, depth, size, minDepth, maxDepth); break;
	case 2 :	discontinuity_kernel<2><<<gridSize, blockSize, 0, stream>>>(mask_out, depth, size, minDepth, maxDepth); break;
	case 1 :	discontinuity_kernel<1><<<gridSize, blockSize, 0, stream>>>(mask_out, depth, size, minDepth, maxDepth); break;
	default:	break;
	}
	cudaSafeCall( hipGetLastError() );

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
#endif
}
