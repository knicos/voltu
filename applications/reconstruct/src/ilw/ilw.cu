#include "hip/hip_runtime.h"
#include "ilw_cuda.hpp"
#include <ftl/cuda/weighting.hpp>

using ftl::cuda::TextureObject;
using ftl::rgbd::Camera;

#define WARP_SIZE 32
#define T_PER_BLOCK 8
#define FULL_MASK 0xffffffff

__device__ inline float warpMin(float e) {
	for (int i = WARP_SIZE/2; i > 0; i /= 2) {
		const float other = __shfl_xor_sync(FULL_MASK, e, i, WARP_SIZE);
		e = min(e, other);
	}
	return e;
}

__device__ inline float warpSum(float e) {
	for (int i = WARP_SIZE/2; i > 0; i /= 2) {
		const float other = __shfl_xor_sync(FULL_MASK, e, i, WARP_SIZE);
		e += other;
	}
	return e;
}

//#define COR_WIN_RADIUS 17
//#define COR_WIN_SIZE (COR_WIN_RADIUS * COR_WIN_RADIUS)

#define WINDOW_RADIUS 1

template<int COR_STEPS> 
__global__ void correspondence_energy_vector_kernel(
        TextureObject<float> d1,
        TextureObject<float> d2,
        TextureObject<uchar4> c1,
        TextureObject<uchar4> c2,
        TextureObject<float> dout,
        TextureObject<float> conf,
        float4x4 pose1,
        float4x4 pose1_inv,
        float4x4 pose2,  // Inverse
        Camera cam1,
        Camera cam2, ftl::cuda::ILWParams params) {

    // Each warp picks point in p1
    //const int tid = (threadIdx.x + threadIdx.y * blockDim.x);
	const int x = (blockIdx.x*blockDim.x + threadIdx.x); // / WARP_SIZE;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    //const float3 world1 = make_float3(p1.tex2D(x, y));
    const float depth1 = d1.tex2D(x,y); //(pose1_inv * world1).z;  // Initial starting depth
	if (depth1 < cam1.minDepth || depth1 > cam1.maxDepth) return;

	// TODO: Temporary hack to ensure depth1 is present
	//const float4 temp = vout.tex2D(x,y);
	//vout(x,y) =  make_float4(depth1, 0.0f, temp.z, temp.w);
	
	const float3 world1 = pose1 * cam1.screenToCam(x,y,depth1);

    const uchar4 colour1 = c1.tex2D(x, y);

    float bestcost = 1.1f;
    float avgcost = 0.0f;
    float bestdepth;
    int count = 0;
    
	const float step_interval = params.range / (COR_STEPS / 2);
	
	const float3 rayStep_world = pose1.getFloat3x3() * cam1.screenToCam(x,y,step_interval);
	const float3 rayStart_2 = pose2 * world1;
	const float3 rayStep_2 = pose2.getFloat3x3() * rayStep_world;

    // Project to p2 using cam2
    // Each thread takes a possible correspondence and calculates a weighting
    //const int lane = tid % WARP_SIZE;
	for (int i=0; i<COR_STEPS; ++i) {
		const int j = i - (COR_STEPS/2);
		const float depth_adjust = (float)j * step_interval + depth1;

        // Calculate adjusted depth 3D point in camera 2 space
        const float3 worldPos = world1 + j * rayStep_world; //(pose1 * cam1.screenToCam(x, y, depth_adjust));
        const float3 camPos = rayStart_2 + j * rayStep_2; //pose2 * worldPos;
        const uint2 screen = cam2.camToScreen<uint2>(camPos);

        if (screen.x >= cam2.width || screen.y >= cam2.height) continue;

        // Small window around suggested point
        //for (int u=-WINDOW_RADIUS; u<=WINDOW_RADIUS; ++u) {
        //for (int v=-WINDOW_RADIUS; v<=WINDOW_RADIUS; ++v) {
        //const int u = 0;
        //const int v = 0;

            // Now do correspondence evaluation at "screen" location in camera 2
            //const float3 world2 = make_float3(p2.tex2D((int)screen.x+u, (int)screen.y+v));
			//if ((params.flags & ftl::cuda::kILWFlag_IgnoreBad) && world2.x == MINF) continue;
			

			const float depth2 = d2.tex2D((int)screen.x, (int)screen.y);

            // Determine degree of correspondence
            float cost = 1.0f - ftl::cuda::weighting(fabs(depth2 - camPos.z), params.spatial_smooth);
            // Point is too far away to even count
			if (cost == 1.0f) continue;
			
			const uchar4 colour2 = c2.tex2D((int)screen.x, (int)screen.y);

            // Mix ratio of colour and distance costs
            const float ccost = 1.0f - ftl::cuda::colourWeighting(colour1, colour2, params.colour_smooth);
			if ((params.flags & ftl::cuda::kILWFlag_SkipBadColour) && ccost == 1.0f) continue;
			
			// Cost eq 1: summed contributions
			cost = params.cost_ratio * (ccost) + (1.0f - params.cost_ratio) * cost;
			
			// Cost eq 2: Multiplied
			//cost = ccost * cost * cost * cost;

            ++count;
            avgcost += (params.flags & ftl::cuda::kILWFlag_ColourConfidenceOnly) ? ccost : cost;
            if (cost < bestcost) {
                bestdepth = depth_adjust;
                bestcost = cost;
            }

        //}
        //}
    }

	//count = warpSum(count);
    const float mincost = bestcost; //warpMin(bestcost);
	//bool best = mincost == bestcost;
	avgcost /= count;
    const float confidence = (params.flags & ftl::cuda::kILWFlag_ColourConfidenceOnly) ? avgcost : (avgcost - mincost);

    if (mincost < 1.0f) {
        //float3 tvecA = pose1 * cam1.screenToCam(x, y, bestdepth);
        //float3 tvecB = pose1 * world1;
        //if (params.flags & ftl::cuda::kILWFlag_RestrictZ) {
        //    tvecA.x = tvecB.x;
        //    tvecA.y = tvecB.y;
        //}
        //tvecA = tvecA - world1;
        float old = conf.tex2D(x,y);

        if ((1.0f - mincost) * confidence > old) {
			dout(x,y) = bestdepth;
			conf(x,y) = (1.0f - mincost) * confidence;
		}
			
		//eout(x,y) = max(eout(x,y), (length(bestpoint-world1) / 0.04f) * 7.0f);
		//eout(x,y) = max(eout(x,y), (1.0f - mincost) * 7.0f);
		//eout(x,y) = max(eout(x, y), (1.0f - mincost) * confidence * (length(bestpoint-world1) / 0.04f) * 12.0f);
		
		//eout(x,y) = max(eout(x, y), confidence * 12.0f);
    }
}

void ftl::cuda::correspondence(
        TextureObject<float> &d1,
        TextureObject<float> &d2,
        TextureObject<uchar4> &c1,
        TextureObject<uchar4> &c2,
        TextureObject<float> &dout,
        TextureObject<float> &conf,
        float4x4 &pose1,
        float4x4 &pose1_inv,
        float4x4 &pose2,
        const Camera &cam1,
        const Camera &cam2, const ILWParams &params, int win,
        hipStream_t stream) {

	const dim3 gridSize((d1.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (d1.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    //printf("COR SIZE %d,%d\n", p1.width(), p1.height());

    correspondence_energy_vector_kernel<16><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, dout, conf, pose1, pose1_inv, pose2, cam1, cam2, params);

    //switch (win) {
    //case 17     : correspondence_energy_vector_kernel<17><<<gridSize, blockSize, 0, stream>>>(p1, p2, c1, c2, vout, eout, pose1, pose1_inv, pose2, cam1, cam2, params); break;
    //case 9      : correspondence_energy_vector_kernel<9><<<gridSize, blockSize, 0, stream>>>(p1, p2, c1, c2, vout, eout, pose1, pose1_inv, pose2, cam1, cam2, params); break;
    //case 5      : correspondence_energy_vector_kernel<5><<<gridSize, blockSize, 0, stream>>>(p1, p2, c1, c2, vout, eout, pose1, pose1_inv, pose2, cam1, cam2, params); break;
    //}
    cudaSafeCall( hipGetLastError() );
}

//==============================================================================

//#define MOTION_RADIUS 9

template <int MOTION_RADIUS>
__global__ void move_points_kernel(
    ftl::cuda::TextureObject<float> d_old,
	ftl::cuda::TextureObject<float> d_new,
	ftl::cuda::TextureObject<float> conf,
    ftl::rgbd::Camera camera,
	float4x4 pose,
	ftl::cuda::ILWParams params,
    float rate) {

    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	const float d0_new = d_new.tex2D((int)x,(int)y);
	const float d0_old = d_old.tex2D((int)x,(int)y);
	if (d0_new == 0.0f) return;  // No correspondence found
    
    if (x < d_old.width() && y < d_old.height()) {
		//const float4 world = p(x,y);
		//if (world.x == MINF) return;

		float delta = 0.0f; //make_float4(0.0f, 0.0f, 0.0f, 0.0f); //ev.tex2D((int)x,(int)y);
		float contrib = 0.0f;

		// Calculate screen space distortion with neighbours
		for (int v=-MOTION_RADIUS; v<=MOTION_RADIUS; ++v) {
			for (int u=-MOTION_RADIUS; u<=MOTION_RADIUS; ++u) {
				const float dn_new = d_new.tex2D((int)x+u,(int)y+v);
				const float dn_old = d_old.tex2D((int)x+u,(int)y+v);
				const float confn = conf.tex2D((int)x+u,(int)y+v);
				//const float3 pn = make_float3(p.tex2D((int)x+u,(int)y+v));
				//if (pn.x == MINF) continue;
				if (dn_new == 0.0f) continue;  // Neighbour has no new correspondence

				const float s = ftl::cuda::weighting(fabs(d0_new - dn_new), params.range);
				contrib += (confn+0.01f) * s;
				delta += (confn+0.01f) * s * ((confn == 0.0f) ? dn_old : dn_new);
			}
		}

        if (contrib > 0.0f) {
            //const float3 newworld = pose * camera.screenToCam(x, y, vec0.x + rate * ((delta / contrib) - vec0.x));
			//p(x,y) = make_float4(newworld, world.w); //world + rate * (vec / contrib);
			
			d_old(x,y) = d0_old + rate * ((delta / contrib) - d0_old);
        }
    }
}


void ftl::cuda::move_points(
        ftl::cuda::TextureObject<float> &d_old,
		ftl::cuda::TextureObject<float> &d_new,
		ftl::cuda::TextureObject<float> &conf,
        const ftl::rgbd::Camera &camera,
		const float4x4 &pose,
		const ftl::cuda::ILWParams &params,
        float rate,
        int radius,
        hipStream_t stream) {

    const dim3 gridSize((d_old.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (d_old.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    switch (radius) {
    case 9 : move_points_kernel<9><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    case 5 : move_points_kernel<5><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    case 3 : move_points_kernel<3><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    case 1 : move_points_kernel<1><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    case 0 : move_points_kernel<0><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    }

    cudaSafeCall( hipGetLastError() );
}
