#include "hip/hip_runtime.h"
#include "ilw_cuda.hpp"
#include <ftl/cuda/weighting.hpp>

using ftl::cuda::TextureObject;
using ftl::rgbd::Camera;

#define WARP_SIZE 32
#define T_PER_BLOCK 8
#define FULL_MASK 0xffffffff

__device__ inline float warpMin(float e) {
	for (int i = WARP_SIZE/2; i > 0; i /= 2) {
		const float other = __shfl_xor_sync(FULL_MASK, e, i, WARP_SIZE);
		e = min(e, other);
	}
	return e;
}

__device__ inline float warpSum(float e) {
	for (int i = WARP_SIZE/2; i > 0; i /= 2) {
		const float other = __shfl_xor_sync(FULL_MASK, e, i, WARP_SIZE);
		e += other;
	}
	return e;
}

//==============================================================================

template <int RADIUS>
__global__ void preprocess_kernel(
    	ftl::cuda::TextureObject<float> depth_in,
		ftl::cuda::TextureObject<float> depth_out,
		ftl::cuda::TextureObject<uchar4> colour,
		ftl::cuda::TextureObject<int> mask,
		ftl::rgbd::Camera camera,
		ftl::cuda::ILWParams params) {

    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float d = depth_in.tex2D((int)x,(int)y);
	uchar4 c = colour.tex2D((int)x,(int)y);

	// Calculate discontinuity mask

	// Fill missing depths
	if (d < camera.minDepth || d > camera.maxDepth) {
		float depth_accum = 0.0f;
		float contrib = 0.0f;

		for (int v=-RADIUS; v<=RADIUS; ++v) {
			for (int u=-RADIUS; u<=RADIUS; ++u) {
				uchar4 c2 = colour.tex2D((int)x+u,(int)y+v);
				float d2 = depth_in.tex2D((int)x+u,(int)y+v);
				if (d2 >= camera.minDepth && d2 <= camera.maxDepth) {
					float w = ftl::cuda::colourWeighting(c, c2, params.colour_smooth);
					depth_accum += d2*w;
					contrib += w;
				}
			}
		}

		if (contrib >= 0.0f) d = depth_accum / contrib;
	}

	depth_out(x,y) = d;
}

void ftl::cuda::preprocess_depth(
		ftl::cuda::TextureObject<float> &depth_in,
		ftl::cuda::TextureObject<float> &depth_out,
		ftl::cuda::TextureObject<uchar4> &colour,
		ftl::cuda::TextureObject<int> &mask,
		const ftl::rgbd::Camera &camera,
		const ftl::cuda::ILWParams &params,
		hipStream_t stream) {

	const dim3 gridSize((depth_in.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (depth_in.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	preprocess_kernel<3><<<gridSize, blockSize, 0, stream>>>(depth_in, depth_out, colour, mask, camera, params);

	cudaSafeCall( hipGetLastError() );
}

//==============================================================================

template<int FUNCTION>
__device__ float costFunction(const ftl::cuda::ILWParams &params, float dweight, float cweight);

template <>
__device__ inline float costFunction<0>(const ftl::cuda::ILWParams &params, float dweight, float cweight) {
	return 1.0f - (params.cost_ratio * (cweight) + (1.0f - params.cost_ratio) * dweight);
}

template <>
__device__ inline float costFunction<1>(const ftl::cuda::ILWParams &param, float dweight, float cweight) {
	return 1.0f - (cweight * cweight * dweight);
}

template <>
__device__ inline float costFunction<2>(const ftl::cuda::ILWParams &param, float dweight, float cweight) {
	return 1.0f - (dweight * dweight * cweight);
}

template <>
__device__ inline float costFunction<3>(const ftl::cuda::ILWParams &params, float dweight, float cweight) {
	return (dweight == 0.0f) ? 1.0f : 1.0f - (params.cost_ratio * (cweight) + (1.0f - params.cost_ratio) * dweight);
}

template<int COR_STEPS, int FUNCTION> 
__global__ void correspondence_energy_vector_kernel(
        TextureObject<float> d1,
        TextureObject<float> d2,
        TextureObject<uchar4> c1,
        TextureObject<uchar4> c2,
        TextureObject<float> dout,
        TextureObject<float> conf,
        float4x4 pose1,
        float4x4 pose1_inv,
        float4x4 pose2,  // Inverse
        Camera cam1,
        Camera cam2, ftl::cuda::ILWParams params) {

    // Each warp picks point in p1
    //const int tid = (threadIdx.x + threadIdx.y * blockDim.x);
	const int x = (blockIdx.x*blockDim.x + threadIdx.x); // / WARP_SIZE;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    //const float3 world1 = make_float3(p1.tex2D(x, y));
    const float depth1 = d1.tex2D(x,y); //(pose1_inv * world1).z;  // Initial starting depth
	if (depth1 < cam1.minDepth || depth1 > cam1.maxDepth) return;

	// TODO: Temporary hack to ensure depth1 is present
	//const float4 temp = vout.tex2D(x,y);
	//vout(x,y) =  make_float4(depth1, 0.0f, temp.z, temp.w);
	
	const float3 world1 = pose1 * cam1.screenToCam(x,y,depth1);

    const uchar4 colour1 = c1.tex2D(x, y);

    float bestcost = 1.1f;
    float avgcost = 0.0f;
    float bestdepth;
    int count = 0;
    
	const float step_interval = params.range / (COR_STEPS / 2);
	
	const float3 rayStep_world = pose1.getFloat3x3() * cam1.screenToCam(x,y,step_interval);
	const float3 rayStart_2 = pose2 * world1;
	const float3 rayStep_2 = pose2.getFloat3x3() * rayStep_world;

    // Project to p2 using cam2
    // Each thread takes a possible correspondence and calculates a weighting
    //const int lane = tid % WARP_SIZE;
	for (int i=0; i<COR_STEPS; ++i) {
		const int j = i - (COR_STEPS/2);
		const float depth_adjust = (float)j * step_interval + depth1;

        // Calculate adjusted depth 3D point in camera 2 space
        const float3 worldPos = world1 + j * rayStep_world; //(pose1 * cam1.screenToCam(x, y, depth_adjust));
        const float3 camPos = rayStart_2 + j * rayStep_2; //pose2 * worldPos;
        const uint2 screen = cam2.camToScreen<uint2>(camPos);

        if (screen.x >= cam2.width || screen.y >= cam2.height) continue;

		// Generate a depth correspondence value
		const float depth2 = d2.tex2D((int)screen.x, (int)screen.y);
		const float dweight = ftl::cuda::weighting(fabs(depth2 - camPos.z), params.spatial_smooth);
		
		// Generate a colour correspondence value
		const uchar4 colour2 = c2.tex2D((int)screen.x, (int)screen.y);
		const float cweight = ftl::cuda::colourWeighting(colour1, colour2, params.colour_smooth);

		const float cost = costFunction<FUNCTION>(params, dweight, cweight);

		// Cost is so bad, don't even consider this a valid option
		if (cost >= params.cost_threshold) continue;

		++count;
		avgcost += cost;
		if (cost < bestcost) {
			bestdepth = depth_adjust;
			bestcost = cost;
		}
    }

	//count = warpSum(count);
    const float mincost = bestcost; //warpMin(bestcost);
	//bool best = mincost == bestcost;
	avgcost /= count;
    const float confidence = (params.flags & ftl::cuda::kILWFlag_ColourConfidenceOnly) ? avgcost : (avgcost - mincost);

    if (mincost < 1.0f) {
        float old = conf.tex2D(x,y);

        if ((1.0f - mincost) * confidence > old) {
			dout(x,y) = bestdepth;
			conf(x,y) = (1.0f - mincost) * confidence;
		}
    }
}

void ftl::cuda::correspondence(
        TextureObject<float> &d1,
        TextureObject<float> &d2,
        TextureObject<uchar4> &c1,
        TextureObject<uchar4> &c2,
        TextureObject<float> &dout,
        TextureObject<float> &conf,
        float4x4 &pose1,
        float4x4 &pose1_inv,
        float4x4 &pose2,
        const Camera &cam1,
        const Camera &cam2, const ILWParams &params, int func,
        hipStream_t stream) {

	const dim3 gridSize((d1.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (d1.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    //printf("COR SIZE %d,%d\n", p1.width(), p1.height());

	switch (func) {
    case 0: correspondence_energy_vector_kernel<16,0><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, dout, conf, pose1, pose1_inv, pose2, cam1, cam2, params);
	case 1: correspondence_energy_vector_kernel<16,1><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, dout, conf, pose1, pose1_inv, pose2, cam1, cam2, params);
	case 2: correspondence_energy_vector_kernel<16,2><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, dout, conf, pose1, pose1_inv, pose2, cam1, cam2, params);
	case 3: correspondence_energy_vector_kernel<16,3><<<gridSize, blockSize, 0, stream>>>(d1, d2, c1, c2, dout, conf, pose1, pose1_inv, pose2, cam1, cam2, params);
	}

    cudaSafeCall( hipGetLastError() );
}

//==============================================================================

//#define MOTION_RADIUS 9

template <int MOTION_RADIUS>
__global__ void move_points_kernel(
    ftl::cuda::TextureObject<float> d_old,
	ftl::cuda::TextureObject<float> d_new,
	ftl::cuda::TextureObject<float> conf,
    ftl::rgbd::Camera camera,
	float4x4 pose,
	ftl::cuda::ILWParams params,
    float rate) {

    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	const float d0_new = d_new.tex2D((int)x,(int)y);
	const float d0_old = d_old.tex2D((int)x,(int)y);
	if (d0_new == 0.0f) return;  // No correspondence found
    
    if (x < d_old.width() && y < d_old.height()) {
		//const float4 world = p(x,y);
		//if (world.x == MINF) return;

		float delta = 0.0f; //make_float4(0.0f, 0.0f, 0.0f, 0.0f); //ev.tex2D((int)x,(int)y);
		float contrib = 0.0f;

		// Calculate screen space distortion with neighbours
		for (int v=-MOTION_RADIUS; v<=MOTION_RADIUS; ++v) {
			for (int u=-MOTION_RADIUS; u<=MOTION_RADIUS; ++u) {
				const float dn_new = d_new.tex2D((int)x+u,(int)y+v);
				const float dn_old = d_old.tex2D((int)x+u,(int)y+v);
				const float confn = conf.tex2D((int)x+u,(int)y+v);
				//const float3 pn = make_float3(p.tex2D((int)x+u,(int)y+v));
				//if (pn.x == MINF) continue;
				if (dn_new == 0.0f) continue;  // Neighbour has no new correspondence

				const float s = ftl::cuda::weighting(fabs(d0_new - dn_new), params.range);
				contrib += (confn+0.01f) * s;
				delta += (confn+0.01f) * s * ((confn == 0.0f) ? dn_old : dn_new);
			}
		}

        if (contrib > 0.0f) {
            //const float3 newworld = pose * camera.screenToCam(x, y, vec0.x + rate * ((delta / contrib) - vec0.x));
			//p(x,y) = make_float4(newworld, world.w); //world + rate * (vec / contrib);
			
			d_old(x,y) = d0_old + rate * ((delta / contrib) - d0_old);
        }
    }
}


void ftl::cuda::move_points(
        ftl::cuda::TextureObject<float> &d_old,
		ftl::cuda::TextureObject<float> &d_new,
		ftl::cuda::TextureObject<float> &conf,
        const ftl::rgbd::Camera &camera,
		const float4x4 &pose,
		const ftl::cuda::ILWParams &params,
        float rate,
        int radius,
        hipStream_t stream) {

    const dim3 gridSize((d_old.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (d_old.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    switch (radius) {
    case 9 : move_points_kernel<9><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    case 5 : move_points_kernel<5><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    case 3 : move_points_kernel<3><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    case 1 : move_points_kernel<1><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    case 0 : move_points_kernel<0><<<gridSize, blockSize, 0, stream>>>(d_old,d_new,conf,camera, pose, params, rate); break;
    }

    cudaSafeCall( hipGetLastError() );
}
