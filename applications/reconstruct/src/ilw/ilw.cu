#include "hip/hip_runtime.h"
#include "ilw_cuda.hpp"
#include <ftl/cuda/weighting.hpp>

using ftl::cuda::TextureObject;
using ftl::rgbd::Camera;

#define WARP_SIZE 32
#define T_PER_BLOCK 8
#define FULL_MASK 0xffffffff

__device__ inline float warpMin(float e) {
	for (int i = WARP_SIZE/2; i > 0; i /= 2) {
		const float other = __shfl_xor_sync(FULL_MASK, e, i, WARP_SIZE);
		e = min(e, other);
	}
	return e;
}

__device__ inline float warpSum(float e) {
	for (int i = WARP_SIZE/2; i > 0; i /= 2) {
		const float other = __shfl_xor_sync(FULL_MASK, e, i, WARP_SIZE);
		e += other;
	}
	return e;
}

//#define COR_WIN_RADIUS 17
//#define COR_WIN_SIZE (COR_WIN_RADIUS * COR_WIN_RADIUS)

template<int COR_STEPS> 
__global__ void correspondence_energy_vector_kernel(
        TextureObject<float4> p1,
        TextureObject<float4> p2,
        TextureObject<uchar4> c1,
        TextureObject<uchar4> c2,
        TextureObject<float4> vout,
        TextureObject<float> eout,
        float4x4 pose1,
        float4x4 pose1_inv,
        float4x4 pose2,  // Inverse
        Camera cam1,
        Camera cam2, ftl::cuda::ILWParams params) {

    // Each warp picks point in p1
    const int tid = (threadIdx.x + threadIdx.y * blockDim.x);
	const int x = (blockIdx.x*blockDim.x + threadIdx.x) / WARP_SIZE;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    const float3 world1 = make_float3(p1.tex2D(x, y));
    const float depth1 = (pose1_inv * world1).z;  // Initial starting depth
    if (depth1 < cam1.minDepth || depth1 > cam1.maxDepth) return;

    const uchar4 colour1 = c1.tex2D(x, y);

    float bestcost = 1.1f;
    float avgcost = 0.0f;
    float bestdepth;
    int count = 0;
    
    const float step_interval = 0.05f / COR_STEPS;

    // Project to p2 using cam2
    // Each thread takes a possible correspondence and calculates a weighting
    const int lane = tid % WARP_SIZE;
	for (int i=lane; i<COR_STEPS; i+=WARP_SIZE) {
        const float depth_adjust = (float)(i - (COR_STEPS / 2)) * step_interval + depth1;

        // Calculate adjusted depth 3D point in camera 2 space
        const float3 worldPos = (pose1 * cam1.screenToCam(x, y, depth_adjust));
        const float3 camPos = pose2 * worldPos;
        const uint2 screen = cam2.camToScreen<uint2>(camPos);

        if (screen.x >= cam2.width || screen.y >= cam2.height) continue;

        // Now do correspondence evaluation at "screen" location in camera 2
        const float3 world2 = make_float3(p2.tex2D((int)screen.x, (int)screen.y));
        if ((params.flags & ftl::cuda::kILWFlag_IgnoreBad) && world2.x == MINF) continue;
        const uchar4 colour2 = c2.tex2D((int)screen.x, (int)screen.y);

        // Determine degree of correspondence
		float cost = 1.0f - ftl::cuda::spatialWeighting(world1, world2, params.spatial_smooth);
		// Point is too far away to even count
		if (world2.x != MINF && cost == 1.0f) continue;

        // Mix ratio of colour and distance costs
        const float ccost = 1.0f - ftl::cuda::colourWeighting(colour1, colour2, params.colour_smooth);
        if ((params.flags & ftl::cuda::kILWFlag_SkipBadColour) && ccost == 1.0f) continue;
        cost = params.cost_ratio * (ccost) + (1.0f - params.cost_ratio) * cost;
        //cost /= 2.0f;

		++count;
		avgcost += cost;
        if (world2.x != MINF && cost < bestcost) {
            bestdepth = depth_adjust;
            bestcost = cost;
        }
    }

	count = warpSum(count);
    const float mincost = warpMin(bestcost);
	bool best = mincost == bestcost;
	avgcost = warpSum(avgcost) / count;
    const float confidence = (avgcost - mincost);

    // FIXME: Multiple threads in warp could match this.
    if (best && mincost < 1.0f) {
        float3 tvecA = pose1 * cam1.screenToCam(x, y, bestdepth);
        //float3 tvecB = pose1 * world1;
        //if (params.flags & ftl::cuda::kILWFlag_RestrictZ) {
        //    tvecA.x = tvecB.x;
        //    tvecA.y = tvecB.y;
        //}
        tvecA = tvecA - world1;
        vout(x,y) =  make_float4(
            tvecA.x, // * (1.0f - mincost) * confidence,
            tvecA.y, // * (1.0f - mincost) * confidence,
            tvecA.z, // * (1.0f - mincost) * confidence,
            (1.0f - mincost) * confidence);
			
		//eout(x,y) = max(eout(x,y), (length(bestpoint-world1) / 0.04f) * 7.0f);
		//eout(x,y) = max(eout(x,y), (1.0f - mincost) * 7.0f);
		//eout(x,y) = max(eout(x, y), (1.0f - mincost) * confidence * (length(bestpoint-world1) / 0.04f) * 12.0f);
		eout(x,y) = max(eout(x, y), (1.0f - mincost) * confidence * 12.0f);
		//eout(x,y) = max(eout(x, y), confidence * 12.0f);
    } else if (mincost >= 1.0f && lane == 0) {
        //vout(x,y) = make_float4(0.0f);
        //eout(x,y) = 0.0f;
    }
}

void ftl::cuda::correspondence_energy_vector(
        TextureObject<float4> &p1,
        TextureObject<float4> &p2,
        TextureObject<uchar4> &c1,
        TextureObject<uchar4> &c2,
        TextureObject<float4> &vout,
        TextureObject<float> &eout,
        float4x4 &pose1,
        float4x4 &pose1_inv,
        float4x4 &pose2,
        const Camera &cam1,
        const Camera &cam2, const ILWParams &params, int win,
        hipStream_t stream) {

    const dim3 gridSize((p1.width() + 2 - 1)/2, (p1.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(2*WARP_SIZE, T_PER_BLOCK);

    //printf("COR SIZE %d,%d\n", p1.width(), p1.height());

    correspondence_energy_vector_kernel<64><<<gridSize, blockSize, 0, stream>>>(p1, p2, c1, c2, vout, eout, pose1, pose1_inv, pose2, cam1, cam2, params);

    //switch (win) {
    //case 17     : correspondence_energy_vector_kernel<17><<<gridSize, blockSize, 0, stream>>>(p1, p2, c1, c2, vout, eout, pose1, pose1_inv, pose2, cam1, cam2, params); break;
    //case 9      : correspondence_energy_vector_kernel<9><<<gridSize, blockSize, 0, stream>>>(p1, p2, c1, c2, vout, eout, pose1, pose1_inv, pose2, cam1, cam2, params); break;
    //case 5      : correspondence_energy_vector_kernel<5><<<gridSize, blockSize, 0, stream>>>(p1, p2, c1, c2, vout, eout, pose1, pose1_inv, pose2, cam1, cam2, params); break;
    //}
    cudaSafeCall( hipGetLastError() );
}

//==============================================================================

//#define MOTION_RADIUS 9

template <int MOTION_RADIUS>
__global__ void move_points_kernel(
    ftl::cuda::TextureObject<float4> p,
    ftl::cuda::TextureObject<float4> ev,
    ftl::rgbd::Camera camera,
    float rate) {

    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    if (x < p.width() && y < p.height()) {
		const float4 world = p(x,y);
		if (world.x == MINF) return;

		float4 vec = make_float4(0.0f, 0.0f, 0.0f, 0.0f); //ev.tex2D((int)x,(int)y);
		float contrib = 0.0f;

		// Calculate screen space distortion with neighbours
		for (int v=-MOTION_RADIUS; v<=MOTION_RADIUS; ++v) {
			for (int u=-MOTION_RADIUS; u<=MOTION_RADIUS; ++u) {
				const float4 vecn = ev.tex2D((int)x+u,(int)y+v);
				const float3 pn = make_float3(p.tex2D((int)x+u,(int)y+v));
				if (pn.x == MINF) continue;

				const float s = ftl::cuda::spatialWeighting(pn, make_float3(world), 0.01f);
				contrib += vecn.w * s;
				vec += vecn.w * s * vecn;
			}
		}

        if (vec.w > 0.0f) {
            p(x,y) = world + rate * (vec / contrib);
        }
    }
}


void ftl::cuda::move_points(
        ftl::cuda::TextureObject<float4> &p,
        ftl::cuda::TextureObject<float4> &v,
        const ftl::rgbd::Camera &camera,
        float rate,
        int radius,
        hipStream_t stream) {

    const dim3 gridSize((p.width() + T_PER_BLOCK - 1)/T_PER_BLOCK, (p.height() + T_PER_BLOCK - 1)/T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    switch (radius) {
    case 9 : move_points_kernel<9><<<gridSize, blockSize, 0, stream>>>(p,v,camera,rate); break;
    case 5 : move_points_kernel<5><<<gridSize, blockSize, 0, stream>>>(p,v,camera,rate); break;
    case 3 : move_points_kernel<3><<<gridSize, blockSize, 0, stream>>>(p,v,camera,rate); break;
    case 1 : move_points_kernel<1><<<gridSize, blockSize, 0, stream>>>(p,v,camera,rate); break;
    case 0 : move_points_kernel<0><<<gridSize, blockSize, 0, stream>>>(p,v,camera,rate); break;
    }

    cudaSafeCall( hipGetLastError() );
}
