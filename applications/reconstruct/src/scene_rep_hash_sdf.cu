#include "hip/hip_runtime.h"
// From: https://github.com/niessner/VoxelHashing/blob/master/DepthSensingCUDA/Source/CUDASceneRepHashSDF.cu

//#include <cutil_inline.h>
//#include <cutil_math.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>

#include <ftl/cuda_matrix_util.hpp>

#include <ftl/voxel_hash.hpp>
#include <ftl/depth_camera.hpp>
#include <ftl/ray_cast_params.hpp>

#define T_PER_BLOCK 8

using ftl::voxhash::HashData;
using ftl::voxhash::HashParams;
using ftl::voxhash::Voxel;
using ftl::voxhash::HashEntry;
using ftl::voxhash::FREE_ENTRY;

// TODO (Nick) Use ftl::cuda::Texture (texture objects)
//texture<float, hipTextureType2D, hipReadModeElementType> depthTextureRef;
//texture<float4, hipTextureType2D, hipReadModeElementType> colorTextureRef;

__device__ __constant__ HashParams c_hashParams;
__device__ __constant__ RayCastParams c_rayCastParams;
__device__ __constant__ DepthCameraParams c_depthCameraParams;

extern "C" void updateConstantHashParams(const HashParams& params) {

	size_t size;
	cudaSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_hashParams)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_hashParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
	}


extern "C" void updateConstantRayCastParams(const RayCastParams& params) {
	//printf("Update ray cast params\n");
	size_t size;
	cudaSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_rayCastParams)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_rayCastParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif

}

extern "C" void updateConstantDepthCameraParams(const DepthCameraParams& params) {
	//printf("Update depth camera params\n");
	size_t size;
	cudaSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_depthCameraParams)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_depthCameraParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif

}

extern "C" void bindInputDepthColorTextures(const DepthCameraData& depthCameraData) 
{
	/*cudaSafeCall(hipBindTextureToArray(depthTextureRef, depthCameraData.d_depthArray, depthCameraData.h_depthChannelDesc));
	cudaSafeCall(hipBindTextureToArray(colorTextureRef, depthCameraData.d_colorArray, depthCameraData.h_colorChannelDesc));
	depthTextureRef.filterMode = hipFilterModePoint;
	colorTextureRef.filterMode = hipFilterModePoint;*/
}

__global__ void resetHeapKernel(HashData hashData) 
{
	const HashParams& hashParams = c_hashParams;
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx == 0) {
		hashData.d_heapCounter[0] = hashParams.m_numSDFBlocks - 1;	//points to the last element of the array
	}
	
	if (idx < hashParams.m_numSDFBlocks) {

		hashData.d_heap[idx] = hashParams.m_numSDFBlocks - idx - 1;
		uint blockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
		uint base_idx = idx * blockSize;
		for (uint i = 0; i < blockSize; i++) {
			hashData.deleteVoxel(base_idx+i);
		}
	}
}

__global__ void resetHashKernel(HashData hashData) 
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		hashData.deleteHashEntry(hashData.d_hash[idx]);
		hashData.deleteHashEntry(hashData.d_hashCompactified[idx]);
	}
}


__global__ void resetHashBucketMutexKernel(HashData hashData) 
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < hashParams.m_hashNumBuckets) {
		hashData.d_hashBucketMutex[idx] = FREE_ENTRY;
	}
}

extern "C" void resetCUDA(HashData& hashData, const HashParams& hashParams)
{
	{
		//resetting the heap and SDF blocks
		const dim3 gridSize((hashParams.m_numSDFBlocks + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
		const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

		resetHeapKernel<<<gridSize, blockSize>>>(hashData);


		#ifdef _DEBUG
			cudaSafeCall(hipDeviceSynchronize());
			//cutilCheckMsg(__FUNCTION__);
		#endif

	}

	{
		//resetting the hash
		const dim3 gridSize((HASH_BUCKET_SIZE * hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
		const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

		resetHashKernel<<<gridSize, blockSize>>>(hashData);

		#ifdef _DEBUG
			cudaSafeCall(hipDeviceSynchronize());
			//cutilCheckMsg(__FUNCTION__);
		#endif
	}

	{
		//resetting the mutex
		const dim3 gridSize((hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
		const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

		resetHashBucketMutexKernel<<<gridSize, blockSize>>>(hashData);

		#ifdef _DEBUG
			cudaSafeCall(hipDeviceSynchronize());
			//cutilCheckMsg(__FUNCTION__);
		#endif
	}


}

extern "C" void resetHashBucketMutexCUDA(HashData& hashData, const HashParams& hashParams)
{
	const dim3 gridSize((hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
	const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

	resetHashBucketMutexKernel<<<gridSize, blockSize>>>(hashData);

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}


__device__
unsigned int linearizeChunkPos(const int3& chunkPos)
{
	int3 p = chunkPos-c_hashParams.m_streamingMinGridPos;
	return  p.z * c_hashParams.m_streamingGridDimensions.x * c_hashParams.m_streamingGridDimensions.y +
			p.y * c_hashParams.m_streamingGridDimensions.x +
			p.x;
}

__device__
int3 worldToChunks(const float3& posWorld)
{
	float3 p;
	p.x = posWorld.x/c_hashParams.m_streamingVoxelExtents.x;
	p.y = posWorld.y/c_hashParams.m_streamingVoxelExtents.y;
	p.z = posWorld.z/c_hashParams.m_streamingVoxelExtents.z;

	float3 s;
	s.x = (float)sign(p.x);
	s.y = (float)sign(p.y);
	s.z = (float)sign(p.z);

	return make_int3(p+s*0.5f);
}

__device__
bool isSDFBlockStreamedOut(const int3& sdfBlock, const HashData& hashData, const unsigned int* d_bitMask)	//TODO MATTHIAS (-> move to HashData)
{
	float3 posWorld = hashData.virtualVoxelPosToWorld(hashData.SDFBlockToVirtualVoxelPos(sdfBlock)); // sdfBlock is assigned to chunk by the bottom right sample pos

	uint index = linearizeChunkPos(worldToChunks(posWorld));
	uint nBitsInT = 32;
	return ((d_bitMask[index/nBitsInT] & (0x1 << (index%nBitsInT))) != 0x0);
}

// Note: bitMask used for Streaming out code... could be set to nullptr if not streaming out
// Note: Allocations might need to be around fat rays since multiple voxels could correspond
// to same depth map pixel at larger distances.
__global__ void allocKernel(HashData hashData, DepthCameraData cameraData, const unsigned int* d_bitMask) 
{
	const HashParams& hashParams = c_hashParams;
	const DepthCameraParams& cameraParams = c_depthCameraParams;

	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < cameraParams.m_imageWidth && y < cameraParams.m_imageHeight)
	{
		float d = tex2D<float>(cameraData.depth_obj_, x, y);
		//if (d == MINF || d < cameraParams.m_sensorDepthWorldMin || d > cameraParams.m_sensorDepthWorldMax)	return;
		if (d == MINF || d == 0.0f)	return;

		if (d >= hashParams.m_maxIntegrationDistance) return;

		// TODO (Nick) Use covariance to include a frustrum of influence
		float t = hashData.getTruncation(d);
		float minDepth = min(hashParams.m_maxIntegrationDistance, d-t);
		float maxDepth = min(hashParams.m_maxIntegrationDistance, d+t);
		if (minDepth >= maxDepth) return;

		// Convert ray from image coords to world
		// Does kinectDepthToSkeleton convert pixel values to coordinates using
		// camera intrinsics? Same as what reprojectTo3D does in OpenCV?
		float3 rayMin = cameraData.kinectDepthToSkeleton(x, y, minDepth);
		// Is the rigid transform then the estimated camera pose?
		rayMin = hashParams.m_rigidTransform * rayMin;
		//printf("Ray min: %f,%f,%f\n", rayMin.x, rayMin.y, rayMin.z);
		float3 rayMax = cameraData.kinectDepthToSkeleton(x, y, maxDepth);
		rayMax = hashParams.m_rigidTransform * rayMax;

		float3 rayDir = normalize(rayMax - rayMin);
	
		// Only ray cast from min possible depth to max depth
		int3 idCurrentVoxel = hashData.worldToSDFBlock(rayMin);
		int3 idEnd = hashData.worldToSDFBlock(rayMax);
		
		float3 step = make_float3(sign(rayDir));
		float3 boundaryPos = hashData.SDFBlockToWorld(idCurrentVoxel+make_int3(clamp(step, 0.0, 1.0f)))-0.5f*hashParams.m_virtualVoxelSize;
		float3 tMax = (boundaryPos-rayMin)/rayDir;
		float3 tDelta = (step*SDF_BLOCK_SIZE*hashParams.m_virtualVoxelSize)/rayDir;
		int3 idBound = make_int3(make_float3(idEnd)+step);

		//#pragma unroll
		//for(int c = 0; c < 3; c++) {
		//	if (rayDir[c] == 0.0f) { tMax[c] = PINF; tDelta[c] = PINF; }
		//	if (boundaryPos[c] - rayMin[c] == 0.0f) { tMax[c] = PINF; tDelta[c] = PINF; }
		//}
		if (rayDir.x == 0.0f) { tMax.x = PINF; tDelta.x = PINF; }
		if (boundaryPos.x - rayMin.x == 0.0f) { tMax.x = PINF; tDelta.x = PINF; }

		if (rayDir.y == 0.0f) { tMax.y = PINF; tDelta.y = PINF; }
		if (boundaryPos.y - rayMin.y == 0.0f) { tMax.y = PINF; tDelta.y = PINF; }

		if (rayDir.z == 0.0f) { tMax.z = PINF; tDelta.z = PINF; }
		if (boundaryPos.z - rayMin.z == 0.0f) { tMax.z = PINF; tDelta.z = PINF; }


		unsigned int iter = 0; // iter < g_MaxLoopIterCount
		unsigned int g_MaxLoopIterCount = 1024;
#pragma unroll 1
		while(iter < g_MaxLoopIterCount) {

			//check if it's in the frustum and not checked out
			if (hashData.isSDFBlockInCameraFrustumApprox(idCurrentVoxel)) { //} && !isSDFBlockStreamedOut(idCurrentVoxel, hashData, d_bitMask)) {		
				hashData.allocBlock(idCurrentVoxel, cameraParams.flags & 0xFF);
				//printf("Allocate block: %d\n",idCurrentVoxel.x);
			}

			// Traverse voxel grid
			if(tMax.x < tMax.y && tMax.x < tMax.z)	{
				idCurrentVoxel.x += step.x;
				if(idCurrentVoxel.x == idBound.x) return;
				tMax.x += tDelta.x;
			}
			else if(tMax.z < tMax.y) {
				idCurrentVoxel.z += step.z;
				if(idCurrentVoxel.z == idBound.z) return;
				tMax.z += tDelta.z;
			}
			else	{
				idCurrentVoxel.y += step.y;
				if(idCurrentVoxel.y == idBound.y) return;
				tMax.y += tDelta.y;
			}

			iter++;
		}
	}
}

extern "C" void allocCUDA(HashData& hashData, const HashParams& hashParams, const DepthCameraData& depthCameraData, const DepthCameraParams& depthCameraParams, const unsigned int* d_bitMask) 
{

	//printf("Allocating: %d\n",depthCameraParams.m_imageWidth);

	const dim3 gridSize((depthCameraParams.m_imageWidth + T_PER_BLOCK - 1)/T_PER_BLOCK, (depthCameraParams.m_imageHeight + T_PER_BLOCK - 1)/T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	allocKernel<<<gridSize, blockSize>>>(hashData, depthCameraData, d_bitMask);

	//cudaSafeCall(hipDeviceSynchronize());

	#ifdef _DEBUG
		cudaSafeCall(hipDeviceSynchronize());
		//cutilCheckMsg(__FUNCTION__);
	#endif
}



__global__ void fillDecisionArrayKernel(HashData hashData, DepthCameraData depthCameraData) 
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		hashData.d_hashDecision[idx] = 0;
		if (hashData.d_hash[idx].ptr != FREE_ENTRY) {
			if (hashData.isSDFBlockInCameraFrustumApprox(hashData.d_hash[idx].pos)) {
				hashData.d_hashDecision[idx] = 1;	//yes
			}
		}
	}
}

extern "C" void fillDecisionArrayCUDA(HashData& hashData, const HashParams& hashParams, const DepthCameraData& depthCameraData)
{
	const dim3 gridSize((HASH_BUCKET_SIZE * hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
	const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

	fillDecisionArrayKernel<<<gridSize, blockSize>>>(hashData, depthCameraData);

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif

}

__global__ void compactifyHashKernel(HashData hashData) 
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		if (hashData.d_hashDecision[idx] == 1) {
			hashData.d_hashCompactified[hashData.d_hashDecisionPrefix[idx]-1] = hashData.d_hash[idx];
		}
	}
}

extern "C" void compactifyHashCUDA(HashData& hashData, const HashParams& hashParams) 
{
	const dim3 gridSize((HASH_BUCKET_SIZE * hashParams.m_hashNumBuckets + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
	const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

	compactifyHashKernel<<<gridSize, blockSize>>>(hashData);

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}


#define COMPACTIFY_HASH_THREADS_PER_BLOCK 256
//#define COMPACTIFY_HASH_SIMPLE
__global__ void compactifyHashAllInOneKernel(HashData hashData)
{
	const HashParams& hashParams = c_hashParams;
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
#ifdef COMPACTIFY_HASH_SIMPLE
	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		if (hashData.d_hash[idx].ptr != FREE_ENTRY) {
			if (hashData.isSDFBlockInCameraFrustumApprox(hashData.d_hash[idx].pos))
			{
				int addr = atomicAdd(hashData.d_hashCompactifiedCounter, 1);
				hashData.d_hashCompactified[addr] = hashData.d_hash[idx];
			}
		}
	}
#else	
	__shared__ int localCounter;
	if (threadIdx.x == 0) localCounter = 0;
	__syncthreads();

	int addrLocal = -1;
	if (idx < hashParams.m_hashNumBuckets * HASH_BUCKET_SIZE) {
		if (hashData.d_hash[idx].ptr != FREE_ENTRY) {
			if (hashData.isSDFBlockInCameraFrustumApprox(hashData.d_hash[idx].pos))
			{
				addrLocal = atomicAdd(&localCounter, 1);
			}
		}
	}

	__syncthreads();

	__shared__ int addrGlobal;
	if (threadIdx.x == 0 && localCounter > 0) {
		addrGlobal = atomicAdd(hashData.d_hashCompactifiedCounter, localCounter);
	}
	__syncthreads();

	if (addrLocal != -1) {
		const unsigned int addr = addrGlobal + addrLocal;
		hashData.d_hashCompactified[addr] = hashData.d_hash[idx];
	}
#endif
}

extern "C" unsigned int compactifyHashAllInOneCUDA(HashData& hashData, const HashParams& hashParams)
{
	const unsigned int threadsPerBlock = COMPACTIFY_HASH_THREADS_PER_BLOCK;
	const dim3 gridSize((HASH_BUCKET_SIZE * hashParams.m_hashNumBuckets + threadsPerBlock - 1) / threadsPerBlock, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	cudaSafeCall(hipMemset(hashData.d_hashCompactifiedCounter, 0, sizeof(int)));
	compactifyHashAllInOneKernel << <gridSize, blockSize >> >(hashData);
	unsigned int res = 0;
	cudaSafeCall(hipMemcpy(&res, hashData.d_hashCompactifiedCounter, sizeof(unsigned int), hipMemcpyDeviceToHost));

#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
	return res;
}

__device__ float4 make_float4(uchar4 c) {
	return make_float4(static_cast<float>(c.x), static_cast<float>(c.y), static_cast<float>(c.z), static_cast<float>(c.w));
}

inline __device__ uchar4 bilinearFilterColor(const float2& screenPos, hipTextureObject_t colorTextureRef) {
	const DepthCameraParams& cameraParams = c_depthCameraParams;
	const int imageWidth = cameraParams.m_imageWidth;
	const int imageHeight = cameraParams.m_imageHeight;
	const int2 p00 = make_int2(screenPos.x+0.5f, screenPos.y+0.5f);
	const int2 dir = sign(make_float2(screenPos.x - p00.x, screenPos.y - p00.y));

	const int2 p01 = p00 + make_int2(0.0f, dir.y);
	const int2 p10 = p00 + make_int2(dir.x, 0.0f);
	const int2 p11 = p00 + make_int2(dir.x, dir.y);

	const float alpha = (screenPos.x - p00.x)*dir.x;
	const float beta  = (screenPos.y - p00.y)*dir.y;

	float4 s0 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float w0 = 0.0f;
	if(p00.x >= 0 && p00.x < imageWidth && p00.y >= 0 && p00.y < imageHeight) { uchar4 v00 = tex2D<uchar4>(colorTextureRef, p00.x, p00.y); if(v00.x != 0) { s0 += (1.0f-alpha)*make_float4(v00); w0 += (1.0f-alpha); } }
	if(p10.x >= 0 && p10.x < imageWidth && p10.y >= 0 && p10.y < imageHeight) { uchar4 v10 = tex2D<uchar4>(colorTextureRef, p10.x, p10.y); if(v10.x != 0) { s0 +=		 alpha *make_float4(v10); w0 +=		 alpha ; } }

	float4 s1 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float w1 = 0.0f;
	if(p01.x >= 0 && p01.x < imageWidth && p01.y >= 0 && p01.y < imageHeight) { uchar4 v01 = tex2D<uchar4>(colorTextureRef, p01.x, p01.y); if(v01.x != 0) { s1 += (1.0f-alpha)*make_float4(v01); w1 += (1.0f-alpha);} }
	if(p11.x >= 0 && p11.x < imageWidth && p11.y >= 0 && p11.y < imageHeight) { uchar4 v11 = tex2D<uchar4>(colorTextureRef, p11.x, p11.y); if(v11.x != 0) { s1 +=		 alpha *make_float4(v11); w1 +=		 alpha ;} }

	const float4 p0 = s0/w0;
	const float4 p1 = s1/w1;

	float4 ss = make_float4(0.0f, 0.0f, 0.0f, 0.0f); float ww = 0.0f;
	if(w0 > 0.0f) { ss += (1.0f-beta)*p0; ww += (1.0f-beta); }
	if(w1 > 0.0f) { ss +=		beta *p1; ww +=		  beta ; }

	if(ww > 0.0f) {
		ss /= ww;
		return make_uchar4(ss.x,ss.y,ss.z,ss.w);
	} else		  return make_uchar4(0, 0, 0, 0);
}

__device__ float colourDistance(const uchar4 &c1, const uchar3 &c2) {
	float x = c1.x-c2.x;
	float y = c1.y-c2.y;
	float z = c1.z-c2.z;
	return x*x + y*y + z*z;
}

__global__ void integrateDepthMapKernel(HashData hashData, DepthCameraData cameraData, hipTextureObject_t depthT, hipTextureObject_t colourT) {
	const HashParams& hashParams = c_hashParams;
	const DepthCameraParams& cameraParams = c_depthCameraParams;

	//TODO check if we should load this in shared memory
	HashEntry& entry = hashData.d_hashCompactified[blockIdx.x];
	//if (entry.ptr == FREE_ENTRY) {
	//	printf("invliad integrate");
	//	return; //should never happen since we did the compactification before
	//}

	int3 pi_base = hashData.SDFBlockToVirtualVoxelPos(entry.pos);

	uint i = threadIdx.x;	//inside of an SDF block
	int3 pi = pi_base + make_int3(hashData.delinearizeVoxelIndex(i));
	float3 pf = hashData.virtualVoxelPosToWorld(pi);

	pf = hashParams.m_rigidTransformInverse * pf;
	uint2 screenPos = make_uint2(cameraData.cameraToKinectScreenInt(pf));

	// For this voxel in hash, get its screen position and check it is on screen
	if (screenPos.x < cameraParams.m_imageWidth && screenPos.y < cameraParams.m_imageHeight) {	//on screen

		//float depth = g_InputDepth[screenPos];
		float depth = tex2D<float>(depthT, screenPos.x, screenPos.y);
		//if (depth > 20.0f) return;

		uchar4 color  = make_uchar4(0, 0, 0, 0);
		//if (cameraData.d_colorData) {
			color = tex2D<uchar4>(colourT, screenPos.x, screenPos.y);
			//color = bilinearFilterColor(cameraData.cameraToKinectScreenFloat(pf));
		//}

		//printf("screen pos %d\n", color.x);
		//return;

		// Depth is within accepted max distance from camera
		if (depth > 0 && depth < hashParams.m_maxIntegrationDistance) { // valid depth and color (Nick: removed colour check)
			float depthZeroOne = cameraData.cameraToKinectProjZ(depth);

			// Calculate SDF of this voxel wrt the depth map value
			float sdf = depth - pf.z;
			float truncation = hashData.getTruncation(depth);

			// Is this voxel close enough to cam for depth map value
			// CHECK Nick: If is too close then free space violation so remove?
			// This isn't enough if the disparity has occlusions that don't cause violations
			// Could RGB changes also cause removals if depth can't be confirmed?
			if (sdf > truncation) {
				uint idx = entry.ptr + i;
				hashData.d_SDFBlocks[idx].weight = 0;
				//hashData.d_SDFBlocks[idx].sdf = PINF;
				hashData.d_SDFBlocks[idx].color = make_uchar3(0,0,0);
			}
			else if (sdf > -truncation) // && depthZeroOne >= 0.0f && depthZeroOne <= 1.0f) //check if in truncation range should already be made in depth map computation
			{
				if (sdf >= 0.0f) {
					sdf = fminf(truncation, sdf);
				} else {
					sdf = fmaxf(-truncation, sdf);
				}


				//printf("SDF: %f\n", sdf);
				//float weightUpdate = g_WeightSample;
				//weightUpdate = (1-depthZeroOne)*5.0f + depthZeroOne*0.05f;
				//weightUpdate *= g_WeightSample;
				float weightUpdate = max(hashParams.m_integrationWeightSample * 1.5f * (1.0f-depthZeroOne), 1.0f);

				Voxel curr;	//construct current voxel
				curr.sdf = sdf;
				curr.weight = weightUpdate;
				curr.color = make_uchar3(color.x, color.y, color.z);
				

				uint idx = entry.ptr + i;

				if (entry.flags != cameraParams.flags & 0xFF) {
					entry.flags = cameraParams.flags & 0xFF;
					hashData.d_SDFBlocks[idx].color = make_uchar3(0,0,0);
				}
				
				Voxel newVoxel;
				//if (color.x == MINF) hashData.combineVoxelDepthOnly(hashData.d_SDFBlocks[idx], curr, newVoxel);
				//else hashData.combineVoxel(hashData.d_SDFBlocks[idx], curr, newVoxel);
				hashData.combineVoxel(hashData.d_SDFBlocks[idx], curr, newVoxel);

				hashData.d_SDFBlocks[idx] = newVoxel;

				//Voxel prev = getVoxel(g_SDFBlocksSDFUAV, g_SDFBlocksRGBWUAV, idx);
				//Voxel newVoxel = combineVoxel(curr, prev);
				//setVoxel(g_SDFBlocksSDFUAV, g_SDFBlocksRGBWUAV, idx, newVoxel);
			}
		} else {
			uint idx = entry.ptr + i;
			float coldist = colourDistance(color, hashData.d_SDFBlocks[idx].color);
			if (depth > 40.0f && coldist > 100.0f) {
				//hashData.d_SDFBlocks[idx].color = make_uchar3(0,0,(uchar)(coldist));
				hashData.d_SDFBlocks[idx].weight = hashData.d_SDFBlocks[idx].weight >> 1;
			}
		}
	}
}


extern "C" void integrateDepthMapCUDA(HashData& hashData, const HashParams& hashParams,
		const DepthCameraData& depthCameraData, const DepthCameraParams& depthCameraParams)
{
	const unsigned int threadsPerBlock = SDF_BLOCK_SIZE*SDF_BLOCK_SIZE*SDF_BLOCK_SIZE;
	const dim3 gridSize(hashParams.m_numOccupiedBlocks, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	if (hashParams.m_numOccupiedBlocks > 0) {	//this guard is important if there is no depth in the current frame (i.e., no blocks were allocated)
		integrateDepthMapKernel << <gridSize, blockSize >> >(hashData, depthCameraData, depthCameraData.depth_obj_, depthCameraData.colour_obj_);
	}

	cudaSafeCall( hipGetLastError() );
#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}



__global__ void starveVoxelsKernel(HashData hashData) {

	const uint idx = blockIdx.x;
	const HashEntry& entry = hashData.d_hashCompactified[idx];

	//is typically exectued only every n'th frame
	int weight = hashData.d_SDFBlocks[entry.ptr + threadIdx.x].weight;
	weight = max(0, weight-1);	
	hashData.d_SDFBlocks[entry.ptr + threadIdx.x].weight = weight;  //CHECK Remove to totally clear previous frame (Nick)
}

extern "C" void starveVoxelsKernelCUDA(HashData& hashData, const HashParams& hashParams)
{
	const unsigned int threadsPerBlock = SDF_BLOCK_SIZE*SDF_BLOCK_SIZE*SDF_BLOCK_SIZE;
	const dim3 gridSize(hashParams.m_numOccupiedBlocks, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	if (hashParams.m_numOccupiedBlocks > 0) {
		starveVoxelsKernel << <gridSize, blockSize >> >(hashData);
	}
#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}


__shared__ float	shared_MinSDF[SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE / 2];
__shared__ uint		shared_MaxWeight[SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE / 2];


__global__ void garbageCollectIdentifyKernel(HashData hashData) {

	const DepthCameraParams& cameraParams = c_depthCameraParams;
	const unsigned int hashIdx = blockIdx.x;
	const HashEntry& entry = hashData.d_hashCompactified[hashIdx];

	// Entire block was not touched in this frame, so remove (Nick)
	/*if (entry.flags != cameraParams.flags & 0xFF) {
		hashData.d_hashDecision[hashIdx] = 1;
		return;
	}*/
	
	//uint h = hashData.computeHashPos(entry.pos);
	//hashData.d_hashDecision[hashIdx] = 1;
	//if (hashData.d_hashBucketMutex[h] == LOCK_ENTRY)	return;

	//if (entry.ptr == FREE_ENTRY) return; //should never happen since we did compactify before
	//const uint linBlockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	const unsigned int idx0 = entry.ptr + 2*threadIdx.x+0;
	const unsigned int idx1 = entry.ptr + 2*threadIdx.x+1;

	Voxel v0 = hashData.d_SDFBlocks[idx0];
	Voxel v1 = hashData.d_SDFBlocks[idx1];

	if (v0.weight == 0)	v0.sdf = PINF;
	if (v1.weight == 0)	v1.sdf = PINF;

	shared_MinSDF[threadIdx.x] = min(fabsf(v0.sdf), fabsf(v1.sdf));	//init shared memory
	shared_MaxWeight[threadIdx.x] = max(v0.weight, v1.weight);
		
#pragma unroll 1
	for (uint stride = 2; stride <= blockDim.x; stride <<= 1) {
		__syncthreads();
		if ((threadIdx.x  & (stride-1)) == (stride-1)) {
			shared_MinSDF[threadIdx.x] = min(shared_MinSDF[threadIdx.x-stride/2], shared_MinSDF[threadIdx.x]);
			shared_MaxWeight[threadIdx.x] = max(shared_MaxWeight[threadIdx.x-stride/2], shared_MaxWeight[threadIdx.x]);
		}
	}

	__syncthreads();

	if (threadIdx.x == blockDim.x - 1) {
		float minSDF = shared_MinSDF[threadIdx.x];
		uint maxWeight = shared_MaxWeight[threadIdx.x];

		float t = hashData.getTruncation(c_depthCameraParams.m_sensorDepthWorldMax);	//MATTHIAS TODO check whether this is a reasonable metric

		if (minSDF >= t || maxWeight == 0) {
			hashData.d_hashDecision[hashIdx] = 1;
		} else {
			hashData.d_hashDecision[hashIdx] = 0; 
		}
	}
}
 
extern "C" void garbageCollectIdentifyCUDA(HashData& hashData, const HashParams& hashParams) {
	
	const unsigned int threadsPerBlock = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE / 2;
	const dim3 gridSize(hashParams.m_numOccupiedBlocks, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	if (hashParams.m_numOccupiedBlocks > 0) {
		garbageCollectIdentifyKernel << <gridSize, blockSize >> >(hashData);
	}
#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}


__global__ void garbageCollectFreeKernel(HashData hashData) {

	//const uint hashIdx = blockIdx.x;
	const uint hashIdx = blockIdx.x*blockDim.x + threadIdx.x;


	if (hashIdx < c_hashParams.m_numOccupiedBlocks && hashData.d_hashDecision[hashIdx] != 0) {	//decision to delete the hash entry

		const HashEntry& entry = hashData.d_hashCompactified[hashIdx];
		//if (entry.ptr == FREE_ENTRY) return; //should never happen since we did compactify before

		if (hashData.deleteHashEntryElement(entry.pos)) {	//delete hash entry from hash (and performs heap append)
			const uint linBlockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

			#pragma unroll 1
			for (uint i = 0; i < linBlockSize; i++) {	//clear sdf block: CHECK TODO another kernel?
				hashData.deleteVoxel(entry.ptr + i);
			}
		}
	}
}


extern "C" void garbageCollectFreeCUDA(HashData& hashData, const HashParams& hashParams) {
	
	const unsigned int threadsPerBlock = T_PER_BLOCK*T_PER_BLOCK;
	const dim3 gridSize((hashParams.m_numOccupiedBlocks + threadsPerBlock - 1) / threadsPerBlock, 1);
	const dim3 blockSize(threadsPerBlock, 1);
	
	if (hashParams.m_numOccupiedBlocks > 0) {
		garbageCollectFreeKernel << <gridSize, blockSize >> >(hashData);
	}
#ifdef _DEBUG
	cudaSafeCall(hipDeviceSynchronize());
	//cutilCheckMsg(__FUNCTION__);
#endif
}
